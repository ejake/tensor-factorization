#include "hip/hip_runtime.h"
#include <Python.h>
#include <iostream>
#include "theano_mod_helper.h"
#include "cuda_ndarray.cuh"
//////////////////////
////  Support Code
//////////////////////


#define INTDIV_POW2(a, b) (a >> b)
#define INTMOD_POW2(a, b) (a & ((1<<b)-1))
        // GpuElemwise{Sub}[(0, 0)]
// node.op.destroy_map={0: [0]}
//    Input   0 CudaNdarrayType(float32, matrix)
//    Input   1 CudaNdarrayType(float32, matrix)
//    Output  0 CudaNdarrayType(float32, matrix)
static __global__ void kernel_Sub_node_884cea2a5e11fb3f992663ca999afaa3_0_1(unsigned int numEls
	, const int dim0
	, const float * i0_data, int i0_str_0
	, const float * i1_data, int i1_str_0
	, float * o0_data, int o0_str_0
	)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    for (int i = idx; i < numEls; i += numThreads) {
        int ii = i;
        const float * ii_i0_data = i0_data;
        const float * ii_i1_data = i1_data;
        float * ii_o0_data = o0_data;
        int pos0 = ii;
        ii_i0_data += pos0 * i0_str_0;
        ii_i1_data += pos0 * i1_str_0;
        ii_o0_data += pos0 * o0_str_0;
npy_float32 o0_i;
        o0_i = ii_i0_data[0] - ii_i1_data[0];
ii_o0_data[0] = o0_i;
    }
}
// GpuElemwise{Sub}[(0, 0)]
// node.op.destroy_map={0: [0]}
//    Input   0 CudaNdarrayType(float32, matrix)
//    Input   1 CudaNdarrayType(float32, matrix)
//    Output  0 CudaNdarrayType(float32, matrix)
static __global__ void kernel_Sub_node_884cea2a5e11fb3f992663ca999afaa3_0_2(unsigned int numEls
	, const int dim0, const int dim1
	, const float * i0_data, int i0_str_0, int i0_str_1
	, const float * i1_data, int i1_str_0, int i1_str_1
	, float * o0_data, int o0_str_0, int o0_str_1
	)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    for (int i = idx; i < numEls; i += numThreads) {
        int ii = i;
        const float * ii_i0_data = i0_data;
        const float * ii_i1_data = i1_data;
        float * ii_o0_data = o0_data;
        int pos1 = ii % dim1;
        ii = ii / dim1;
        ii_i0_data += pos1 * i0_str_1;
        ii_i1_data += pos1 * i1_str_1;
        ii_o0_data += pos1 * o0_str_1;
        int pos0 = ii;
        ii_i0_data += pos0 * i0_str_0;
        ii_i1_data += pos0 * i1_str_0;
        ii_o0_data += pos0 * o0_str_0;
npy_float32 o0_i;
        o0_i = ii_i0_data[0] - ii_i1_data[0];
ii_o0_data[0] = o0_i;
    }
}
// GpuElemwise{Sub}[(0, 0)]
// node.op.destroy_map={0: [0]}
//    Input   0 CudaNdarrayType(float32, matrix)
//    Input   1 CudaNdarrayType(float32, matrix)
//    Output  0 CudaNdarrayType(float32, matrix)
static __global__ void kernel_Sub_node_884cea2a5e11fb3f992663ca999afaa3_0_Ccontiguous (unsigned int numEls
	, const float * i0_data
	, const float * i1_data
	, float * o0_data
	)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    for (int i = idx; i < numEls; i += numThreads) {
npy_float32 o0_i;
        o0_i = i0_data[i] - i1_data[i];
o0_data[i] = o0_i;
    }
}

        static void can_collapse_node_884cea2a5e11fb3f992663ca999afaa3_0(int nd, const int * dims, const int * strides, int collapse[])
        {
            //can we collapse dims[i] and dims[i-1]
            for(int i=nd-1;i>0;i--){
                if(strides[i]*dims[i]==strides[i-1]){//the dims nd-1 are not strided again dimension nd
                    collapse[i]=1;
                }else collapse[i]=0;
            }
        }
        

        static int callkernel_node_884cea2a5e11fb3f992663ca999afaa3_0(unsigned int numEls, const int d,
            const int * dims,
            const float * i0_data, const int * i0_str, const float * i1_data, const int * i1_str,
            float * o0_data, const int * o0_str)
        {
            numEls = dims[0]*dims[1]*1;
        
int local_dims[2];

            int local_str[2][2];
            int local_ostr[1][2];
            

        int nd_collapse = 2;
        for(int i=0;i<2;i++){//init new dim
          local_dims[i]=dims[i];
        }
        

            for(int i=0;i<2;i++){//init new strides
              local_str[0][i]=i0_str[i];
            }
            

            for(int i=0;i<2;i++){//init new strides
              local_str[1][i]=i1_str[i];
            }
            

            for(int i=0;i<2;i++){//init new strides
              local_ostr[0][i]=o0_str[i];
            }
            

        for(int id=0;id<nd_collapse;id++){

          bool all_broadcast=true;
          for(int input_id=0;input_id<2;input_id++){
            if(local_str[input_id][id]!=0 || local_dims[id]!=1) all_broadcast= false;
          }
          for(int input_id=0;input_id<1;input_id++){
            if(local_ostr[input_id][id]!=0 || local_dims[id]!=1) all_broadcast= false;
          }
          if(all_broadcast){
            for(int j=id+1;j<nd_collapse;j++)//remove dims i from the array
              local_dims[j-1]=local_dims[j];
            for(int input_id=0;input_id<2;input_id++){
              for(int j=id+1;j<nd_collapse;j++){//remove dims i from the array
                local_str[input_id][j-1]=local_str[input_id][j];
              }
            }
            for(int output_id=0;output_id<1;output_id++){
              for(int j=id+1;j<nd_collapse;j++){//remove dims i from the array
                local_ostr[output_id][j-1]=local_ostr[output_id][j];
              }
            }
            nd_collapse--; id--;
          }
        }
        
int nd_collapse_[2] = {1,1};

                        int nd_collapse_0[2] = {1,1};

can_collapse_node_884cea2a5e11fb3f992663ca999afaa3_0(nd_collapse, local_dims, local_str[0], nd_collapse_0);
for(int i=0;i<nd_collapse;i++){
if(nd_collapse_0[i]==0)
nd_collapse_[i]=0;
}
                

                        int nd_collapse_1[2] = {1,1};

can_collapse_node_884cea2a5e11fb3f992663ca999afaa3_0(nd_collapse, local_dims, local_str[1], nd_collapse_1);
for(int i=0;i<nd_collapse;i++){
if(nd_collapse_1[i]==0)
nd_collapse_[i]=0;
}
                

            for(int i=nd_collapse-1;i>0;i--){
              if(nd_collapse_[i]==1){
                local_str[0][i-1]=local_str[0][i];//set new strides
                for(int j=i+1;j<nd_collapse;j++)//remove stride i from the array
                  local_str[0][j-1]=local_str[0][j];
                }
            }
            

            for(int i=nd_collapse-1;i>0;i--){
              if(nd_collapse_[i]==1){
                local_str[1][i-1]=local_str[1][i];//set new strides
                for(int j=i+1;j<nd_collapse;j++)//remove stride i from the array
                  local_str[1][j-1]=local_str[1][j];
                }
            }
            

            for(int i=nd_collapse-1;i>0;i--){
              if(nd_collapse_[i]==1){
                local_ostr[0][i-1]=local_ostr[0][i];//set new strides
                for(int j=i+1;j<nd_collapse;j++)//remove stride i from the array
                  local_ostr[0][j-1]=local_ostr[0][j];
                }
            }
            

        for(int i=nd_collapse-1;i>0;i--){
          if(nd_collapse_[i]==1){
            local_dims[i-1]*=local_dims[i];//set new dims
            for(int j=i+1;j<nd_collapse;j++)//remove dims i from the array
              local_dims[j-1]=local_dims[j];
          }
        }
        

        for(int i=1, end=nd_collapse;i<end;i++){
          if(nd_collapse_[i]==1)nd_collapse--;
        }
        if(nd_collapse == 1 
 &&  local_str[0][nd_collapse-1]==1  && local_str[1][nd_collapse-1]==1  && local_ostr[0][nd_collapse-1]==1 
){nd_collapse=0;} 
if(numEls==0) return 0;
switch (nd_collapse==0?0:min(2,nd_collapse)) {
case 0: {

                //first use at least a full warp
                int threads_per_block = std::min(numEls,  (unsigned int)32); //WARP SIZE

                //next start adding multiprocessors
                int n_blocks = std::min(numEls/threads_per_block + (numEls % threads_per_block?1:0), (unsigned int)30); // UP TO NUMBER OF MULTIPROCESSORS

                // next start adding more warps per multiprocessor
                if (threads_per_block * n_blocks < numEls)
                    threads_per_block = std::min(numEls/n_blocks, (unsigned int)NUM_VECTOR_OP_THREADS_PER_BLOCK);
                kernel_Sub_node_884cea2a5e11fb3f992663ca999afaa3_0_Ccontiguous<<<n_blocks, threads_per_block>>>(numEls, i0_data, i1_data, o0_data);

                //std::cerr << "calling callkernel returned\n";
                

                CNDA_THREAD_SYNC;
                hipError_t err = hipGetLastError();
                if( hipSuccess != err)
                {
                    PyErr_Format(PyExc_RuntimeError, "Cuda error: %s: %s.\n    n_blocks=%i threads_per_block=%i\n   Call: %s\n",
                         "GpuElemwise node_884cea2a5e11fb3f992663ca999afaa3_0 Sub", hipGetErrorString(err),
                         n_blocks, threads_per_block,
                         "kernel_Sub_node_884cea2a5e11fb3f992663ca999afaa3_0_Ccontiguous<<<n_blocks, threads_per_block>>>(numEls, i0_data, i1_data, o0_data)");
                    return -1;

                }
                
                return 0;
                
        } break;
case 1: {

                //first use at least a full warp
                int threads_per_block = std::min(numEls, (unsigned int)32); //WARP SIZE

                //next start adding multiprocessors
                int n_blocks = std::min(numEls/threads_per_block + (numEls % threads_per_block?1:0), (unsigned int)30); // UP TO NUMBER OF MULTIPROCESSORS

                // next start adding more warps per multiprocessor
                if (threads_per_block * n_blocks < numEls)
                    threads_per_block = std::min(numEls/n_blocks, (unsigned int)NUM_VECTOR_OP_THREADS_PER_BLOCK);

                kernel_Sub_node_884cea2a5e11fb3f992663ca999afaa3_0_1<<<n_blocks, threads_per_block>>>(numEls, local_dims[0], i0_data, local_str[0][0], i1_data, local_str[1][0], o0_data, local_ostr[0][0]);
                

                CNDA_THREAD_SYNC;
                hipError_t err = hipGetLastError();
                if( hipSuccess != err)
                {
                    PyErr_Format(PyExc_RuntimeError, "Cuda error: %s: %s.\n    n_blocks=%i threads_per_block=%i\n   Call: %s\n",
                         "GpuElemwise node_884cea2a5e11fb3f992663ca999afaa3_0 Sub", hipGetErrorString(err),
                         n_blocks, threads_per_block,
                         "kernel_Sub_node_884cea2a5e11fb3f992663ca999afaa3_0_Ccontiguous<<<n_blocks, threads_per_block>>>(numEls, local_dims[0], i0_data, local_str[0][0], i1_data, local_str[1][0], o0_data, local_ostr[0][0])");
                    return -1;

                }
                return 0;
                
        } break;
case 2: {

                //first use at least a full warp
                int threads_per_block = std::min(numEls, (unsigned int)32); //WARP SIZE

                //next start adding multiprocessors
                int n_blocks = std::min(numEls/threads_per_block + (numEls % threads_per_block?1:0), (unsigned int)30); // UP TO NUMBER OF MULTIPROCESSORS

                // next start adding more warps per multiprocessor
                if (threads_per_block * n_blocks < numEls)
                    threads_per_block = std::min(numEls/n_blocks, (unsigned int)NUM_VECTOR_OP_THREADS_PER_BLOCK);

                kernel_Sub_node_884cea2a5e11fb3f992663ca999afaa3_0_2<<<n_blocks, threads_per_block>>>(numEls, local_dims[0], local_dims[1], i0_data, local_str[0][0], local_str[0][1], i1_data, local_str[1][0], local_str[1][1], o0_data, local_ostr[0][0], local_ostr[0][1]);
                

                CNDA_THREAD_SYNC;
                hipError_t err = hipGetLastError();
                if( hipSuccess != err)
                {
                    PyErr_Format(PyExc_RuntimeError, "Cuda error: %s: %s.\n    n_blocks=%i threads_per_block=%i\n   Call: %s\n",
                         "GpuElemwise node_884cea2a5e11fb3f992663ca999afaa3_0 Sub", hipGetErrorString(err),
                         n_blocks, threads_per_block,
                         "kernel_Sub_node_884cea2a5e11fb3f992663ca999afaa3_0_Ccontiguous<<<n_blocks, threads_per_block>>>(numEls, local_dims[0], local_dims[1], i0_data, local_str[0][0], local_str[0][1], i1_data, local_str[1][0], local_str[1][1], o0_data, local_ostr[0][0], local_ostr[0][1])");
                    return -1;

                }
                return 0;
                
        } break;
}
return -2;
}


    namespace {
    struct __struct_compiled_op_884cea2a5e11fb3f992663ca999afaa3 {
        PyObject* __ERROR;

        PyObject* storage_V3;
PyObject* storage_V5;
PyObject* storage_V1;
        

        __struct_compiled_op_884cea2a5e11fb3f992663ca999afaa3() {
            // This is only somewhat safe because we:
            //  1) Are not a virtual class
            //  2) Do not use any virtual classes in the members
            //  3) Deal with mostly POD and pointers

            // If this changes, we would have to revise this, but for
            // now I am tired of chasing segfaults because
            // initialization code had an error and some pointer has
            // a junk value.
            memset(this, 0, sizeof(*this));
        }
        ~__struct_compiled_op_884cea2a5e11fb3f992663ca999afaa3(void) {
            cleanup();
        }

        int init(PyObject* __ERROR, PyObject* storage_V3, PyObject* storage_V5, PyObject* storage_V1) {
            Py_XINCREF(storage_V3);
Py_XINCREF(storage_V5);
Py_XINCREF(storage_V1);
            this->storage_V3 = storage_V3;
this->storage_V5 = storage_V5;
this->storage_V1 = storage_V1;
            




            this->__ERROR = __ERROR;
            return 0;
        }
        void cleanup(void) {
            __label_1:

double __DUMMY_1;
__label_3:

double __DUMMY_3;
__label_5:

double __DUMMY_5;
__label_8:

double __DUMMY_8;

            Py_XDECREF(this->storage_V3);
Py_XDECREF(this->storage_V5);
Py_XDECREF(this->storage_V1);
        }
        int run(void) {
            int __failure = 0;
            
    PyObject* py_V1;
     CudaNdarray * V1;
    PyObject* py_V3;
     CudaNdarray * V3;
    PyObject* py_V5;
     CudaNdarray * V5;
{

    py_V1 = PyList_GET_ITEM(storage_V1, 0);
    {Py_XINCREF(py_V1);}
    
        if (py_V1 == Py_None)
        {
            V1 = NULL;
        }
        else
        {
            
        assert(py_V1->ob_refcnt >= 2); // There should be at least one ref from the container object,
        // and one ref from the local scope.

        if (CudaNdarray_Check(py_V1))
        {
            //fprintf(stderr, "c_extract CNDA object w refcnt %p %i\n", py_V1, (py_V1->ob_refcnt));
            V1 = (CudaNdarray*)py_V1;
            //std::cerr << "c_extract " << V1 << '\n';
        

                if (V1->nd != 2)
                {
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has rank %i, it was supposed to have rank 2",
                                 V1->nd);
                    V1 = NULL;
                    {
        __failure = 2;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_2;};
                }
                //std::cerr << "c_extract " << V1 << " nd check passed\n";
            

                assert(V1);
                Py_INCREF(py_V1);
            }
            else if (py_V1 == Py_None)
            {
                PyErr_SetString(PyExc_TypeError,
                                "expected a CudaNdarray, not None");
                V1 = NULL;
                {
        __failure = 2;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_2;};
            }
            else
            {
                //fprintf(stderr, "FAILING c_extract CNDA object w refcnt %p %i\n", py_V1, (py_V1->ob_refcnt));
                PyErr_SetString(PyExc_TypeError, "Argument not a CudaNdarray");
                V1 = NULL;
                {
        __failure = 2;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_2;};
            }
            //std::cerr << "c_extract done " << V1 << '\n';
            

        }
        
{

    py_V3 = PyList_GET_ITEM(storage_V3, 0);
    {Py_XINCREF(py_V3);}
    
        assert(py_V3->ob_refcnt >= 2); // There should be at least one ref from the container object,
        // and one ref from the local scope.

        if (CudaNdarray_Check(py_V3))
        {
            //fprintf(stderr, "c_extract CNDA object w refcnt %p %i\n", py_V3, (py_V3->ob_refcnt));
            V3 = (CudaNdarray*)py_V3;
            //std::cerr << "c_extract " << V3 << '\n';
        

                if (V3->nd != 2)
                {
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has rank %i, it was supposed to have rank 2",
                                 V3->nd);
                    V3 = NULL;
                    {
        __failure = 4;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_4;};
                }
                //std::cerr << "c_extract " << V3 << " nd check passed\n";
            

                assert(V3);
                Py_INCREF(py_V3);
            }
            else if (py_V3 == Py_None)
            {
                PyErr_SetString(PyExc_TypeError,
                                "expected a CudaNdarray, not None");
                V3 = NULL;
                {
        __failure = 4;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_4;};
            }
            else
            {
                //fprintf(stderr, "FAILING c_extract CNDA object w refcnt %p %i\n", py_V3, (py_V3->ob_refcnt));
                PyErr_SetString(PyExc_TypeError, "Argument not a CudaNdarray");
                V3 = NULL;
                {
        __failure = 4;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_4;};
            }
            //std::cerr << "c_extract done " << V3 << '\n';
            

{

    py_V5 = PyList_GET_ITEM(storage_V5, 0);
    {Py_XINCREF(py_V5);}
    
        assert(py_V5->ob_refcnt >= 2); // There should be at least one ref from the container object,
        // and one ref from the local scope.

        if (CudaNdarray_Check(py_V5))
        {
            //fprintf(stderr, "c_extract CNDA object w refcnt %p %i\n", py_V5, (py_V5->ob_refcnt));
            V5 = (CudaNdarray*)py_V5;
            //std::cerr << "c_extract " << V5 << '\n';
        

                if (V5->nd != 2)
                {
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has rank %i, it was supposed to have rank 2",
                                 V5->nd);
                    V5 = NULL;
                    {
        __failure = 6;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_6;};
                }
                //std::cerr << "c_extract " << V5 << " nd check passed\n";
            

                assert(V5);
                Py_INCREF(py_V5);
            }
            else if (py_V5 == Py_None)
            {
                PyErr_SetString(PyExc_TypeError,
                                "expected a CudaNdarray, not None");
                V5 = NULL;
                {
        __failure = 6;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_6;};
            }
            else
            {
                //fprintf(stderr, "FAILING c_extract CNDA object w refcnt %p %i\n", py_V5, (py_V5->ob_refcnt));
                PyErr_SetString(PyExc_TypeError, "Argument not a CudaNdarray");
                V5 = NULL;
                {
        __failure = 6;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_6;};
            }
            //std::cerr << "c_extract done " << V5 << '\n';
            

{
// Op class GpuElemwise

        //std::cerr << "C_CODE Sub START\n";
        //standard elemwise size checks
            

            int dims[2] = {1,1};
            

                int broadcasts_V3[2] = {0, 0};
                

                int broadcasts_V5[2] = {0, 0};
                

        //std::cerr << "C_CODE Sub checking input V3\n";
        if (2 != V3->nd)
        {
            PyErr_Format(PyExc_TypeError,
                         "need 2 dims, not %i", V3->nd);
            {
        __failure = 7;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_7;};
        }
        for (int i = 0; i< 2; ++i)
        {
            dims[i] = (dims[i] == 1) ? CudaNdarray_HOST_DIMS(V3)[i] : dims[i];
            if ((!(broadcasts_V3[i] &&
                 CudaNdarray_HOST_DIMS(V3)[i] == 1)) &&
                (dims[i] != CudaNdarray_HOST_DIMS(V3)[i]))
            {
                //std::cerr << "C_CODE Sub checking input V3 failed\n";
                PyErr_Format(PyExc_ValueError,
                             "GpuElemwise. Input dimension mis-match. Input"
                             " 0 (indices start at 0) has shape[%i] == %i"
                             ", but the output's size on that axis is %i.",
                             i,
                             CudaNdarray_HOST_DIMS(V3)[i],
                             dims[i]
                            );
                {
        __failure = 7;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_7;};
            }
        }
            

        //std::cerr << "C_CODE Sub checking input V5\n";
        if (2 != V5->nd)
        {
            PyErr_Format(PyExc_TypeError,
                         "need 2 dims, not %i", V5->nd);
            {
        __failure = 7;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_7;};
        }
        for (int i = 0; i< 2; ++i)
        {
            dims[i] = (dims[i] == 1) ? CudaNdarray_HOST_DIMS(V5)[i] : dims[i];
            if ((!(broadcasts_V5[i] &&
                 CudaNdarray_HOST_DIMS(V5)[i] == 1)) &&
                (dims[i] != CudaNdarray_HOST_DIMS(V5)[i]))
            {
                //std::cerr << "C_CODE Sub checking input V5 failed\n";
                PyErr_Format(PyExc_ValueError,
                             "GpuElemwise. Input dimension mis-match. Input"
                             " 1 (indices start at 0) has shape[%i] == %i"
                             ", but the output's size on that axis is %i.",
                             i,
                             CudaNdarray_HOST_DIMS(V5)[i],
                             dims[i]
                            );
                {
        __failure = 7;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_7;};
            }
        }
            

        Py_XDECREF(V1);
        V1 = V3;
        Py_INCREF(V1);
        for (int i = 0; (i< 2) && (V1); ++i) {
            if (dims[i] != CudaNdarray_HOST_DIMS(V1)[i])
            {
                PyErr_Format(PyExc_ValueError,
                             "GpuElemwise. Output dimension mis-match. Output"
                             " 0 (indices start at 0), working inplace"
                             " on input 0, has shape[%i] == %i"
                             ", but the output's size on that axis is %i.",
                             i,
                             CudaNdarray_HOST_DIMS(V1)[i],
                             dims[i]
                            );
                Py_DECREF(V1);
                V1 = NULL;
                {
        __failure = 7;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_7;};
            }
        }
        //std::cerr << "ELEMWISE NEW V1 nd" << V1->nd << "\n";
        //std::cerr << "ELEMWISE NEW V1 data" << V1->devdata << "\n";
        

        {
            //new block so that failure gotos don't skip over variable initialization
            //std::cerr << "calling callkernel\n";
            if (callkernel_node_884cea2a5e11fb3f992663ca999afaa3_0(1, 0, dims
            

                        , CudaNdarray_DEV_DATA(V3), CudaNdarray_HOST_STRIDES(V3)
            

                        , CudaNdarray_DEV_DATA(V5), CudaNdarray_HOST_STRIDES(V5)
            

                        , CudaNdarray_DEV_DATA(V1), CudaNdarray_HOST_STRIDES(V1)
            

                        ))
            {
                 // error
            

                Py_DECREF(V1);
                V1 = NULL;
                

                {
        __failure = 7;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_7;};
            }
            else // no error
            {
            }
        }
        //std::cerr << "C_CODE Sub END\n";
        
__label_7:

double __DUMMY_7;

}
__label_6:

        //std::cerr << "cleanup " << py_V5 << " " << V5 << "\n";
        //fprintf(stderr, "c_cleanup CNDA py_object w refcnt %p %i\n", py_V5, (py_V5->ob_refcnt));
        if (V5)
        {
            //fprintf(stderr, "c_cleanup CNDA cn_object w refcnt %p %i\n", V5, (V5->ob_refcnt));
            Py_XDECREF(V5);
        }
        //std::cerr << "cleanup done" << py_V5 << "\n";
        
    {Py_XDECREF(py_V5);}
    
double __DUMMY_6;

}
__label_4:

        //std::cerr << "cleanup " << py_V3 << " " << V3 << "\n";
        //fprintf(stderr, "c_cleanup CNDA py_object w refcnt %p %i\n", py_V3, (py_V3->ob_refcnt));
        if (V3)
        {
            //fprintf(stderr, "c_cleanup CNDA cn_object w refcnt %p %i\n", V3, (V3->ob_refcnt));
            Py_XDECREF(V3);
        }
        //std::cerr << "cleanup done" << py_V3 << "\n";
        
    {Py_XDECREF(py_V3);}
    
double __DUMMY_4;

}
__label_2:

    if (!__failure) {
      
        //std::cerr << "sync\n";
        if (NULL == V1) {
            // failure: sync None to storage
            Py_XDECREF(py_V1);
            py_V1 = Py_None;
            Py_INCREF(py_V1);
        }
        else
        {
            if (py_V1 != (PyObject*)V1)
            {
                Py_XDECREF(py_V1);
                py_V1 = (PyObject*)V1;
                Py_INCREF(py_V1);
            }
            assert(py_V1->ob_refcnt);
        }
        
      PyObject* old = PyList_GET_ITEM(storage_V1, 0);
      {Py_XINCREF(py_V1);}
      PyList_SET_ITEM(storage_V1, 0, py_V1);
      {Py_XDECREF(old);}
    }
    
        //std::cerr << "cleanup " << py_V1 << " " << V1 << "\n";
        //fprintf(stderr, "c_cleanup CNDA py_object w refcnt %p %i\n", py_V1, (py_V1->ob_refcnt));
        if (V1)
        {
            //fprintf(stderr, "c_cleanup CNDA cn_object w refcnt %p %i\n", V1, (V1->ob_refcnt));
            Py_XDECREF(V1);
        }
        //std::cerr << "cleanup done" << py_V1 << "\n";
        
    {Py_XDECREF(py_V1);}
    
double __DUMMY_2;

}

            
        if (__failure) {
            // When there is a failure, this code puts the exception
            // in __ERROR.
            PyObject* err_type = NULL;
            PyObject* err_msg = NULL;
            PyObject* err_traceback = NULL;
            PyErr_Fetch(&err_type, &err_msg, &err_traceback);
            if (!err_type) {err_type = Py_None;Py_INCREF(Py_None);}
            if (!err_msg) {err_msg = Py_None; Py_INCREF(Py_None);}
            if (!err_traceback) {err_traceback = Py_None; Py_INCREF(Py_None);}
            PyObject* old_err_type = PyList_GET_ITEM(__ERROR, 0);
            PyObject* old_err_msg = PyList_GET_ITEM(__ERROR, 1);
            PyObject* old_err_traceback = PyList_GET_ITEM(__ERROR, 2);
            PyList_SET_ITEM(__ERROR, 0, err_type);
            PyList_SET_ITEM(__ERROR, 1, err_msg);
            PyList_SET_ITEM(__ERROR, 2, err_traceback);
            {Py_XDECREF(old_err_type);}
            {Py_XDECREF(old_err_msg);}
            {Py_XDECREF(old_err_traceback);}
        }
        // The failure code is returned to index what code block failed.
        return __failure;
        
        }
    };
    }
    

        static int __struct_compiled_op_884cea2a5e11fb3f992663ca999afaa3_executor(__struct_compiled_op_884cea2a5e11fb3f992663ca999afaa3* self) {
            return self->run();
        }

        static void __struct_compiled_op_884cea2a5e11fb3f992663ca999afaa3_destructor(void* executor, void* self) {
            delete ((__struct_compiled_op_884cea2a5e11fb3f992663ca999afaa3*)self);
        }
        
//////////////////////
////  Functions
//////////////////////
static PyObject * instantiate(PyObject * self, PyObject *argtuple) {
  assert(PyTuple_Check(argtuple));
  if (4 != PyTuple_Size(argtuple)){ 
     PyErr_Format(PyExc_TypeError, "Wrong number of arguments, expected 4, got %i", (int)PyTuple_Size(argtuple));
     return NULL;
  }
  __struct_compiled_op_884cea2a5e11fb3f992663ca999afaa3* struct_ptr = new __struct_compiled_op_884cea2a5e11fb3f992663ca999afaa3();
  if (struct_ptr->init( PyTuple_GET_ITEM(argtuple, 0),PyTuple_GET_ITEM(argtuple, 1),PyTuple_GET_ITEM(argtuple, 2),PyTuple_GET_ITEM(argtuple, 3) ) != 0) {
    delete struct_ptr;
    return NULL;
  }
  PyObject* thunk = PyCObject_FromVoidPtrAndDesc((void*)(&__struct_compiled_op_884cea2a5e11fb3f992663ca999afaa3_executor), struct_ptr, __struct_compiled_op_884cea2a5e11fb3f992663ca999afaa3_destructor);
  return thunk; }

//////////////////////
////  Module init
//////////////////////
static PyMethodDef MyMethods[] = {
	{"instantiate", instantiate, METH_VARARGS, "undocumented"} ,
	{NULL, NULL, 0, NULL}
};
PyMODINIT_FUNC init884cea2a5e11fb3f992663ca999afaa3(void){
   (void) Py_InitModule("884cea2a5e11fb3f992663ca999afaa3", MyMethods);
}
