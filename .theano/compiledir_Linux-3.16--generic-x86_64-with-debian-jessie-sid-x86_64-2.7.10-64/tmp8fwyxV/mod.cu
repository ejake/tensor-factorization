#include "hip/hip_runtime.h"
#include <Python.h>
#include <iostream>
#include "theano_mod_helper.h"
#include "cuda_ndarray.cuh"
//////////////////////
////  Support Code
//////////////////////


            static __global__ void kernel_reduce_010_node_49998ffbdefdf18d464596bd95591cdb_0(
                    const int d0,
                    const int d1,
                    const int d2,
                    const float *A, const int sA0,
                    const int sA1, const int sA2,
                    float * Z, const int sZ0, const int sZ1)
            {
                const int threadCount = blockDim.x;
                const int threadNum = threadIdx.x;
                extern __shared__ float buf[];

                if (warpSize != 32)
                {
                    return;  //TODO: set error code
                }


                for (int i0 = blockIdx.x; i0 < d0; i0 += gridDim.x)
                {
                    for (int i2 = blockIdx.y; i2 < d2; i2 += gridDim.y)
                    {
                        float myresult = 0;
                        for (int i1 = threadIdx.x; i1 < d1; i1 += blockDim.x)
                        {
                            myresult = myresult + A[i0 * sA0 + i1 * sA1 + i2 * sA2];;
                        }
                        
        __syncthreads(); // some kernel do multiple reduction.
        buf[threadNum] = myresult;
        __syncthreads();

        // rest of function is handled by one warp
        if (threadNum < warpSize)
        {
            //round up all the partial sums into the first `warpSize` elements
            for (int i = threadNum + warpSize; i < threadCount; i += warpSize)
            {
                myresult = myresult + buf[i];
            }
            buf[threadNum] = myresult;
        /*Comment this optimization as it don't work on Fermi GPU.
        TODO: find why it don't work or put the GPU compute capability into the version
            // no sync because only one warp is running
            if(threadCount >32)
            {buf[threadNum] = buf[threadNum] + buf[threadNum+16];buf[threadNum] = buf[threadNum] + buf[threadNum+8];buf[threadNum] = buf[threadNum] + buf[threadNum+4];buf[threadNum] = buf[threadNum] + buf[threadNum+2];buf[threadNum] = buf[threadNum] + buf[threadNum+1];
                if (threadNum == 0)
                {
                    Z[i0 * sZ0 + i2*sZ1] = buf[0];
                }

            }
            else */
            if (threadNum < 16)
            {
                //reduce so that threadNum 0 has the reduction of everything
                if (threadNum + 16 < threadCount) buf[threadNum] = buf[threadNum] + buf[threadNum+16];if (threadNum + 8 < threadCount) buf[threadNum] = buf[threadNum] + buf[threadNum+8];if (threadNum + 4 < threadCount) buf[threadNum] = buf[threadNum] + buf[threadNum+4];if (threadNum + 2 < threadCount) buf[threadNum] = buf[threadNum] + buf[threadNum+2];if (threadNum + 1 < threadCount) buf[threadNum] = buf[threadNum] + buf[threadNum+1];
                if (threadNum == 0)
                {
                    Z[i0 * sZ0 + i2*sZ1] = buf[0];
                }
            }
        }
        
                    }
                }

            }
            

            static __global__ void kernel_reduce_010_AD_node_49998ffbdefdf18d464596bd95591cdb_0(
                    const int A,
                    const int B,
                    const int C,
                    const int D,
                    //const int E, // THIS is 32
                    const float *X, const int sX0,
                    const int sX1, const int sX2,
                    float * Z, const int sZ0, const int sZ1)
            {
                const int threadCount = blockDim.x;
                const int threadNum = threadIdx.x;
                float myresult = 0.0f;

                if (warpSize != 32)
                {
                    return;  //TODO: set error code
                }

                for (int a = blockIdx.x; a < A; a += gridDim.x)
                {
                    for (int i2_D = blockIdx.y; i2_D < D; i2_D += gridDim.y)
                    {
                        int c = i2_D * 32 + threadIdx.x;
                        if (c < C)
                        {
                            myresult = 0;
                            for (int b = 0; b < B; ++b)
                            {
                                myresult = myresult + X[a * sX0 + b * sX1 + c * sX2];;
                            }
                            Z[a * sZ0 + c * sZ1] = myresult;
                        }
                    }
                }

            }
            


    namespace {
    struct __struct_compiled_op_49998ffbdefdf18d464596bd95591cdb {
        PyObject* __ERROR;

        PyObject* storage_V3;
PyObject* storage_V1;
        

        __struct_compiled_op_49998ffbdefdf18d464596bd95591cdb() {
            // This is only somewhat safe because we:
            //  1) Are not a virtual class
            //  2) Do not use any virtual classes in the members
            //  3) Deal with mostly POD and pointers

            // If this changes, we would have to revise this, but for
            // now I am tired of chasing segfaults because
            // initialization code had an error and some pointer has
            // a junk value.
            memset(this, 0, sizeof(*this));
        }
        ~__struct_compiled_op_49998ffbdefdf18d464596bd95591cdb(void) {
            cleanup();
        }

        int init(PyObject* __ERROR, PyObject* storage_V3, PyObject* storage_V1) {
            Py_XINCREF(storage_V3);
Py_XINCREF(storage_V1);
            this->storage_V3 = storage_V3;
this->storage_V1 = storage_V1;
            



            this->__ERROR = __ERROR;
            return 0;
        }
        void cleanup(void) {
            __label_1:

double __DUMMY_1;
__label_3:

double __DUMMY_3;
__label_6:

double __DUMMY_6;

            Py_XDECREF(this->storage_V3);
Py_XDECREF(this->storage_V1);
        }
        int run(void) {
            int __failure = 0;
            
    PyObject* py_V1;
     CudaNdarray * V1;
    PyObject* py_V3;
     CudaNdarray * V3;
{

    py_V1 = PyList_GET_ITEM(storage_V1, 0);
    {Py_XINCREF(py_V1);}
    
        if (py_V1 == Py_None)
        {
            V1 = NULL;
        }
        else
        {
            
        assert(py_V1->ob_refcnt >= 2); // There should be at least one ref from the container object,
        // and one ref from the local scope.

        if (CudaNdarray_Check(py_V1))
        {
            //fprintf(stderr, "c_extract CNDA object w refcnt %p %i\n", py_V1, (py_V1->ob_refcnt));
            V1 = (CudaNdarray*)py_V1;
            //std::cerr << "c_extract " << V1 << '\n';
        

                if (V1->nd != 1)
                {
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has rank %i, it was supposed to have rank 1",
                                 V1->nd);
                    V1 = NULL;
                    {
        __failure = 2;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_2;};
                }
                //std::cerr << "c_extract " << V1 << " nd check passed\n";
            

                assert(V1);
                Py_INCREF(py_V1);
            }
            else if (py_V1 == Py_None)
            {
                PyErr_SetString(PyExc_TypeError,
                                "expected a CudaNdarray, not None");
                V1 = NULL;
                {
        __failure = 2;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_2;};
            }
            else
            {
                //fprintf(stderr, "FAILING c_extract CNDA object w refcnt %p %i\n", py_V1, (py_V1->ob_refcnt));
                PyErr_SetString(PyExc_TypeError, "Argument not a CudaNdarray");
                V1 = NULL;
                {
        __failure = 2;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_2;};
            }
            //std::cerr << "c_extract done " << V1 << '\n';
            

        }
        
{

    py_V3 = PyList_GET_ITEM(storage_V3, 0);
    {Py_XINCREF(py_V3);}
    
        assert(py_V3->ob_refcnt >= 2); // There should be at least one ref from the container object,
        // and one ref from the local scope.

        if (CudaNdarray_Check(py_V3))
        {
            //fprintf(stderr, "c_extract CNDA object w refcnt %p %i\n", py_V3, (py_V3->ob_refcnt));
            V3 = (CudaNdarray*)py_V3;
            //std::cerr << "c_extract " << V3 << '\n';
        

                if (V3->nd != 2)
                {
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has rank %i, it was supposed to have rank 2",
                                 V3->nd);
                    V3 = NULL;
                    {
        __failure = 4;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_4;};
                }
                //std::cerr << "c_extract " << V3 << " nd check passed\n";
            

                assert(V3);
                Py_INCREF(py_V3);
            }
            else if (py_V3 == Py_None)
            {
                PyErr_SetString(PyExc_TypeError,
                                "expected a CudaNdarray, not None");
                V3 = NULL;
                {
        __failure = 4;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_4;};
            }
            else
            {
                //fprintf(stderr, "FAILING c_extract CNDA object w refcnt %p %i\n", py_V3, (py_V3->ob_refcnt));
                PyErr_SetString(PyExc_TypeError, "Argument not a CudaNdarray");
                V3 = NULL;
                {
        __failure = 4;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_4;};
            }
            //std::cerr << "c_extract done " << V3 << '\n';
            

{
// Op class GpuCAReduce

        if (V3->nd != 2)
        {
            PyErr_Format(PyExc_TypeError,
                         "required nd=2, got nd=%i", V3->nd);
            {
        __failure = 5;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_5;};
        }
        

        if (  !V1
           || (V1->nd != 1)
        
 || (CudaNdarray_HOST_DIMS(V1)[0] != CudaNdarray_HOST_DIMS(V3)[1]) 

           )
        {
            
int new_dims[1]; 
new_dims[0] = CudaNdarray_HOST_DIMS(V3)[1];

            Py_XDECREF(V1);
            V1 = (CudaNdarray*) CudaNdarray_NewDims(1, new_dims);
            if (NULL == V1)
            {
                {
        __failure = 5;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_5;};
            }
        }
        

        if (CudaNdarray_SIZE(V1) && ! CudaNdarray_SIZE(V3)){
            hipMemset(V1->devdata, 0, CudaNdarray_SIZE(V1) * sizeof(float));
        }
        else if (CudaNdarray_SIZE(V1))
        {
        

    {
        int verbose = 0;
        if(CudaNdarray_HOST_STRIDES(V3)[0] >
           CudaNdarray_HOST_STRIDES(V3)[1]){
                // If there are a lot of summations to do, then we can use simple parallelization -
                // use each thread to do one sum.

                // we might as well launch blocks of 32 threads because that's the warp size.
                // we could schedule more threads if we were maxing out the gridsize below, but
                // the gridsize is way more than the physical hardware and I think 32 threads
                // on a huge grid is enough to fully use the hardware.
                dim3 n_threads(32,1,1);

                // We kindof reshape the input implicitly to something 4D:
                //  the shape A,B,C    ->   A, B, D, E
                //  where C <= D*E < C+32
                //  where E==32

                int A = 1;
                int B = CudaNdarray_HOST_DIMS(V3)[0];
                int C = CudaNdarray_HOST_DIMS(V3)[1];
                int D = C/32;
                if (32*D < C) D+= 1;
                assert ((C <= 32*D) && (32*D < C+32));

                // The gridsize would ideally be (A, D).  But we do the following logic to make
                // sure we don't ask for a grid that is too big.
                dim3 n_blocks(A,D);
                if (n_blocks.x > NUM_VECTOR_OP_BLOCKS) n_blocks.x = NUM_VECTOR_OP_BLOCKS;
                if (n_blocks.x*n_blocks.y > NUM_VECTOR_OP_BLOCKS) n_blocks.y = NUM_VECTOR_OP_BLOCKS/n_blocks.x;
                kernel_reduce_010_AD_node_49998ffbdefdf18d464596bd95591cdb_0<<<n_blocks, n_threads>>>(
                A,B,C,D,
                        CudaNdarray_DEV_DATA(V3),
                        1,
                        CudaNdarray_HOST_STRIDES(V3)[0],
                        CudaNdarray_HOST_STRIDES(V3)[1],
                        CudaNdarray_DEV_DATA(V1),
                        1,
                        CudaNdarray_HOST_STRIDES(V1)[0]
                        );

            CNDA_THREAD_SYNC;
            hipError_t sts = hipGetLastError();
            if (hipSuccess != sts)
            {
                PyErr_Format(PyExc_RuntimeError,
                    "Cuda error: %s: %s."
                    " (grid: %i x %i; block: %i x %i x %i)\n",
                    "kernel_reduce_10_ADnode_49998ffbdefdf18d464596bd95591cdb_0",
                    hipGetErrorString(sts),
                    n_blocks.x,
                    n_blocks.y,
                    n_threads.x,
                    n_threads.y,
                    n_threads.z);
                {
        __failure = 5;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_5;};
            }
        }else{
            dim3 n_threads(
                    std::min(CudaNdarray_HOST_DIMS(V3)[0],
                            NUM_VECTOR_OP_THREADS_PER_BLOCK));
            dim3 n_blocks(1,
                std::min(CudaNdarray_HOST_DIMS(V3)[1],
                    NUM_VECTOR_OP_BLOCKS));
            if (verbose) {
              fprintf(stderr,
                "running kernel_reduce_10_node_49998ffbdefdf18d464596bd95591cdb_0 n_blocks=(%i,%i)\n",
                n_blocks.x,
                n_blocks.y);
            }
            assert(CudaNdarray_HOST_DIMS(V3)[1] == CudaNdarray_HOST_DIMS(V1)[0]);
            int n_shared = sizeof(float) * n_threads.x;
            kernel_reduce_010_node_49998ffbdefdf18d464596bd95591cdb_0<<<n_blocks, n_threads, n_shared>>>(
                    1,
                    CudaNdarray_HOST_DIMS(V3)[0],
                    CudaNdarray_HOST_DIMS(V3)[1],
                    CudaNdarray_DEV_DATA(V3),
                    1,
                    CudaNdarray_HOST_STRIDES(V3)[0],
                    CudaNdarray_HOST_STRIDES(V3)[1],
                    CudaNdarray_DEV_DATA(V1),
                    1,
                    CudaNdarray_HOST_STRIDES(V1)[0]
                    );
            CNDA_THREAD_SYNC;
            hipError_t sts = hipGetLastError();
            if (hipSuccess != sts)
            {
                PyErr_Format(PyExc_RuntimeError,
                    "Cuda error: %s: %s."
                    " (grid: %i x %i; block: %i x %i x %i)\n",
                    "kernel_reduce_010_node_49998ffbdefdf18d464596bd95591cdb_0",
                    hipGetErrorString(sts),
                    n_blocks.x,
                    n_blocks.y,
                    n_threads.x,
                    n_threads.y,
                    n_threads.z);
                {
        __failure = 5;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_5;};
            }
        }
    }
        

        }
        
__label_5:

double __DUMMY_5;

}
__label_4:

        //std::cerr << "cleanup " << py_V3 << " " << V3 << "\n";
        //fprintf(stderr, "c_cleanup CNDA py_object w refcnt %p %i\n", py_V3, (py_V3->ob_refcnt));
        if (V3)
        {
            //fprintf(stderr, "c_cleanup CNDA cn_object w refcnt %p %i\n", V3, (V3->ob_refcnt));
            Py_XDECREF(V3);
        }
        //std::cerr << "cleanup done" << py_V3 << "\n";
        
    {Py_XDECREF(py_V3);}
    
double __DUMMY_4;

}
__label_2:

    if (!__failure) {
      
        //std::cerr << "sync\n";
        if (NULL == V1) {
            // failure: sync None to storage
            Py_XDECREF(py_V1);
            py_V1 = Py_None;
            Py_INCREF(py_V1);
        }
        else
        {
            if (py_V1 != (PyObject*)V1)
            {
                Py_XDECREF(py_V1);
                py_V1 = (PyObject*)V1;
                Py_INCREF(py_V1);
            }
            assert(py_V1->ob_refcnt);
        }
        
      PyObject* old = PyList_GET_ITEM(storage_V1, 0);
      {Py_XINCREF(py_V1);}
      PyList_SET_ITEM(storage_V1, 0, py_V1);
      {Py_XDECREF(old);}
    }
    
        //std::cerr << "cleanup " << py_V1 << " " << V1 << "\n";
        //fprintf(stderr, "c_cleanup CNDA py_object w refcnt %p %i\n", py_V1, (py_V1->ob_refcnt));
        if (V1)
        {
            //fprintf(stderr, "c_cleanup CNDA cn_object w refcnt %p %i\n", V1, (V1->ob_refcnt));
            Py_XDECREF(V1);
        }
        //std::cerr << "cleanup done" << py_V1 << "\n";
        
    {Py_XDECREF(py_V1);}
    
double __DUMMY_2;

}

            
        if (__failure) {
            // When there is a failure, this code puts the exception
            // in __ERROR.
            PyObject* err_type = NULL;
            PyObject* err_msg = NULL;
            PyObject* err_traceback = NULL;
            PyErr_Fetch(&err_type, &err_msg, &err_traceback);
            if (!err_type) {err_type = Py_None;Py_INCREF(Py_None);}
            if (!err_msg) {err_msg = Py_None; Py_INCREF(Py_None);}
            if (!err_traceback) {err_traceback = Py_None; Py_INCREF(Py_None);}
            PyObject* old_err_type = PyList_GET_ITEM(__ERROR, 0);
            PyObject* old_err_msg = PyList_GET_ITEM(__ERROR, 1);
            PyObject* old_err_traceback = PyList_GET_ITEM(__ERROR, 2);
            PyList_SET_ITEM(__ERROR, 0, err_type);
            PyList_SET_ITEM(__ERROR, 1, err_msg);
            PyList_SET_ITEM(__ERROR, 2, err_traceback);
            {Py_XDECREF(old_err_type);}
            {Py_XDECREF(old_err_msg);}
            {Py_XDECREF(old_err_traceback);}
        }
        // The failure code is returned to index what code block failed.
        return __failure;
        
        }
    };
    }
    

        static int __struct_compiled_op_49998ffbdefdf18d464596bd95591cdb_executor(__struct_compiled_op_49998ffbdefdf18d464596bd95591cdb* self) {
            return self->run();
        }

        static void __struct_compiled_op_49998ffbdefdf18d464596bd95591cdb_destructor(void* executor, void* self) {
            delete ((__struct_compiled_op_49998ffbdefdf18d464596bd95591cdb*)self);
        }
        
//////////////////////
////  Functions
//////////////////////
static PyObject * instantiate(PyObject * self, PyObject *argtuple) {
  assert(PyTuple_Check(argtuple));
  if (3 != PyTuple_Size(argtuple)){ 
     PyErr_Format(PyExc_TypeError, "Wrong number of arguments, expected 3, got %i", (int)PyTuple_Size(argtuple));
     return NULL;
  }
  __struct_compiled_op_49998ffbdefdf18d464596bd95591cdb* struct_ptr = new __struct_compiled_op_49998ffbdefdf18d464596bd95591cdb();
  if (struct_ptr->init( PyTuple_GET_ITEM(argtuple, 0),PyTuple_GET_ITEM(argtuple, 1),PyTuple_GET_ITEM(argtuple, 2) ) != 0) {
    delete struct_ptr;
    return NULL;
  }
  PyObject* thunk = PyCObject_FromVoidPtrAndDesc((void*)(&__struct_compiled_op_49998ffbdefdf18d464596bd95591cdb_executor), struct_ptr, __struct_compiled_op_49998ffbdefdf18d464596bd95591cdb_destructor);
  return thunk; }

//////////////////////
////  Module init
//////////////////////
static PyMethodDef MyMethods[] = {
	{"instantiate", instantiate, METH_VARARGS, "undocumented"} ,
	{NULL, NULL, 0, NULL}
};
PyMODINIT_FUNC init49998ffbdefdf18d464596bd95591cdb(void){
   (void) Py_InitModule("49998ffbdefdf18d464596bd95591cdb", MyMethods);
}
