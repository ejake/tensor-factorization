#include <Python.h>
#include <iostream>
#include "theano_mod_helper.h"
#include "cuda_ndarray.cuh"
#include <math.h>
#include <numpy/arrayobject.h>
#include <numpy/arrayscalars.h>
//////////////////////
////  Support Code
//////////////////////


    namespace {
    struct __struct_compiled_op_ef5673747c99044f59681d9ee3ac8b53 {
        PyObject* __ERROR;

        PyObject* storage_V3;
PyObject* storage_V5;
PyObject* storage_V7;
PyObject* storage_V1;
        

        __struct_compiled_op_ef5673747c99044f59681d9ee3ac8b53() {
            // This is only somewhat safe because we:
            //  1) Are not a virtual class
            //  2) Do not use any virtual classes in the members
            //  3) Deal with mostly POD and pointers

            // If this changes, we would have to revise this, but for
            // now I am tired of chasing segfaults because
            // initialization code had an error and some pointer has
            // a junk value.
            memset(this, 0, sizeof(*this));
        }
        ~__struct_compiled_op_ef5673747c99044f59681d9ee3ac8b53(void) {
            cleanup();
        }

        int init(PyObject* __ERROR, PyObject* storage_V3, PyObject* storage_V5, PyObject* storage_V7, PyObject* storage_V1) {
            Py_XINCREF(storage_V3);
Py_XINCREF(storage_V5);
Py_XINCREF(storage_V7);
Py_XINCREF(storage_V1);
            this->storage_V3 = storage_V3;
this->storage_V5 = storage_V5;
this->storage_V7 = storage_V7;
this->storage_V1 = storage_V1;
            





            this->__ERROR = __ERROR;
            return 0;
        }
        void cleanup(void) {
            __label_1:

double __DUMMY_1;
__label_3:

double __DUMMY_3;
__label_5:

double __DUMMY_5;
__label_7:

double __DUMMY_7;
__label_10:

double __DUMMY_10;

            Py_XDECREF(this->storage_V3);
Py_XDECREF(this->storage_V5);
Py_XDECREF(this->storage_V7);
Py_XDECREF(this->storage_V1);
        }
        int run(void) {
            int __failure = 0;
            
    PyObject* py_V1;
     CudaNdarray * V1;
    PyObject* py_V3;
     CudaNdarray * V3;
    PyObject* py_V5;
    
        PyArrayObject* V5;
        
            typedef npy_int64 dtype_V5;
            
    PyObject* py_V7;
    
        PyArrayObject* V7;
        
            typedef npy_int64 dtype_V7;
            
{

    py_V1 = PyList_GET_ITEM(storage_V1, 0);
    {Py_XINCREF(py_V1);}
    
        if (py_V1 == Py_None)
        {
            V1 = NULL;
        }
        else
        {
            
        assert(py_V1->ob_refcnt >= 2); // There should be at least one ref from the container object,
        // and one ref from the local scope.

        if (CudaNdarray_Check(py_V1))
        {
            //fprintf(stderr, "c_extract CNDA object w refcnt %p %i\n", py_V1, (py_V1->ob_refcnt));
            V1 = (CudaNdarray*)py_V1;
            //std::cerr << "c_extract " << V1 << '\n';
        

                if (V1->nd != 2)
                {
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has rank %i, it was supposed to have rank 2",
                                 V1->nd);
                    V1 = NULL;
                    {
        __failure = 2;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_2;};
                }
                //std::cerr << "c_extract " << V1 << " nd check passed\n";
            

                assert(V1);
                Py_INCREF(py_V1);
            }
            else if (py_V1 == Py_None)
            {
                PyErr_SetString(PyExc_TypeError,
                                "expected a CudaNdarray, not None");
                V1 = NULL;
                {
        __failure = 2;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_2;};
            }
            else
            {
                //fprintf(stderr, "FAILING c_extract CNDA object w refcnt %p %i\n", py_V1, (py_V1->ob_refcnt));
                PyErr_SetString(PyExc_TypeError, "Argument not a CudaNdarray");
                V1 = NULL;
                {
        __failure = 2;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_2;};
            }
            //std::cerr << "c_extract done " << V1 << '\n';
            

        }
        
{

    py_V3 = PyList_GET_ITEM(storage_V3, 0);
    {Py_XINCREF(py_V3);}
    
        assert(py_V3->ob_refcnt >= 2); // There should be at least one ref from the container object,
        // and one ref from the local scope.

        if (CudaNdarray_Check(py_V3))
        {
            //fprintf(stderr, "c_extract CNDA object w refcnt %p %i\n", py_V3, (py_V3->ob_refcnt));
            V3 = (CudaNdarray*)py_V3;
            //std::cerr << "c_extract " << V3 << '\n';
        

                if (V3->nd != 2)
                {
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has rank %i, it was supposed to have rank 2",
                                 V3->nd);
                    V3 = NULL;
                    {
        __failure = 4;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_4;};
                }
                //std::cerr << "c_extract " << V3 << " nd check passed\n";
            

                if (CudaNdarray_HOST_DIMS(V3)[0] != 1)
                {
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has dim %i on broadcastable dimension %i",
                                 CudaNdarray_HOST_DIMS(V3)[0], 0);
                    V3 = NULL;
                    {
        __failure = 4;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_4;};
                }
                //std::cerr << "c_extract " << V3 << "dim check 0 passed\n";
                //std::cerr << "c_extract " << V3 << "checking bcast 0 <" << V3->str<< ">\n";
                //std::cerr << "c_extract " << V3->str[0] << "\n";
                if (CudaNdarray_HOST_STRIDES(V3)[0])
                {
                    //std::cerr << "c_extract bad stride detected...\n";
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has a nonzero stride %i on a broadcastable dimension %i",
                                 CudaNdarray_HOST_STRIDES(V3)[0], 0);
                    V3 = NULL;
                    {
        __failure = 4;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_4;};
                }
                //std::cerr << "c_extract " << V3 << "bcast check 0 passed\n";
                    

                if (CudaNdarray_HOST_DIMS(V3)[1] != 1)
                {
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has dim %i on broadcastable dimension %i",
                                 CudaNdarray_HOST_DIMS(V3)[1], 1);
                    V3 = NULL;
                    {
        __failure = 4;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_4;};
                }
                //std::cerr << "c_extract " << V3 << "dim check 1 passed\n";
                //std::cerr << "c_extract " << V3 << "checking bcast 1 <" << V3->str<< ">\n";
                //std::cerr << "c_extract " << V3->str[1] << "\n";
                if (CudaNdarray_HOST_STRIDES(V3)[1])
                {
                    //std::cerr << "c_extract bad stride detected...\n";
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has a nonzero stride %i on a broadcastable dimension %i",
                                 CudaNdarray_HOST_STRIDES(V3)[1], 1);
                    V3 = NULL;
                    {
        __failure = 4;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_4;};
                }
                //std::cerr << "c_extract " << V3 << "bcast check 1 passed\n";
                    

                assert(V3);
                Py_INCREF(py_V3);
            }
            else if (py_V3 == Py_None)
            {
                PyErr_SetString(PyExc_TypeError,
                                "expected a CudaNdarray, not None");
                V3 = NULL;
                {
        __failure = 4;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_4;};
            }
            else
            {
                //fprintf(stderr, "FAILING c_extract CNDA object w refcnt %p %i\n", py_V3, (py_V3->ob_refcnt));
                PyErr_SetString(PyExc_TypeError, "Argument not a CudaNdarray");
                V3 = NULL;
                {
        __failure = 4;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_4;};
            }
            //std::cerr << "c_extract done " << V3 << '\n';
            

{

    py_V5 = PyList_GET_ITEM(storage_V5, 0);
    {Py_XINCREF(py_V5);}
    
            V5 = NULL;
            if (py_V5 == Py_None) {
                // We can either fail here or set V5 to NULL and rely on Ops
                // using tensors to handle the NULL case, but if they fail to do so
                // they'll end up with nasty segfaults, so this is public service.
                PyErr_SetString(PyExc_ValueError, "expected an ndarray, not None");
                {
        __failure = 6;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_6;}
            }
            if (!PyArray_Check(py_V5)) {
                PyErr_SetString(PyExc_ValueError, "expected an ndarray");
                {
        __failure = 6;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_6;}
            }
            // We expect NPY_INT64
            if (!PyArray_ISALIGNED((PyArrayObject*) py_V5)) {
                PyArrayObject * tmp = (PyArrayObject*) py_V5;
                PyErr_Format(PyExc_NotImplementedError,
                             "expected an aligned array of type %ld "
                             "(NPY_INT64), got non-aligned array of type %ld"
                             " with %ld dimensions, with 3 last dims "
                             "%ld, %ld, %ld"
                             " and 3 last strides %ld %ld, %ld.",
                             (long int) NPY_INT64,
                             (long int) PyArray_TYPE((PyArrayObject*) py_V5),
                             (long int) PyArray_NDIM(tmp),
                             (long int) PyArray_NDIM(tmp) >= 3 ?
            PyArray_DIMS(tmp)[PyArray_NDIM(tmp)-3] : -1,
                             (long int) PyArray_NDIM(tmp) >= 2 ?
            PyArray_DIMS(tmp)[PyArray_NDIM(tmp)-2] : -1,
                             (long int) PyArray_NDIM(tmp) >= 1 ?
            PyArray_DIMS(tmp)[PyArray_NDIM(tmp)-1] : -1,
                             (long int) PyArray_NDIM(tmp) >= 3 ?
            PyArray_STRIDES(tmp)[PyArray_NDIM(tmp)-3] : -1,
                             (long int) PyArray_NDIM(tmp) >= 2 ?
            PyArray_STRIDES(tmp)[PyArray_NDIM(tmp)-2] : -1,
                             (long int) PyArray_NDIM(tmp) >= 1 ?
            PyArray_STRIDES(tmp)[PyArray_NDIM(tmp)-1] : -1
            );
                {
        __failure = 6;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_6;}
            }
            // This is a TypeError to be consistent with DEBUG_MODE
            // Note: DEBUG_MODE also tells the name of the container
            if (PyArray_TYPE((PyArrayObject*) py_V5) != NPY_INT64) {
                PyErr_Format(PyExc_TypeError,
                             "expected type_num %d (NPY_INT64) got %d",
                             NPY_INT64, PyArray_TYPE((PyArrayObject*) py_V5));
                {
        __failure = 6;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_6;}
            }
            
        V5 = (PyArrayObject*)(py_V5);
        Py_XINCREF(V5);
        
{

    py_V7 = PyList_GET_ITEM(storage_V7, 0);
    {Py_XINCREF(py_V7);}
    
            V7 = NULL;
            if (py_V7 == Py_None) {
                // We can either fail here or set V7 to NULL and rely on Ops
                // using tensors to handle the NULL case, but if they fail to do so
                // they'll end up with nasty segfaults, so this is public service.
                PyErr_SetString(PyExc_ValueError, "expected an ndarray, not None");
                {
        __failure = 8;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_8;}
            }
            if (!PyArray_Check(py_V7)) {
                PyErr_SetString(PyExc_ValueError, "expected an ndarray");
                {
        __failure = 8;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_8;}
            }
            // We expect NPY_INT64
            if (!PyArray_ISALIGNED((PyArrayObject*) py_V7)) {
                PyArrayObject * tmp = (PyArrayObject*) py_V7;
                PyErr_Format(PyExc_NotImplementedError,
                             "expected an aligned array of type %ld "
                             "(NPY_INT64), got non-aligned array of type %ld"
                             " with %ld dimensions, with 3 last dims "
                             "%ld, %ld, %ld"
                             " and 3 last strides %ld %ld, %ld.",
                             (long int) NPY_INT64,
                             (long int) PyArray_TYPE((PyArrayObject*) py_V7),
                             (long int) PyArray_NDIM(tmp),
                             (long int) PyArray_NDIM(tmp) >= 3 ?
            PyArray_DIMS(tmp)[PyArray_NDIM(tmp)-3] : -1,
                             (long int) PyArray_NDIM(tmp) >= 2 ?
            PyArray_DIMS(tmp)[PyArray_NDIM(tmp)-2] : -1,
                             (long int) PyArray_NDIM(tmp) >= 1 ?
            PyArray_DIMS(tmp)[PyArray_NDIM(tmp)-1] : -1,
                             (long int) PyArray_NDIM(tmp) >= 3 ?
            PyArray_STRIDES(tmp)[PyArray_NDIM(tmp)-3] : -1,
                             (long int) PyArray_NDIM(tmp) >= 2 ?
            PyArray_STRIDES(tmp)[PyArray_NDIM(tmp)-2] : -1,
                             (long int) PyArray_NDIM(tmp) >= 1 ?
            PyArray_STRIDES(tmp)[PyArray_NDIM(tmp)-1] : -1
            );
                {
        __failure = 8;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_8;}
            }
            // This is a TypeError to be consistent with DEBUG_MODE
            // Note: DEBUG_MODE also tells the name of the container
            if (PyArray_TYPE((PyArrayObject*) py_V7) != NPY_INT64) {
                PyErr_Format(PyExc_TypeError,
                             "expected type_num %d (NPY_INT64) got %d",
                             NPY_INT64, PyArray_TYPE((PyArrayObject*) py_V7));
                {
        __failure = 8;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_8;}
            }
            
        V7 = (PyArrayObject*)(py_V7);
        Py_XINCREF(V7);
        
{
// Op class GpuAlloc
int dims[2];
dims[0] = PyInt_AsLong((PyObject*)V5);
dims[1] = PyInt_AsLong((PyObject*)V7);
if(V1==NULL
||CudaNdarray_HOST_DIMS(V1)[0]!=dims[0]||CudaNdarray_HOST_DIMS(V1)[1]!=dims[1]){
            Py_XDECREF(V1);
            V1 = (CudaNdarray*)CudaNdarray_New();
            if (!V1)
            {
                // exception already set
                {
        __failure = 9;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_9;};
            }
            if (CudaNdarray_alloc_contiguous(V1, 2, dims))
            {
                // exception already set
                Py_XDECREF(V1);
                V1 = NULL;
                {
        __failure = 9;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_9;};
            }
        }
        
        if (1 && CudaNdarray_is_c_contiguous(V1))
        {
            hipError_t err = hipMemset(V1->devdata, 0,
                                         CudaNdarray_SIZE(V1) * 4);
            if (hipSuccess != err)
            {
                PyErr_Format(PyExc_MemoryError,
                             "GpuAlloc: Error memsetting %ld"
                             " bytes of device memory. %s",
                             (long)(CudaNdarray_SIZE(V1) * 4),
                             hipGetErrorString(err));
                Py_XDECREF(V1);
                V1 = NULL;
                {
        __failure = 9;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_9;};
            }
        }
        else if (CudaNdarray_CopyFromCudaNdarray(V1, V3, true))
        {
            // exception already set
            Py_XDECREF(V1);
            V1 = NULL;
            {
        __failure = 9;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_9;};
        }
        __label_9:

double __DUMMY_9;

}
__label_8:

        if (V7) {
            Py_XDECREF(V7);
        }
        
    {Py_XDECREF(py_V7);}
    
double __DUMMY_8;

}
__label_6:

        if (V5) {
            Py_XDECREF(V5);
        }
        
    {Py_XDECREF(py_V5);}
    
double __DUMMY_6;

}
__label_4:

        //std::cerr << "cleanup " << py_V3 << " " << V3 << "\n";
        //fprintf(stderr, "c_cleanup CNDA py_object w refcnt %p %i\n", py_V3, (py_V3->ob_refcnt));
        if (V3)
        {
            //fprintf(stderr, "c_cleanup CNDA cn_object w refcnt %p %i\n", V3, (V3->ob_refcnt));
            Py_XDECREF(V3);
        }
        //std::cerr << "cleanup done" << py_V3 << "\n";
        
    {Py_XDECREF(py_V3);}
    
double __DUMMY_4;

}
__label_2:

    if (!__failure) {
      
        //std::cerr << "sync\n";
        if (NULL == V1) {
            // failure: sync None to storage
            Py_XDECREF(py_V1);
            py_V1 = Py_None;
            Py_INCREF(py_V1);
        }
        else
        {
            if (py_V1 != (PyObject*)V1)
            {
                Py_XDECREF(py_V1);
                py_V1 = (PyObject*)V1;
                Py_INCREF(py_V1);
            }
            assert(py_V1->ob_refcnt);
        }
        
      PyObject* old = PyList_GET_ITEM(storage_V1, 0);
      {Py_XINCREF(py_V1);}
      PyList_SET_ITEM(storage_V1, 0, py_V1);
      {Py_XDECREF(old);}
    }
    
        //std::cerr << "cleanup " << py_V1 << " " << V1 << "\n";
        //fprintf(stderr, "c_cleanup CNDA py_object w refcnt %p %i\n", py_V1, (py_V1->ob_refcnt));
        if (V1)
        {
            //fprintf(stderr, "c_cleanup CNDA cn_object w refcnt %p %i\n", V1, (V1->ob_refcnt));
            Py_XDECREF(V1);
        }
        //std::cerr << "cleanup done" << py_V1 << "\n";
        
    {Py_XDECREF(py_V1);}
    
double __DUMMY_2;

}

            
        if (__failure) {
            // When there is a failure, this code puts the exception
            // in __ERROR.
            PyObject* err_type = NULL;
            PyObject* err_msg = NULL;
            PyObject* err_traceback = NULL;
            PyErr_Fetch(&err_type, &err_msg, &err_traceback);
            if (!err_type) {err_type = Py_None;Py_INCREF(Py_None);}
            if (!err_msg) {err_msg = Py_None; Py_INCREF(Py_None);}
            if (!err_traceback) {err_traceback = Py_None; Py_INCREF(Py_None);}
            PyObject* old_err_type = PyList_GET_ITEM(__ERROR, 0);
            PyObject* old_err_msg = PyList_GET_ITEM(__ERROR, 1);
            PyObject* old_err_traceback = PyList_GET_ITEM(__ERROR, 2);
            PyList_SET_ITEM(__ERROR, 0, err_type);
            PyList_SET_ITEM(__ERROR, 1, err_msg);
            PyList_SET_ITEM(__ERROR, 2, err_traceback);
            {Py_XDECREF(old_err_type);}
            {Py_XDECREF(old_err_msg);}
            {Py_XDECREF(old_err_traceback);}
        }
        // The failure code is returned to index what code block failed.
        return __failure;
        
        }
    };
    }
    

        static int __struct_compiled_op_ef5673747c99044f59681d9ee3ac8b53_executor(__struct_compiled_op_ef5673747c99044f59681d9ee3ac8b53* self) {
            return self->run();
        }

        static void __struct_compiled_op_ef5673747c99044f59681d9ee3ac8b53_destructor(void* executor, void* self) {
            delete ((__struct_compiled_op_ef5673747c99044f59681d9ee3ac8b53*)self);
        }
        
//////////////////////
////  Functions
//////////////////////
static PyObject * instantiate(PyObject * self, PyObject *argtuple) {
  assert(PyTuple_Check(argtuple));
  if (5 != PyTuple_Size(argtuple)){ 
     PyErr_Format(PyExc_TypeError, "Wrong number of arguments, expected 5, got %i", (int)PyTuple_Size(argtuple));
     return NULL;
  }
  __struct_compiled_op_ef5673747c99044f59681d9ee3ac8b53* struct_ptr = new __struct_compiled_op_ef5673747c99044f59681d9ee3ac8b53();
  if (struct_ptr->init( PyTuple_GET_ITEM(argtuple, 0),PyTuple_GET_ITEM(argtuple, 1),PyTuple_GET_ITEM(argtuple, 2),PyTuple_GET_ITEM(argtuple, 3),PyTuple_GET_ITEM(argtuple, 4) ) != 0) {
    delete struct_ptr;
    return NULL;
  }
  PyObject* thunk = PyCObject_FromVoidPtrAndDesc((void*)(&__struct_compiled_op_ef5673747c99044f59681d9ee3ac8b53_executor), struct_ptr, __struct_compiled_op_ef5673747c99044f59681d9ee3ac8b53_destructor);
  return thunk; }

//////////////////////
////  Module init
//////////////////////
static PyMethodDef MyMethods[] = {
	{"instantiate", instantiate, METH_VARARGS, "undocumented"} ,
	{NULL, NULL, 0, NULL}
};
PyMODINIT_FUNC initef5673747c99044f59681d9ee3ac8b53(void){
   import_array();
   (void) Py_InitModule("ef5673747c99044f59681d9ee3ac8b53", MyMethods);
}
