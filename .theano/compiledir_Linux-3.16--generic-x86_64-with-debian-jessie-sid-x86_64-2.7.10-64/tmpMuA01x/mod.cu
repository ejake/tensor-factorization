#include <Python.h>
#include <iostream>
#include "theano_mod_helper.h"
#include "cuda_ndarray.cuh"
#include <math.h>
#include <numpy/arrayobject.h>
#include <numpy/arrayscalars.h>
//////////////////////
////  Support Code
//////////////////////


    namespace {
    struct __struct_compiled_op_9854bfa6f034fe0d8f1c2131d4045172 {
        PyObject* __ERROR;

        PyObject* storage_V3;
PyObject* storage_V5;
PyObject* storage_V1;
        

        __struct_compiled_op_9854bfa6f034fe0d8f1c2131d4045172() {
            // This is only somewhat safe because we:
            //  1) Are not a virtual class
            //  2) Do not use any virtual classes in the members
            //  3) Deal with mostly POD and pointers

            // If this changes, we would have to revise this, but for
            // now I am tired of chasing segfaults because
            // initialization code had an error and some pointer has
            // a junk value.
            memset(this, 0, sizeof(*this));
        }
        ~__struct_compiled_op_9854bfa6f034fe0d8f1c2131d4045172(void) {
            cleanup();
        }

        int init(PyObject* __ERROR, PyObject* storage_V3, PyObject* storage_V5, PyObject* storage_V1) {
            Py_XINCREF(storage_V3);
Py_XINCREF(storage_V5);
Py_XINCREF(storage_V1);
            this->storage_V3 = storage_V3;
this->storage_V5 = storage_V5;
this->storage_V1 = storage_V1;
            




            this->__ERROR = __ERROR;
            return 0;
        }
        void cleanup(void) {
            __label_1:

double __DUMMY_1;
__label_3:

double __DUMMY_3;
__label_5:

double __DUMMY_5;
__label_8:

double __DUMMY_8;

            Py_XDECREF(this->storage_V3);
Py_XDECREF(this->storage_V5);
Py_XDECREF(this->storage_V1);
        }
        int run(void) {
            int __failure = 0;
            
    PyObject* py_V1;
     CudaNdarray * V1;
    PyObject* py_V3;
     CudaNdarray * V3;
    PyObject* py_V5;
    
        PyArrayObject* V5;
        
{

    py_V1 = PyList_GET_ITEM(storage_V1, 0);
    {Py_XINCREF(py_V1);}
    
        if (py_V1 == Py_None)
        {
            V1 = NULL;
        }
        else
        {
            
        assert(py_V1->ob_refcnt >= 2); // There should be at least one ref from the container object,
        // and one ref from the local scope.

        if (CudaNdarray_Check(py_V1))
        {
            //fprintf(stderr, "c_extract CNDA object w refcnt %p %i\n", py_V1, (py_V1->ob_refcnt));
            V1 = (CudaNdarray*)py_V1;
            //std::cerr << "c_extract " << V1 << '\n';
        

                assert(V1);
                Py_INCREF(py_V1);
            }
            

        }
        
{

    py_V3 = PyList_GET_ITEM(storage_V3, 0);
    {Py_XINCREF(py_V3);}
    
        assert(py_V3->ob_refcnt >= 2); // There should be at least one ref from the container object,
        // and one ref from the local scope.

        if (CudaNdarray_Check(py_V3))
        {
            //fprintf(stderr, "c_extract CNDA object w refcnt %p %i\n", py_V3, (py_V3->ob_refcnt));
            V3 = (CudaNdarray*)py_V3;
            //std::cerr << "c_extract " << V3 << '\n';
        

                assert(V3);
                Py_INCREF(py_V3);
            }
            

{

    py_V5 = PyList_GET_ITEM(storage_V5, 0);
    {Py_XINCREF(py_V5);}
    
        V5 = (PyArrayObject*)(py_V5);
        Py_XINCREF(V5);
        
{
// Op class GpuReshape

        PyObject *new_shape = PyTuple_New(2);
        size_t total = 1;
        int compute_axis = -1;

        assert (PyArray_NDIM(V5) == 1);
        if (PyArray_DIM(V5, 0) != 2)
        {
            Py_XDECREF(new_shape);
            PyErr_Format(PyExc_ValueError,
                         "GpuReshape: given shape is of incorrect "
                         "length (%d should be %d).",
                         PyArray_DIM(V5, 0), 2);
            {
        __failure = 7;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_7;};
        }

        for (size_t i = 0; i < 2; ++i)
        {
            long dimension = ((npy_int64*)(
                    PyArray_BYTES(V5) +
                    i * PyArray_STRIDES(V5)[0]))[0];
            if (dimension == -1)
            {
                if (compute_axis != -1)
                {
                    Py_XDECREF(new_shape);
                    PyErr_Format(PyExc_ValueError,
                                 "GpuReshape: only one -1 is accepted "
                                 "in the new shape, but got two at "
                                 "indices %d and %zu.",
                                 compute_axis, i);
                    {
        __failure = 7;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_7;};
                }
                compute_axis = i;
            }
            else
            {
                total *= dimension;
                PyObject *py_dimension = PyInt_FromLong(dimension);
                PyTuple_SetItem(new_shape, i, py_dimension);
            }
        }

        if (compute_axis != -1)
        {
            long dimension = CudaNdarray_SIZE(V3) / total;
            total *= dimension;
            PyObject *py_dimension = PyInt_FromLong(dimension);
            PyTuple_SetItem(new_shape, compute_axis, py_dimension);
        }

        if (total != CudaNdarray_SIZE(V3))
        {
            const int *shape_from_py = CudaNdarray_HOST_DIMS(V3);

            char shape_from[128];
            size_t offset = 0;
            for (size_t i = 0; i < V3->nd; ++i)
            {
                int ws = snprintf(shape_from + offset, 128 - offset,
                        " %d,", shape_from_py[i]);
                offset += ws;
                if ( ws < 0 || offset >= 128 )
                    break;
            }

            shape_from[0]='(';
            if(offset < 128)
                shape_from[offset>0 ? offset-1 : 1] = ')';
            else
                for(size_t i=124; i<127; ++i)
                    shape_from[i] = '.';

            PyObject *shape_to_py = PyObject_Str(new_shape);
            const char *shape_to = PyString_AsString(shape_to_py);
            Py_XDECREF(new_shape);
            PyErr_Format(PyExc_ValueError,
                         "GpuReshape: cannot reshape input of shape "
                         "%s to shape %s.", shape_from, shape_to);
            {
        __failure = 7;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_7;};
        }

        Py_XDECREF(V1);
        V1 = (CudaNdarray*) CudaNdarray_Reshape(V3, new_shape);
        Py_XDECREF(new_shape);
        if (V1 == NULL)
        {
            {
        __failure = 7;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_7;};
        }
        __label_7:

double __DUMMY_7;

}
__label_6:

        if (V5) {
            Py_XDECREF(V5);
        }
        
    {Py_XDECREF(py_V5);}
    
double __DUMMY_6;

}
__label_4:

        //std::cerr << "cleanup " << py_V3 << " " << V3 << "\n";
        //fprintf(stderr, "c_cleanup CNDA py_object w refcnt %p %i\n", py_V3, (py_V3->ob_refcnt));
        if (V3)
        {
            //fprintf(stderr, "c_cleanup CNDA cn_object w refcnt %p %i\n", V3, (V3->ob_refcnt));
            Py_XDECREF(V3);
        }
        //std::cerr << "cleanup done" << py_V3 << "\n";
        
    {Py_XDECREF(py_V3);}
    
double __DUMMY_4;

}
__label_2:

    if (!__failure) {
      
        //std::cerr << "sync\n";
        if (NULL == V1) {
            // failure: sync None to storage
            Py_XDECREF(py_V1);
            py_V1 = Py_None;
            Py_INCREF(py_V1);
        }
        else
        {
            if (py_V1 != (PyObject*)V1)
            {
                Py_XDECREF(py_V1);
                py_V1 = (PyObject*)V1;
                Py_INCREF(py_V1);
            }
            assert(py_V1->ob_refcnt);
        }
        
      PyObject* old = PyList_GET_ITEM(storage_V1, 0);
      {Py_XINCREF(py_V1);}
      PyList_SET_ITEM(storage_V1, 0, py_V1);
      {Py_XDECREF(old);}
    }
    
        //std::cerr << "cleanup " << py_V1 << " " << V1 << "\n";
        //fprintf(stderr, "c_cleanup CNDA py_object w refcnt %p %i\n", py_V1, (py_V1->ob_refcnt));
        if (V1)
        {
            //fprintf(stderr, "c_cleanup CNDA cn_object w refcnt %p %i\n", V1, (V1->ob_refcnt));
            Py_XDECREF(V1);
        }
        //std::cerr << "cleanup done" << py_V1 << "\n";
        
    {Py_XDECREF(py_V1);}
    
double __DUMMY_2;

}

            
        if (__failure) {
            // When there is a failure, this code puts the exception
            // in __ERROR.
            PyObject* err_type = NULL;
            PyObject* err_msg = NULL;
            PyObject* err_traceback = NULL;
            PyErr_Fetch(&err_type, &err_msg, &err_traceback);
            if (!err_type) {err_type = Py_None;Py_INCREF(Py_None);}
            if (!err_msg) {err_msg = Py_None; Py_INCREF(Py_None);}
            if (!err_traceback) {err_traceback = Py_None; Py_INCREF(Py_None);}
            PyObject* old_err_type = PyList_GET_ITEM(__ERROR, 0);
            PyObject* old_err_msg = PyList_GET_ITEM(__ERROR, 1);
            PyObject* old_err_traceback = PyList_GET_ITEM(__ERROR, 2);
            PyList_SET_ITEM(__ERROR, 0, err_type);
            PyList_SET_ITEM(__ERROR, 1, err_msg);
            PyList_SET_ITEM(__ERROR, 2, err_traceback);
            {Py_XDECREF(old_err_type);}
            {Py_XDECREF(old_err_msg);}
            {Py_XDECREF(old_err_traceback);}
        }
        // The failure code is returned to index what code block failed.
        return __failure;
        
        }
    };
    }
    

        static int __struct_compiled_op_9854bfa6f034fe0d8f1c2131d4045172_executor(__struct_compiled_op_9854bfa6f034fe0d8f1c2131d4045172* self) {
            return self->run();
        }

        static void __struct_compiled_op_9854bfa6f034fe0d8f1c2131d4045172_destructor(void* executor, void* self) {
            delete ((__struct_compiled_op_9854bfa6f034fe0d8f1c2131d4045172*)self);
        }
        
//////////////////////
////  Functions
//////////////////////
static PyObject * instantiate(PyObject * self, PyObject *argtuple) {
  assert(PyTuple_Check(argtuple));
  if (4 != PyTuple_Size(argtuple)){ 
     PyErr_Format(PyExc_TypeError, "Wrong number of arguments, expected 4, got %i", (int)PyTuple_Size(argtuple));
     return NULL;
  }
  __struct_compiled_op_9854bfa6f034fe0d8f1c2131d4045172* struct_ptr = new __struct_compiled_op_9854bfa6f034fe0d8f1c2131d4045172();
  if (struct_ptr->init( PyTuple_GET_ITEM(argtuple, 0),PyTuple_GET_ITEM(argtuple, 1),PyTuple_GET_ITEM(argtuple, 2),PyTuple_GET_ITEM(argtuple, 3) ) != 0) {
    delete struct_ptr;
    return NULL;
  }
  PyObject* thunk = PyCObject_FromVoidPtrAndDesc((void*)(&__struct_compiled_op_9854bfa6f034fe0d8f1c2131d4045172_executor), struct_ptr, __struct_compiled_op_9854bfa6f034fe0d8f1c2131d4045172_destructor);
  return thunk; }

//////////////////////
////  Module init
//////////////////////
static PyMethodDef MyMethods[] = {
	{"instantiate", instantiate, METH_VARARGS, "undocumented"} ,
	{NULL, NULL, 0, NULL}
};
PyMODINIT_FUNC init9854bfa6f034fe0d8f1c2131d4045172(void){
   import_array();
   (void) Py_InitModule("9854bfa6f034fe0d8f1c2131d4045172", MyMethods);
}
