#include "hip/hip_runtime.h"
#include <Python.h>
#include <iostream>
#include "theano_mod_helper.h"
#include "cuda_ndarray.cuh"
//////////////////////
////  Support Code
//////////////////////


            static __global__ void kernel_reduce_ccontig_node_4894639462a290346189bb38dab7bb7e_0(
                    const unsigned int d0,
                    const float *A,
                    float * Z)
            {
                const int threadCount = blockDim.x;
                const int threadNum = threadIdx.x;
                extern __shared__ float buf[];
                float myresult = 0;

                if (warpSize != 32)
                {
                    return;  //TODO: set error code
                }

                for (int i0 = threadIdx.x; i0 < d0; i0 += blockDim.x)
                {
                    myresult = myresult + A[i0];
                }
                
        __syncthreads(); // some kernel do multiple reduction.
        buf[threadNum] = myresult;
        __syncthreads();

        // rest of function is handled by one warp
        if (threadNum < warpSize)
        {
            //round up all the partial sums into the first `warpSize` elements
            for (int i = threadNum + warpSize; i < threadCount; i += warpSize)
            {
                myresult = myresult + buf[i];
            }
            buf[threadNum] = myresult;
        /*Comment this optimization as it don't work on Fermi GPU.
        TODO: find why it don't work or put the GPU compute capability into the version
            // no sync because only one warp is running
            if(threadCount >32)
            {buf[threadNum] = buf[threadNum] + buf[threadNum+16];buf[threadNum] = buf[threadNum] + buf[threadNum+8];buf[threadNum] = buf[threadNum] + buf[threadNum+4];buf[threadNum] = buf[threadNum] + buf[threadNum+2];buf[threadNum] = buf[threadNum] + buf[threadNum+1];
                if (threadNum == 0)
                {
                    Z[0] = buf[0];
                }

            }
            else */
            if (threadNum < 16)
            {
                //reduce so that threadNum 0 has the reduction of everything
                if (threadNum + 16 < threadCount) buf[threadNum] = buf[threadNum] + buf[threadNum+16];if (threadNum + 8 < threadCount) buf[threadNum] = buf[threadNum] + buf[threadNum+8];if (threadNum + 4 < threadCount) buf[threadNum] = buf[threadNum] + buf[threadNum+4];if (threadNum + 2 < threadCount) buf[threadNum] = buf[threadNum] + buf[threadNum+2];if (threadNum + 1 < threadCount) buf[threadNum] = buf[threadNum] + buf[threadNum+1];
                if (threadNum == 0)
                {
                    Z[0] = buf[0];
                }
            }
        }
        
            }
            

            static __global__ void kernel_reduce_1_node_4894639462a290346189bb38dab7bb7e_0(
                    const unsigned int d0,
                    const float *A, const int sA0,
                    float * Z)
            {
                const int threadCount = blockDim.x;
                const int threadNum = threadIdx.x;
                extern __shared__ float buf[];
                float myresult = 0;

                if (warpSize != 32)
                {
                    return;  //TODO: set error code
                }

                for (int i0 = threadIdx.x; i0 < d0; i0 += blockDim.x)
                {
                    myresult = myresult + A[i0 * sA0];
                }
                
        __syncthreads(); // some kernel do multiple reduction.
        buf[threadNum] = myresult;
        __syncthreads();

        // rest of function is handled by one warp
        if (threadNum < warpSize)
        {
            //round up all the partial sums into the first `warpSize` elements
            for (int i = threadNum + warpSize; i < threadCount; i += warpSize)
            {
                myresult = myresult + buf[i];
            }
            buf[threadNum] = myresult;
        /*Comment this optimization as it don't work on Fermi GPU.
        TODO: find why it don't work or put the GPU compute capability into the version
            // no sync because only one warp is running
            if(threadCount >32)
            {buf[threadNum] = buf[threadNum] + buf[threadNum+16];buf[threadNum] = buf[threadNum] + buf[threadNum+8];buf[threadNum] = buf[threadNum] + buf[threadNum+4];buf[threadNum] = buf[threadNum] + buf[threadNum+2];buf[threadNum] = buf[threadNum] + buf[threadNum+1];
                if (threadNum == 0)
                {
                    Z[0] = buf[0];
                }

            }
            else */
            if (threadNum < 16)
            {
                //reduce so that threadNum 0 has the reduction of everything
                if (threadNum + 16 < threadCount) buf[threadNum] = buf[threadNum] + buf[threadNum+16];if (threadNum + 8 < threadCount) buf[threadNum] = buf[threadNum] + buf[threadNum+8];if (threadNum + 4 < threadCount) buf[threadNum] = buf[threadNum] + buf[threadNum+4];if (threadNum + 2 < threadCount) buf[threadNum] = buf[threadNum] + buf[threadNum+2];if (threadNum + 1 < threadCount) buf[threadNum] = buf[threadNum] + buf[threadNum+1];
                if (threadNum == 0)
                {
                    Z[0] = buf[0];
                }
            }
        }
        
            }
            


    namespace {
    struct __struct_compiled_op_4894639462a290346189bb38dab7bb7e {
        PyObject* __ERROR;

        PyObject* storage_V3;
PyObject* storage_V1;
        

        __struct_compiled_op_4894639462a290346189bb38dab7bb7e() {
            // This is only somewhat safe because we:
            //  1) Are not a virtual class
            //  2) Do not use any virtual classes in the members
            //  3) Deal with mostly POD and pointers

            // If this changes, we would have to revise this, but for
            // now I am tired of chasing segfaults because
            // initialization code had an error and some pointer has
            // a junk value.
            memset(this, 0, sizeof(*this));
        }
        ~__struct_compiled_op_4894639462a290346189bb38dab7bb7e(void) {
            cleanup();
        }

        int init(PyObject* __ERROR, PyObject* storage_V3, PyObject* storage_V1) {
            Py_XINCREF(storage_V3);
Py_XINCREF(storage_V1);
            this->storage_V3 = storage_V3;
this->storage_V1 = storage_V1;
            



            this->__ERROR = __ERROR;
            return 0;
        }
        void cleanup(void) {
            __label_1:

double __DUMMY_1;
__label_3:

double __DUMMY_3;
__label_6:

double __DUMMY_6;

            Py_XDECREF(this->storage_V3);
Py_XDECREF(this->storage_V1);
        }
        int run(void) {
            int __failure = 0;
            
    PyObject* py_V1;
     CudaNdarray * V1;
    PyObject* py_V3;
     CudaNdarray * V3;
{

    py_V1 = PyList_GET_ITEM(storage_V1, 0);
    {Py_XINCREF(py_V1);}
    
        if (py_V1 == Py_None)
        {
            V1 = NULL;
        }
        else
        {
            
        assert(py_V1->ob_refcnt >= 2); // There should be at least one ref from the container object,
        // and one ref from the local scope.

        if (CudaNdarray_Check(py_V1))
        {
            //fprintf(stderr, "c_extract CNDA object w refcnt %p %i\n", py_V1, (py_V1->ob_refcnt));
            V1 = (CudaNdarray*)py_V1;
            //std::cerr << "c_extract " << V1 << '\n';
        

                if (V1->nd != 0)
                {
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has rank %i, it was supposed to have rank 0",
                                 V1->nd);
                    V1 = NULL;
                    {
        __failure = 2;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_2;};
                }
                //std::cerr << "c_extract " << V1 << " nd check passed\n";
            

                assert(V1);
                Py_INCREF(py_V1);
            }
            else if (py_V1 == Py_None)
            {
                PyErr_SetString(PyExc_TypeError,
                                "expected a CudaNdarray, not None");
                V1 = NULL;
                {
        __failure = 2;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_2;};
            }
            else
            {
                //fprintf(stderr, "FAILING c_extract CNDA object w refcnt %p %i\n", py_V1, (py_V1->ob_refcnt));
                PyErr_SetString(PyExc_TypeError, "Argument not a CudaNdarray");
                V1 = NULL;
                {
        __failure = 2;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_2;};
            }
            //std::cerr << "c_extract done " << V1 << '\n';
            

        }
        
{

    py_V3 = PyList_GET_ITEM(storage_V3, 0);
    {Py_XINCREF(py_V3);}
    
        assert(py_V3->ob_refcnt >= 2); // There should be at least one ref from the container object,
        // and one ref from the local scope.

        if (CudaNdarray_Check(py_V3))
        {
            //fprintf(stderr, "c_extract CNDA object w refcnt %p %i\n", py_V3, (py_V3->ob_refcnt));
            V3 = (CudaNdarray*)py_V3;
            //std::cerr << "c_extract " << V3 << '\n';
        

                if (V3->nd != 1)
                {
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has rank %i, it was supposed to have rank 1",
                                 V3->nd);
                    V3 = NULL;
                    {
        __failure = 4;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_4;};
                }
                //std::cerr << "c_extract " << V3 << " nd check passed\n";
            

                assert(V3);
                Py_INCREF(py_V3);
            }
            else if (py_V3 == Py_None)
            {
                PyErr_SetString(PyExc_TypeError,
                                "expected a CudaNdarray, not None");
                V3 = NULL;
                {
        __failure = 4;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_4;};
            }
            else
            {
                //fprintf(stderr, "FAILING c_extract CNDA object w refcnt %p %i\n", py_V3, (py_V3->ob_refcnt));
                PyErr_SetString(PyExc_TypeError, "Argument not a CudaNdarray");
                V3 = NULL;
                {
        __failure = 4;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_4;};
            }
            //std::cerr << "c_extract done " << V3 << '\n';
            

{
// Op class GpuCAReduce

        if (V3->nd != 1)
        {
            PyErr_Format(PyExc_TypeError,
                         "required nd=1, got nd=%i", V3->nd);
            {
        __failure = 5;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_5;};
        }
        

        if (  !V1
           || (V1->nd != 0)
        

           )
        {
            
int *new_dims=NULL; 

            Py_XDECREF(V1);
            V1 = (CudaNdarray*) CudaNdarray_NewDims(0, new_dims);
            if (NULL == V1)
            {
                {
        __failure = 5;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_5;};
            }
        }
        

        if (CudaNdarray_SIZE(V1) && ! CudaNdarray_SIZE(V3)){
            hipMemset(V1->devdata, 0, CudaNdarray_SIZE(V1) * sizeof(float));
        }
        else if (CudaNdarray_SIZE(V1))
        {
        
if(CudaNdarray_is_c_contiguous( V3)){

        {
          if(CudaNdarray_SIZE(V3)==0){
            hipMemset(V1->devdata, 0, CudaNdarray_SIZE(V1) * sizeof(float));
          }else{
            int verbose = 0;
            dim3 n_threads(
                    std::min(CudaNdarray_SIZE(V3),
                             (size_t) NUM_VECTOR_OP_THREADS_PER_BLOCK));
            dim3 n_blocks(1);
            if (verbose) printf("running kernel_reduce_ccontig_node_4894639462a290346189bb38dab7bb7e_0"
                                " n_threads.x=%d, size=%d, ndim=%d\n",
                                n_threads.x,CudaNdarray_SIZE(V3),V3->nd);
            int n_shared = sizeof(float) * n_threads.x;
            kernel_reduce_ccontig_node_4894639462a290346189bb38dab7bb7e_0<<<n_blocks, n_threads, n_shared>>>(
                    CudaNdarray_SIZE(V3),
                    CudaNdarray_DEV_DATA(V3),
                    CudaNdarray_DEV_DATA(V1));
            CNDA_THREAD_SYNC;
            hipError_t sts = hipGetLastError();
            if (hipSuccess != sts)
            {
                PyErr_Format(PyExc_RuntimeError,
                             "Cuda error: %s: %s."
                             " (grid: %i x %i; block: %i x %i x %i)\n",
                    "kernel_reduce_ccontig_node_4894639462a290346189bb38dab7bb7e_0",
                    hipGetErrorString(sts),
                    n_blocks.x,
                    n_blocks.y,
                    n_threads.x,
                    n_threads.y,
                    n_threads.z);
                {
        __failure = 5;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_5;};
            }
         }
        }
        
}else{

        {
            int verbose = 0;
            dim3 n_threads(
                    std::min(CudaNdarray_HOST_DIMS(V3)[0],
                            NUM_VECTOR_OP_THREADS_PER_BLOCK));
            dim3 n_blocks(1);
            
            if (verbose)
                printf("running kernel_reduce_1_node_4894639462a290346189bb38dab7bb7e_0\n");
            int n_shared = sizeof(float) * n_threads.x * n_threads.y * n_threads.z;
            if (verbose>1)
                printf("n_threads.x=%d, n_threads.y=%d, n_threads.z=%d,"
                       " nb_threads=%d, n_blocks.x=%d, n_blocks.y=%d,"
                       " nb_block=%d, n_shared=%d, shape=(%d)\n",
                                  n_threads.x,n_threads.y,n_threads.z,
                                  n_threads.x*n_threads.y*n_threads.z,
                                  n_blocks.x,n_blocks.y,
                                  n_blocks.x*n_blocks.y, n_shared, CudaNdarray_HOST_DIMS(V3)[0]);
            kernel_reduce_1_node_4894639462a290346189bb38dab7bb7e_0<<<n_blocks, n_threads, n_shared>>>(
            

                    CudaNdarray_HOST_DIMS(V3)[0],
            

                    CudaNdarray_DEV_DATA(V3)
            

                    ,CudaNdarray_HOST_STRIDES(V3)[0]
            

                    ,CudaNdarray_DEV_DATA(V1)
            

                    );
            CNDA_THREAD_SYNC;
            hipError_t sts = hipGetLastError();
            if (hipSuccess != sts)
            {
                PyErr_Format(PyExc_RuntimeError,
                    "Cuda error: %s: %s."
                    " (grid: %i x %i; block: %i x %i x %i)"
                    " shape=(%d) \n",
                    "kernel_reduce_1_node_4894639462a290346189bb38dab7bb7e_0",
                    hipGetErrorString(sts),
                    n_blocks.x,
                    n_blocks.y,
                    n_threads.x,
                    n_threads.y,
                    n_threads.z,
                    CudaNdarray_HOST_DIMS(V3)[0]);
                {
        __failure = 5;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_5;};
            }
        

        }
        
}

        }
        
__label_5:

double __DUMMY_5;

}
__label_4:

        //std::cerr << "cleanup " << py_V3 << " " << V3 << "\n";
        //fprintf(stderr, "c_cleanup CNDA py_object w refcnt %p %i\n", py_V3, (py_V3->ob_refcnt));
        if (V3)
        {
            //fprintf(stderr, "c_cleanup CNDA cn_object w refcnt %p %i\n", V3, (V3->ob_refcnt));
            Py_XDECREF(V3);
        }
        //std::cerr << "cleanup done" << py_V3 << "\n";
        
    {Py_XDECREF(py_V3);}
    
double __DUMMY_4;

}
__label_2:

    if (!__failure) {
      
        //std::cerr << "sync\n";
        if (NULL == V1) {
            // failure: sync None to storage
            Py_XDECREF(py_V1);
            py_V1 = Py_None;
            Py_INCREF(py_V1);
        }
        else
        {
            if (py_V1 != (PyObject*)V1)
            {
                Py_XDECREF(py_V1);
                py_V1 = (PyObject*)V1;
                Py_INCREF(py_V1);
            }
            assert(py_V1->ob_refcnt);
        }
        
      PyObject* old = PyList_GET_ITEM(storage_V1, 0);
      {Py_XINCREF(py_V1);}
      PyList_SET_ITEM(storage_V1, 0, py_V1);
      {Py_XDECREF(old);}
    }
    
        //std::cerr << "cleanup " << py_V1 << " " << V1 << "\n";
        //fprintf(stderr, "c_cleanup CNDA py_object w refcnt %p %i\n", py_V1, (py_V1->ob_refcnt));
        if (V1)
        {
            //fprintf(stderr, "c_cleanup CNDA cn_object w refcnt %p %i\n", V1, (V1->ob_refcnt));
            Py_XDECREF(V1);
        }
        //std::cerr << "cleanup done" << py_V1 << "\n";
        
    {Py_XDECREF(py_V1);}
    
double __DUMMY_2;

}

            
        if (__failure) {
            // When there is a failure, this code puts the exception
            // in __ERROR.
            PyObject* err_type = NULL;
            PyObject* err_msg = NULL;
            PyObject* err_traceback = NULL;
            PyErr_Fetch(&err_type, &err_msg, &err_traceback);
            if (!err_type) {err_type = Py_None;Py_INCREF(Py_None);}
            if (!err_msg) {err_msg = Py_None; Py_INCREF(Py_None);}
            if (!err_traceback) {err_traceback = Py_None; Py_INCREF(Py_None);}
            PyObject* old_err_type = PyList_GET_ITEM(__ERROR, 0);
            PyObject* old_err_msg = PyList_GET_ITEM(__ERROR, 1);
            PyObject* old_err_traceback = PyList_GET_ITEM(__ERROR, 2);
            PyList_SET_ITEM(__ERROR, 0, err_type);
            PyList_SET_ITEM(__ERROR, 1, err_msg);
            PyList_SET_ITEM(__ERROR, 2, err_traceback);
            {Py_XDECREF(old_err_type);}
            {Py_XDECREF(old_err_msg);}
            {Py_XDECREF(old_err_traceback);}
        }
        // The failure code is returned to index what code block failed.
        return __failure;
        
        }
    };
    }
    

        static int __struct_compiled_op_4894639462a290346189bb38dab7bb7e_executor(__struct_compiled_op_4894639462a290346189bb38dab7bb7e* self) {
            return self->run();
        }

        static void __struct_compiled_op_4894639462a290346189bb38dab7bb7e_destructor(void* executor, void* self) {
            delete ((__struct_compiled_op_4894639462a290346189bb38dab7bb7e*)self);
        }
        
//////////////////////
////  Functions
//////////////////////
static PyObject * instantiate(PyObject * self, PyObject *argtuple) {
  assert(PyTuple_Check(argtuple));
  if (3 != PyTuple_Size(argtuple)){ 
     PyErr_Format(PyExc_TypeError, "Wrong number of arguments, expected 3, got %i", (int)PyTuple_Size(argtuple));
     return NULL;
  }
  __struct_compiled_op_4894639462a290346189bb38dab7bb7e* struct_ptr = new __struct_compiled_op_4894639462a290346189bb38dab7bb7e();
  if (struct_ptr->init( PyTuple_GET_ITEM(argtuple, 0),PyTuple_GET_ITEM(argtuple, 1),PyTuple_GET_ITEM(argtuple, 2) ) != 0) {
    delete struct_ptr;
    return NULL;
  }
  PyObject* thunk = PyCObject_FromVoidPtrAndDesc((void*)(&__struct_compiled_op_4894639462a290346189bb38dab7bb7e_executor), struct_ptr, __struct_compiled_op_4894639462a290346189bb38dab7bb7e_destructor);
  return thunk; }

//////////////////////
////  Module init
//////////////////////
static PyMethodDef MyMethods[] = {
	{"instantiate", instantiate, METH_VARARGS, "undocumented"} ,
	{NULL, NULL, 0, NULL}
};
PyMODINIT_FUNC init4894639462a290346189bb38dab7bb7e(void){
   (void) Py_InitModule("4894639462a290346189bb38dab7bb7e", MyMethods);
}
