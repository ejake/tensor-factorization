#include <Python.h>
#include <iostream>
#include "theano_mod_helper.h"
#include "hipDNN.h"
//////////////////////
////  Support Code
//////////////////////

#if PY_MAJOR_VERSION >= 3
#define PyInt_FromLong PyLong_FromLong
#endif


    namespace {
    struct __struct_compiled_op_265abc51f7c376c224983485238ff1a5 {
        PyObject* __ERROR;

        PyObject* storage_V1;
        

        __struct_compiled_op_265abc51f7c376c224983485238ff1a5() {
            // This is only somewhat safe because we:
            //  1) Are not a virtual class
            //  2) Do not use any virtual classes in the members
            //  3) Deal with mostly POD and pointers

            // If this changes, we would have to revise this, but for
            // now I am tired of chasing segfaults because
            // initialization code had an error and some pointer has
            // a junk value.
            memset(this, 0, sizeof(*this));
        }
        ~__struct_compiled_op_265abc51f7c376c224983485238ff1a5(void) {
            cleanup();
        }

        int init(PyObject* __ERROR, PyObject* storage_V1) {
            Py_XINCREF(storage_V1);
            this->storage_V1 = storage_V1;
            


            this->__ERROR = __ERROR;
            return 0;
        }
        void cleanup(void) {
            __label_1:

double __DUMMY_1;
__label_4:

double __DUMMY_4;

            Py_XDECREF(this->storage_V1);
        }
        int run(void) {
            int __failure = 0;
            
    PyObject* py_V1;
    
        PyObject* V1;
        
{

    py_V1 = Py_None;
    {Py_XINCREF(py_V1);}
    
        V1 = NULL;
        
{
// Op class DnnVersion

        #if defined(CUDNN_VERSION)
        V1 = PyTuple_Pack(2, PyInt_FromLong(CUDNN_VERSION), PyInt_FromLong(hipdnnGetVersion()));
        #else
        V1 = PyInt_FromLong(-1);
        #endif
        __label_3:

double __DUMMY_3;

}
__label_2:

    if (!__failure) {
      
        assert(py_V1->ob_refcnt > 1);
        Py_DECREF(py_V1);
        py_V1 = V1 ? V1 : Py_None;
        Py_INCREF(py_V1);
        
      PyObject* old = PyList_GET_ITEM(storage_V1, 0);
      {Py_XINCREF(py_V1);}
      PyList_SET_ITEM(storage_V1, 0, py_V1);
      {Py_XDECREF(old);}
    }
    
        Py_XDECREF(V1);
        
    {Py_XDECREF(py_V1);}
    
double __DUMMY_2;

}

            
        if (__failure) {
            // When there is a failure, this code puts the exception
            // in __ERROR.
            PyObject* err_type = NULL;
            PyObject* err_msg = NULL;
            PyObject* err_traceback = NULL;
            PyErr_Fetch(&err_type, &err_msg, &err_traceback);
            if (!err_type) {err_type = Py_None;Py_INCREF(Py_None);}
            if (!err_msg) {err_msg = Py_None; Py_INCREF(Py_None);}
            if (!err_traceback) {err_traceback = Py_None; Py_INCREF(Py_None);}
            PyObject* old_err_type = PyList_GET_ITEM(__ERROR, 0);
            PyObject* old_err_msg = PyList_GET_ITEM(__ERROR, 1);
            PyObject* old_err_traceback = PyList_GET_ITEM(__ERROR, 2);
            PyList_SET_ITEM(__ERROR, 0, err_type);
            PyList_SET_ITEM(__ERROR, 1, err_msg);
            PyList_SET_ITEM(__ERROR, 2, err_traceback);
            {Py_XDECREF(old_err_type);}
            {Py_XDECREF(old_err_msg);}
            {Py_XDECREF(old_err_traceback);}
        }
        // The failure code is returned to index what code block failed.
        return __failure;
        
        }
    };
    }
    

        static int __struct_compiled_op_265abc51f7c376c224983485238ff1a5_executor(__struct_compiled_op_265abc51f7c376c224983485238ff1a5* self) {
            return self->run();
        }

        static void __struct_compiled_op_265abc51f7c376c224983485238ff1a5_destructor(void* executor, void* self) {
            delete ((__struct_compiled_op_265abc51f7c376c224983485238ff1a5*)self);
        }
        
//////////////////////
////  Functions
//////////////////////
static PyObject * instantiate(PyObject * self, PyObject *argtuple) {
  assert(PyTuple_Check(argtuple));
  if (2 != PyTuple_Size(argtuple)){ 
     PyErr_Format(PyExc_TypeError, "Wrong number of arguments, expected 2, got %i", (int)PyTuple_Size(argtuple));
     return NULL;
  }
  __struct_compiled_op_265abc51f7c376c224983485238ff1a5* struct_ptr = new __struct_compiled_op_265abc51f7c376c224983485238ff1a5();
  if (struct_ptr->init( PyTuple_GET_ITEM(argtuple, 0),PyTuple_GET_ITEM(argtuple, 1) ) != 0) {
    delete struct_ptr;
    return NULL;
  }
  PyObject* thunk = PyCObject_FromVoidPtrAndDesc((void*)(&__struct_compiled_op_265abc51f7c376c224983485238ff1a5_executor), struct_ptr, __struct_compiled_op_265abc51f7c376c224983485238ff1a5_destructor);
  return thunk; }

//////////////////////
////  Module init
//////////////////////
static PyMethodDef MyMethods[] = {
	{"instantiate", instantiate, METH_VARARGS, "undocumented"} ,
	{NULL, NULL, 0, NULL}
};
PyMODINIT_FUNC init265abc51f7c376c224983485238ff1a5(void){
   (void) Py_InitModule("265abc51f7c376c224983485238ff1a5", MyMethods);
}
