#include "hip/hip_runtime.h"
#include <Python.h>
#include <iostream>
#include "theano_mod_helper.h"
#include "cuda_ndarray.cuh"
//////////////////////
////  Support Code
//////////////////////


#define INTDIV_POW2(a, b) (a >> b)
#define INTMOD_POW2(a, b) (a & ((1<<b)-1))
        // GpuElemwise{Composite{(((-i0) / i1) * i2)}}[(0, 0)]
// node.op.destroy_map={0: [0]}
//    Input   0 CudaNdarrayType(float32, vector)
//    Input   1 CudaNdarrayType(float32, (True,))
//    Input   2 CudaNdarrayType(float32, vector)
//    Output  0 CudaNdarrayType(float32, vector)
static __global__ void kernel_Composite_node_3d6ea06e14a2eac7e20cc661f60176f0_0_1(unsigned int numEls
	, const int dim0
	, const float * i0_data, int i0_str_0
	, const float * i1_data, int i1_str_0
	, const float * i2_data, int i2_str_0
	, float * o0_data, int o0_str_0
	)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    const float ii_i1_value = i1_data[0];
    for (int i = idx; i < numEls; i += numThreads) {
        int ii = i;
        const float * ii_i0_data = i0_data;
        const float * ii_i2_data = i2_data;
        float * ii_o0_data = o0_data;
        int pos0 = ii;
        ii_i0_data += pos0 * i0_str_0;
        ii_i2_data += pos0 * i2_str_0;
        ii_o0_data += pos0 * o0_str_0;
npy_float32 o0_i;
        {
npy_float32 V_DUMMY_ID__tmp1;
V_DUMMY_ID__tmp1 = -ii_i0_data[0];
npy_float32 V_DUMMY_ID__tmp2;
V_DUMMY_ID__tmp2 = V_DUMMY_ID__tmp1 / ii_i1_value;
o0_i = V_DUMMY_ID__tmp2 * ii_i2_data[0];
}

ii_o0_data[0] = o0_i;
    }
}
// GpuElemwise{Composite{(((-i0) / i1) * i2)}}[(0, 0)]
// node.op.destroy_map={0: [0]}
//    Input   0 CudaNdarrayType(float32, vector)
//    Input   1 CudaNdarrayType(float32, (True,))
//    Input   2 CudaNdarrayType(float32, vector)
//    Output  0 CudaNdarrayType(float32, vector)
static __global__ void kernel_Composite_node_3d6ea06e14a2eac7e20cc661f60176f0_0_Ccontiguous (unsigned int numEls
	, const float * i0_data
	, const float * i1_data
	, const float * i2_data
	, float * o0_data
	)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    const float ii_i1_value = i1_data[0];
    for (int i = idx; i < numEls; i += numThreads) {
npy_float32 o0_i;
        {
npy_float32 V_DUMMY_ID__tmp1;
V_DUMMY_ID__tmp1 = -i0_data[i];
npy_float32 V_DUMMY_ID__tmp2;
V_DUMMY_ID__tmp2 = V_DUMMY_ID__tmp1 / ii_i1_value;
o0_i = V_DUMMY_ID__tmp2 * i2_data[i];
}

o0_data[i] = o0_i;
    }
}

        static void can_collapse_node_3d6ea06e14a2eac7e20cc661f60176f0_0(int nd, const int * dims, const int * strides, int collapse[])
        {
            //can we collapse dims[i] and dims[i-1]
            for(int i=nd-1;i>0;i--){
                if(strides[i]*dims[i]==strides[i-1]){//the dims nd-1 are not strided again dimension nd
                    collapse[i]=1;
                }else collapse[i]=0;
            }
        }
        

        static int callkernel_node_3d6ea06e14a2eac7e20cc661f60176f0_0(unsigned int numEls, const int d,
            const int * dims,
            const float * i0_data, const int * i0_str, const float * i1_data, const int * i1_str, const float * i2_data, const int * i2_str,
            float * o0_data, const int * o0_str)
        {
            numEls = dims[0]*1;
        
int local_dims[1];

            int local_str[3][1];
            int local_ostr[1][1];
            

        int nd_collapse = 1;
        for(int i=0;i<1;i++){//init new dim
          local_dims[i]=dims[i];
        }
        

            for(int i=0;i<1;i++){//init new strides
              local_str[0][i]=i0_str[i];
            }
            

            for(int i=0;i<1;i++){//init new strides
              local_str[1][i]=i1_str[i];
            }
            

            for(int i=0;i<1;i++){//init new strides
              local_str[2][i]=i2_str[i];
            }
            

            for(int i=0;i<1;i++){//init new strides
              local_ostr[0][i]=o0_str[i];
            }
            

        for(int id=0;id<nd_collapse;id++){

          bool all_broadcast=true;
          for(int input_id=0;input_id<3;input_id++){
            if(local_str[input_id][id]!=0 || local_dims[id]!=1) all_broadcast= false;
          }
          for(int input_id=0;input_id<1;input_id++){
            if(local_ostr[input_id][id]!=0 || local_dims[id]!=1) all_broadcast= false;
          }
          if(all_broadcast){
            for(int j=id+1;j<nd_collapse;j++)//remove dims i from the array
              local_dims[j-1]=local_dims[j];
            for(int input_id=0;input_id<3;input_id++){
              for(int j=id+1;j<nd_collapse;j++){//remove dims i from the array
                local_str[input_id][j-1]=local_str[input_id][j];
              }
            }
            for(int output_id=0;output_id<1;output_id++){
              for(int j=id+1;j<nd_collapse;j++){//remove dims i from the array
                local_ostr[output_id][j-1]=local_ostr[output_id][j];
              }
            }
            nd_collapse--; id--;
          }
        }
        
int nd_collapse_[1] = {1};

                        int nd_collapse_0[1] = {1};

can_collapse_node_3d6ea06e14a2eac7e20cc661f60176f0_0(nd_collapse, local_dims, local_str[0], nd_collapse_0);
for(int i=0;i<nd_collapse;i++){
if(nd_collapse_0[i]==0)
nd_collapse_[i]=0;
}
                

                        int nd_collapse_2[1] = {1};

can_collapse_node_3d6ea06e14a2eac7e20cc661f60176f0_0(nd_collapse, local_dims, local_str[2], nd_collapse_2);
for(int i=0;i<nd_collapse;i++){
if(nd_collapse_2[i]==0)
nd_collapse_[i]=0;
}
                

            for(int i=nd_collapse-1;i>0;i--){
              if(nd_collapse_[i]==1){
                local_str[0][i-1]=local_str[0][i];//set new strides
                for(int j=i+1;j<nd_collapse;j++)//remove stride i from the array
                  local_str[0][j-1]=local_str[0][j];
                }
            }
            

            for(int i=nd_collapse-1;i>0;i--){
              if(nd_collapse_[i]==1){
                local_str[1][i-1]=local_str[1][i];//set new strides
                for(int j=i+1;j<nd_collapse;j++)//remove stride i from the array
                  local_str[1][j-1]=local_str[1][j];
                }
            }
            

            for(int i=nd_collapse-1;i>0;i--){
              if(nd_collapse_[i]==1){
                local_str[2][i-1]=local_str[2][i];//set new strides
                for(int j=i+1;j<nd_collapse;j++)//remove stride i from the array
                  local_str[2][j-1]=local_str[2][j];
                }
            }
            

            for(int i=nd_collapse-1;i>0;i--){
              if(nd_collapse_[i]==1){
                local_ostr[0][i-1]=local_ostr[0][i];//set new strides
                for(int j=i+1;j<nd_collapse;j++)//remove stride i from the array
                  local_ostr[0][j-1]=local_ostr[0][j];
                }
            }
            

        for(int i=nd_collapse-1;i>0;i--){
          if(nd_collapse_[i]==1){
            local_dims[i-1]*=local_dims[i];//set new dims
            for(int j=i+1;j<nd_collapse;j++)//remove dims i from the array
              local_dims[j-1]=local_dims[j];
          }
        }
        

        for(int i=1, end=nd_collapse;i<end;i++){
          if(nd_collapse_[i]==1)nd_collapse--;
        }
        if(nd_collapse == 1 
 &&  local_str[0][nd_collapse-1]==1  && local_str[2][nd_collapse-1]==1  && local_ostr[0][nd_collapse-1]==1 
){nd_collapse=0;} 
if(numEls==0) return 0;
switch (nd_collapse==0?0:min(1,nd_collapse)) {
case 0: {

                //first use at least a full warp
                int threads_per_block = std::min(numEls,  (unsigned int)32); //WARP SIZE

                //next start adding multiprocessors
                int n_blocks = std::min(numEls/threads_per_block + (numEls % threads_per_block?1:0), (unsigned int)30); // UP TO NUMBER OF MULTIPROCESSORS

                // next start adding more warps per multiprocessor
                if (threads_per_block * n_blocks < numEls)
                    threads_per_block = std::min(numEls/n_blocks, (unsigned int)NUM_VECTOR_OP_THREADS_PER_BLOCK);
                kernel_Composite_node_3d6ea06e14a2eac7e20cc661f60176f0_0_Ccontiguous<<<n_blocks, threads_per_block>>>(numEls, i0_data, i1_data, i2_data, o0_data);

                //std::cerr << "calling callkernel returned\n";
                

                CNDA_THREAD_SYNC;
                hipError_t err = hipGetLastError();
                if( hipSuccess != err)
                {
                    PyErr_Format(PyExc_RuntimeError, "Cuda error: %s: %s.\n    n_blocks=%i threads_per_block=%i\n   Call: %s\n",
                         "GpuElemwise node_3d6ea06e14a2eac7e20cc661f60176f0_0 Composite", hipGetErrorString(err),
                         n_blocks, threads_per_block,
                         "kernel_Composite_node_3d6ea06e14a2eac7e20cc661f60176f0_0_Ccontiguous<<<n_blocks, threads_per_block>>>(numEls, i0_data, i1_data, i2_data, o0_data)");
                    return -1;

                }
                
                return 0;
                
        } break;
case 1: {

                //first use at least a full warp
                int threads_per_block = std::min(numEls, (unsigned int)32); //WARP SIZE

                //next start adding multiprocessors
                int n_blocks = std::min(numEls/threads_per_block + (numEls % threads_per_block?1:0), (unsigned int)30); // UP TO NUMBER OF MULTIPROCESSORS

                // next start adding more warps per multiprocessor
                if (threads_per_block * n_blocks < numEls)
                    threads_per_block = std::min(numEls/n_blocks, (unsigned int)NUM_VECTOR_OP_THREADS_PER_BLOCK);

                kernel_Composite_node_3d6ea06e14a2eac7e20cc661f60176f0_0_1<<<n_blocks, threads_per_block>>>(numEls, local_dims[0], i0_data, local_str[0][0], i1_data, local_str[1][0], i2_data, local_str[2][0], o0_data, local_ostr[0][0]);
                

                CNDA_THREAD_SYNC;
                hipError_t err = hipGetLastError();
                if( hipSuccess != err)
                {
                    PyErr_Format(PyExc_RuntimeError, "Cuda error: %s: %s.\n    n_blocks=%i threads_per_block=%i\n   Call: %s\n",
                         "GpuElemwise node_3d6ea06e14a2eac7e20cc661f60176f0_0 Composite", hipGetErrorString(err),
                         n_blocks, threads_per_block,
                         "kernel_Composite_node_3d6ea06e14a2eac7e20cc661f60176f0_0_Ccontiguous<<<n_blocks, threads_per_block>>>(numEls, local_dims[0], i0_data, local_str[0][0], i1_data, local_str[1][0], i2_data, local_str[2][0], o0_data, local_ostr[0][0])");
                    return -1;

                }
                return 0;
                
        } break;
}
return -2;
}


    namespace {
    struct __struct_compiled_op_3d6ea06e14a2eac7e20cc661f60176f0 {
        PyObject* __ERROR;

        PyObject* storage_V3;
PyObject* storage_V5;
PyObject* storage_V7;
PyObject* storage_V1;
        

        __struct_compiled_op_3d6ea06e14a2eac7e20cc661f60176f0() {
            // This is only somewhat safe because we:
            //  1) Are not a virtual class
            //  2) Do not use any virtual classes in the members
            //  3) Deal with mostly POD and pointers

            // If this changes, we would have to revise this, but for
            // now I am tired of chasing segfaults because
            // initialization code had an error and some pointer has
            // a junk value.
            memset(this, 0, sizeof(*this));
        }
        ~__struct_compiled_op_3d6ea06e14a2eac7e20cc661f60176f0(void) {
            cleanup();
        }

        int init(PyObject* __ERROR, PyObject* storage_V3, PyObject* storage_V5, PyObject* storage_V7, PyObject* storage_V1) {
            Py_XINCREF(storage_V3);
Py_XINCREF(storage_V5);
Py_XINCREF(storage_V7);
Py_XINCREF(storage_V1);
            this->storage_V3 = storage_V3;
this->storage_V5 = storage_V5;
this->storage_V7 = storage_V7;
this->storage_V1 = storage_V1;
            





            this->__ERROR = __ERROR;
            return 0;
        }
        void cleanup(void) {
            __label_1:

double __DUMMY_1;
__label_3:

double __DUMMY_3;
__label_5:

double __DUMMY_5;
__label_7:

double __DUMMY_7;
__label_10:

double __DUMMY_10;

            Py_XDECREF(this->storage_V3);
Py_XDECREF(this->storage_V5);
Py_XDECREF(this->storage_V7);
Py_XDECREF(this->storage_V1);
        }
        int run(void) {
            int __failure = 0;
            
    PyObject* py_V1;
     CudaNdarray * V1;
    PyObject* py_V3;
     CudaNdarray * V3;
    PyObject* py_V5;
     CudaNdarray * V5;
    PyObject* py_V7;
     CudaNdarray * V7;
{

    py_V1 = PyList_GET_ITEM(storage_V1, 0);
    {Py_XINCREF(py_V1);}
    
        if (py_V1 == Py_None)
        {
            V1 = NULL;
        }
        else
        {
            
        assert(py_V1->ob_refcnt >= 2); // There should be at least one ref from the container object,
        // and one ref from the local scope.

        if (CudaNdarray_Check(py_V1))
        {
            //fprintf(stderr, "c_extract CNDA object w refcnt %p %i\n", py_V1, (py_V1->ob_refcnt));
            V1 = (CudaNdarray*)py_V1;
            //std::cerr << "c_extract " << V1 << '\n';
        

                if (V1->nd != 1)
                {
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has rank %i, it was supposed to have rank 1",
                                 V1->nd);
                    V1 = NULL;
                    {
        __failure = 2;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_2;};
                }
                //std::cerr << "c_extract " << V1 << " nd check passed\n";
            

                assert(V1);
                Py_INCREF(py_V1);
            }
            else if (py_V1 == Py_None)
            {
                PyErr_SetString(PyExc_TypeError,
                                "expected a CudaNdarray, not None");
                V1 = NULL;
                {
        __failure = 2;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_2;};
            }
            else
            {
                //fprintf(stderr, "FAILING c_extract CNDA object w refcnt %p %i\n", py_V1, (py_V1->ob_refcnt));
                PyErr_SetString(PyExc_TypeError, "Argument not a CudaNdarray");
                V1 = NULL;
                {
        __failure = 2;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_2;};
            }
            //std::cerr << "c_extract done " << V1 << '\n';
            

        }
        
{

    py_V3 = PyList_GET_ITEM(storage_V3, 0);
    {Py_XINCREF(py_V3);}
    
        assert(py_V3->ob_refcnt >= 2); // There should be at least one ref from the container object,
        // and one ref from the local scope.

        if (CudaNdarray_Check(py_V3))
        {
            //fprintf(stderr, "c_extract CNDA object w refcnt %p %i\n", py_V3, (py_V3->ob_refcnt));
            V3 = (CudaNdarray*)py_V3;
            //std::cerr << "c_extract " << V3 << '\n';
        

                if (V3->nd != 1)
                {
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has rank %i, it was supposed to have rank 1",
                                 V3->nd);
                    V3 = NULL;
                    {
        __failure = 4;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_4;};
                }
                //std::cerr << "c_extract " << V3 << " nd check passed\n";
            

                assert(V3);
                Py_INCREF(py_V3);
            }
            else if (py_V3 == Py_None)
            {
                PyErr_SetString(PyExc_TypeError,
                                "expected a CudaNdarray, not None");
                V3 = NULL;
                {
        __failure = 4;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_4;};
            }
            else
            {
                //fprintf(stderr, "FAILING c_extract CNDA object w refcnt %p %i\n", py_V3, (py_V3->ob_refcnt));
                PyErr_SetString(PyExc_TypeError, "Argument not a CudaNdarray");
                V3 = NULL;
                {
        __failure = 4;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_4;};
            }
            //std::cerr << "c_extract done " << V3 << '\n';
            

{

    py_V5 = PyList_GET_ITEM(storage_V5, 0);
    {Py_XINCREF(py_V5);}
    
        assert(py_V5->ob_refcnt >= 2); // There should be at least one ref from the container object,
        // and one ref from the local scope.

        if (CudaNdarray_Check(py_V5))
        {
            //fprintf(stderr, "c_extract CNDA object w refcnt %p %i\n", py_V5, (py_V5->ob_refcnt));
            V5 = (CudaNdarray*)py_V5;
            //std::cerr << "c_extract " << V5 << '\n';
        

                if (V5->nd != 1)
                {
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has rank %i, it was supposed to have rank 1",
                                 V5->nd);
                    V5 = NULL;
                    {
        __failure = 6;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_6;};
                }
                //std::cerr << "c_extract " << V5 << " nd check passed\n";
            

                if (CudaNdarray_HOST_DIMS(V5)[0] != 1)
                {
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has dim %i on broadcastable dimension %i",
                                 CudaNdarray_HOST_DIMS(V5)[0], 0);
                    V5 = NULL;
                    {
        __failure = 6;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_6;};
                }
                //std::cerr << "c_extract " << V5 << "dim check 0 passed\n";
                //std::cerr << "c_extract " << V5 << "checking bcast 0 <" << V5->str<< ">\n";
                //std::cerr << "c_extract " << V5->str[0] << "\n";
                if (CudaNdarray_HOST_STRIDES(V5)[0])
                {
                    //std::cerr << "c_extract bad stride detected...\n";
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has a nonzero stride %i on a broadcastable dimension %i",
                                 CudaNdarray_HOST_STRIDES(V5)[0], 0);
                    V5 = NULL;
                    {
        __failure = 6;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_6;};
                }
                //std::cerr << "c_extract " << V5 << "bcast check 0 passed\n";
                    

                assert(V5);
                Py_INCREF(py_V5);
            }
            else if (py_V5 == Py_None)
            {
                PyErr_SetString(PyExc_TypeError,
                                "expected a CudaNdarray, not None");
                V5 = NULL;
                {
        __failure = 6;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_6;};
            }
            else
            {
                //fprintf(stderr, "FAILING c_extract CNDA object w refcnt %p %i\n", py_V5, (py_V5->ob_refcnt));
                PyErr_SetString(PyExc_TypeError, "Argument not a CudaNdarray");
                V5 = NULL;
                {
        __failure = 6;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_6;};
            }
            //std::cerr << "c_extract done " << V5 << '\n';
            

{

    py_V7 = PyList_GET_ITEM(storage_V7, 0);
    {Py_XINCREF(py_V7);}
    
        assert(py_V7->ob_refcnt >= 2); // There should be at least one ref from the container object,
        // and one ref from the local scope.

        if (CudaNdarray_Check(py_V7))
        {
            //fprintf(stderr, "c_extract CNDA object w refcnt %p %i\n", py_V7, (py_V7->ob_refcnt));
            V7 = (CudaNdarray*)py_V7;
            //std::cerr << "c_extract " << V7 << '\n';
        

                if (V7->nd != 1)
                {
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has rank %i, it was supposed to have rank 1",
                                 V7->nd);
                    V7 = NULL;
                    {
        __failure = 8;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_8;};
                }
                //std::cerr << "c_extract " << V7 << " nd check passed\n";
            

                assert(V7);
                Py_INCREF(py_V7);
            }
            else if (py_V7 == Py_None)
            {
                PyErr_SetString(PyExc_TypeError,
                                "expected a CudaNdarray, not None");
                V7 = NULL;
                {
        __failure = 8;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_8;};
            }
            else
            {
                //fprintf(stderr, "FAILING c_extract CNDA object w refcnt %p %i\n", py_V7, (py_V7->ob_refcnt));
                PyErr_SetString(PyExc_TypeError, "Argument not a CudaNdarray");
                V7 = NULL;
                {
        __failure = 8;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_8;};
            }
            //std::cerr << "c_extract done " << V7 << '\n';
            

{
// Op class GpuElemwise

        //std::cerr << "C_CODE Composite{(((-i0) / i1) * i2)} START\n";
        //standard elemwise size checks
            

            int dims[1] = {1};
            

                int broadcasts_V3[1] = {0};
                

                int broadcasts_V5[1] = {1};
                

                int broadcasts_V7[1] = {0};
                

        //std::cerr << "C_CODE Composite{(((-i0) / i1) * i2)} checking input V3\n";
        if (1 != V3->nd)
        {
            PyErr_Format(PyExc_TypeError,
                         "need 1 dims, not %i", V3->nd);
            {
        __failure = 9;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_9;};
        }
        for (int i = 0; i< 1; ++i)
        {
            dims[i] = (dims[i] == 1) ? CudaNdarray_HOST_DIMS(V3)[i] : dims[i];
            if ((!(broadcasts_V3[i] &&
                 CudaNdarray_HOST_DIMS(V3)[i] == 1)) &&
                (dims[i] != CudaNdarray_HOST_DIMS(V3)[i]))
            {
                //std::cerr << "C_CODE Composite{(((-i0) / i1) * i2)} checking input V3 failed\n";
                PyErr_Format(PyExc_ValueError,
                             "GpuElemwise. Input dimension mis-match. Input"
                             " 0 (indices start at 0) has shape[%i] == %i"
                             ", but the output's size on that axis is %i.",
                             i,
                             CudaNdarray_HOST_DIMS(V3)[i],
                             dims[i]
                            );
                {
        __failure = 9;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_9;};
            }
        }
            

        //std::cerr << "C_CODE Composite{(((-i0) / i1) * i2)} checking input V5\n";
        if (1 != V5->nd)
        {
            PyErr_Format(PyExc_TypeError,
                         "need 1 dims, not %i", V5->nd);
            {
        __failure = 9;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_9;};
        }
        for (int i = 0; i< 1; ++i)
        {
            dims[i] = (dims[i] == 1) ? CudaNdarray_HOST_DIMS(V5)[i] : dims[i];
            if ((!(broadcasts_V5[i] &&
                 CudaNdarray_HOST_DIMS(V5)[i] == 1)) &&
                (dims[i] != CudaNdarray_HOST_DIMS(V5)[i]))
            {
                //std::cerr << "C_CODE Composite{(((-i0) / i1) * i2)} checking input V5 failed\n";
                PyErr_Format(PyExc_ValueError,
                             "GpuElemwise. Input dimension mis-match. Input"
                             " 1 (indices start at 0) has shape[%i] == %i"
                             ", but the output's size on that axis is %i.",
                             i,
                             CudaNdarray_HOST_DIMS(V5)[i],
                             dims[i]
                            );
                {
        __failure = 9;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_9;};
            }
        }
            

        //std::cerr << "C_CODE Composite{(((-i0) / i1) * i2)} checking input V7\n";
        if (1 != V7->nd)
        {
            PyErr_Format(PyExc_TypeError,
                         "need 1 dims, not %i", V7->nd);
            {
        __failure = 9;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_9;};
        }
        for (int i = 0; i< 1; ++i)
        {
            dims[i] = (dims[i] == 1) ? CudaNdarray_HOST_DIMS(V7)[i] : dims[i];
            if ((!(broadcasts_V7[i] &&
                 CudaNdarray_HOST_DIMS(V7)[i] == 1)) &&
                (dims[i] != CudaNdarray_HOST_DIMS(V7)[i]))
            {
                //std::cerr << "C_CODE Composite{(((-i0) / i1) * i2)} checking input V7 failed\n";
                PyErr_Format(PyExc_ValueError,
                             "GpuElemwise. Input dimension mis-match. Input"
                             " 2 (indices start at 0) has shape[%i] == %i"
                             ", but the output's size on that axis is %i.",
                             i,
                             CudaNdarray_HOST_DIMS(V7)[i],
                             dims[i]
                            );
                {
        __failure = 9;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_9;};
            }
        }
            

        Py_XDECREF(V1);
        V1 = V3;
        Py_INCREF(V1);
        for (int i = 0; (i< 1) && (V1); ++i) {
            if (dims[i] != CudaNdarray_HOST_DIMS(V1)[i])
            {
                PyErr_Format(PyExc_ValueError,
                             "GpuElemwise. Output dimension mis-match. Output"
                             " 0 (indices start at 0), working inplace"
                             " on input 0, has shape[%i] == %i"
                             ", but the output's size on that axis is %i.",
                             i,
                             CudaNdarray_HOST_DIMS(V1)[i],
                             dims[i]
                            );
                Py_DECREF(V1);
                V1 = NULL;
                {
        __failure = 9;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_9;};
            }
        }
        //std::cerr << "ELEMWISE NEW V1 nd" << V1->nd << "\n";
        //std::cerr << "ELEMWISE NEW V1 data" << V1->devdata << "\n";
        

        {
            //new block so that failure gotos don't skip over variable initialization
            //std::cerr << "calling callkernel\n";
            if (callkernel_node_3d6ea06e14a2eac7e20cc661f60176f0_0(1, 0, dims
            

                        , CudaNdarray_DEV_DATA(V3), CudaNdarray_HOST_STRIDES(V3)
            

                        , CudaNdarray_DEV_DATA(V5), CudaNdarray_HOST_STRIDES(V5)
            

                        , CudaNdarray_DEV_DATA(V7), CudaNdarray_HOST_STRIDES(V7)
            

                        , CudaNdarray_DEV_DATA(V1), CudaNdarray_HOST_STRIDES(V1)
            

                        ))
            {
                 // error
            

                Py_DECREF(V1);
                V1 = NULL;
                

                {
        __failure = 9;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_9;};
            }
            else // no error
            {
            }
        }
        //std::cerr << "C_CODE Composite{(((-i0) / i1) * i2)} END\n";
        
__label_9:

double __DUMMY_9;

}
__label_8:

        //std::cerr << "cleanup " << py_V7 << " " << V7 << "\n";
        //fprintf(stderr, "c_cleanup CNDA py_object w refcnt %p %i\n", py_V7, (py_V7->ob_refcnt));
        if (V7)
        {
            //fprintf(stderr, "c_cleanup CNDA cn_object w refcnt %p %i\n", V7, (V7->ob_refcnt));
            Py_XDECREF(V7);
        }
        //std::cerr << "cleanup done" << py_V7 << "\n";
        
    {Py_XDECREF(py_V7);}
    
double __DUMMY_8;

}
__label_6:

        //std::cerr << "cleanup " << py_V5 << " " << V5 << "\n";
        //fprintf(stderr, "c_cleanup CNDA py_object w refcnt %p %i\n", py_V5, (py_V5->ob_refcnt));
        if (V5)
        {
            //fprintf(stderr, "c_cleanup CNDA cn_object w refcnt %p %i\n", V5, (V5->ob_refcnt));
            Py_XDECREF(V5);
        }
        //std::cerr << "cleanup done" << py_V5 << "\n";
        
    {Py_XDECREF(py_V5);}
    
double __DUMMY_6;

}
__label_4:

        //std::cerr << "cleanup " << py_V3 << " " << V3 << "\n";
        //fprintf(stderr, "c_cleanup CNDA py_object w refcnt %p %i\n", py_V3, (py_V3->ob_refcnt));
        if (V3)
        {
            //fprintf(stderr, "c_cleanup CNDA cn_object w refcnt %p %i\n", V3, (V3->ob_refcnt));
            Py_XDECREF(V3);
        }
        //std::cerr << "cleanup done" << py_V3 << "\n";
        
    {Py_XDECREF(py_V3);}
    
double __DUMMY_4;

}
__label_2:

    if (!__failure) {
      
        //std::cerr << "sync\n";
        if (NULL == V1) {
            // failure: sync None to storage
            Py_XDECREF(py_V1);
            py_V1 = Py_None;
            Py_INCREF(py_V1);
        }
        else
        {
            if (py_V1 != (PyObject*)V1)
            {
                Py_XDECREF(py_V1);
                py_V1 = (PyObject*)V1;
                Py_INCREF(py_V1);
            }
            assert(py_V1->ob_refcnt);
        }
        
      PyObject* old = PyList_GET_ITEM(storage_V1, 0);
      {Py_XINCREF(py_V1);}
      PyList_SET_ITEM(storage_V1, 0, py_V1);
      {Py_XDECREF(old);}
    }
    
        //std::cerr << "cleanup " << py_V1 << " " << V1 << "\n";
        //fprintf(stderr, "c_cleanup CNDA py_object w refcnt %p %i\n", py_V1, (py_V1->ob_refcnt));
        if (V1)
        {
            //fprintf(stderr, "c_cleanup CNDA cn_object w refcnt %p %i\n", V1, (V1->ob_refcnt));
            Py_XDECREF(V1);
        }
        //std::cerr << "cleanup done" << py_V1 << "\n";
        
    {Py_XDECREF(py_V1);}
    
double __DUMMY_2;

}

            
        if (__failure) {
            // When there is a failure, this code puts the exception
            // in __ERROR.
            PyObject* err_type = NULL;
            PyObject* err_msg = NULL;
            PyObject* err_traceback = NULL;
            PyErr_Fetch(&err_type, &err_msg, &err_traceback);
            if (!err_type) {err_type = Py_None;Py_INCREF(Py_None);}
            if (!err_msg) {err_msg = Py_None; Py_INCREF(Py_None);}
            if (!err_traceback) {err_traceback = Py_None; Py_INCREF(Py_None);}
            PyObject* old_err_type = PyList_GET_ITEM(__ERROR, 0);
            PyObject* old_err_msg = PyList_GET_ITEM(__ERROR, 1);
            PyObject* old_err_traceback = PyList_GET_ITEM(__ERROR, 2);
            PyList_SET_ITEM(__ERROR, 0, err_type);
            PyList_SET_ITEM(__ERROR, 1, err_msg);
            PyList_SET_ITEM(__ERROR, 2, err_traceback);
            {Py_XDECREF(old_err_type);}
            {Py_XDECREF(old_err_msg);}
            {Py_XDECREF(old_err_traceback);}
        }
        // The failure code is returned to index what code block failed.
        return __failure;
        
        }
    };
    }
    

        static int __struct_compiled_op_3d6ea06e14a2eac7e20cc661f60176f0_executor(__struct_compiled_op_3d6ea06e14a2eac7e20cc661f60176f0* self) {
            return self->run();
        }

        static void __struct_compiled_op_3d6ea06e14a2eac7e20cc661f60176f0_destructor(void* executor, void* self) {
            delete ((__struct_compiled_op_3d6ea06e14a2eac7e20cc661f60176f0*)self);
        }
        
//////////////////////
////  Functions
//////////////////////
static PyObject * instantiate(PyObject * self, PyObject *argtuple) {
  assert(PyTuple_Check(argtuple));
  if (5 != PyTuple_Size(argtuple)){ 
     PyErr_Format(PyExc_TypeError, "Wrong number of arguments, expected 5, got %i", (int)PyTuple_Size(argtuple));
     return NULL;
  }
  __struct_compiled_op_3d6ea06e14a2eac7e20cc661f60176f0* struct_ptr = new __struct_compiled_op_3d6ea06e14a2eac7e20cc661f60176f0();
  if (struct_ptr->init( PyTuple_GET_ITEM(argtuple, 0),PyTuple_GET_ITEM(argtuple, 1),PyTuple_GET_ITEM(argtuple, 2),PyTuple_GET_ITEM(argtuple, 3),PyTuple_GET_ITEM(argtuple, 4) ) != 0) {
    delete struct_ptr;
    return NULL;
  }
  PyObject* thunk = PyCObject_FromVoidPtrAndDesc((void*)(&__struct_compiled_op_3d6ea06e14a2eac7e20cc661f60176f0_executor), struct_ptr, __struct_compiled_op_3d6ea06e14a2eac7e20cc661f60176f0_destructor);
  return thunk; }

//////////////////////
////  Module init
//////////////////////
static PyMethodDef MyMethods[] = {
	{"instantiate", instantiate, METH_VARARGS, "undocumented"} ,
	{NULL, NULL, 0, NULL}
};
PyMODINIT_FUNC init3d6ea06e14a2eac7e20cc661f60176f0(void){
   (void) Py_InitModule("3d6ea06e14a2eac7e20cc661f60176f0", MyMethods);
}
