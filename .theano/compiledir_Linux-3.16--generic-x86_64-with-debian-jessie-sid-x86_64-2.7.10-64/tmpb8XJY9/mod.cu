#include "hip/hip_runtime.h"
#include <Python.h>
#include <iostream>
#include "theano_mod_helper.h"
#include "cuda_ndarray.cuh"
#include <math.h>
#include <numpy/arrayobject.h>
#include <numpy/arrayscalars.h>
//////////////////////
////  Support Code
//////////////////////


        // FB: I disable the printing of the warning, as we
        //receive too much email about this and this don't help
        //people. I'm not even sure if the "fix" to give the info about
        //the shape statically give a speed up. So I consider this
        //warning as useless until proved it can speed the user code.
        static int node_03591e6f7304156e3b956ee04cd01af3_0_printed_warning = 1;

        static __global__ void node_03591e6f7304156e3b956ee04cd01af3_0_mrg_uniform(
                float*sample_data,
                npy_int32*state_data,
                const int Nsamples,
                const int Nstreams_used)
        {
            const npy_int32 i0 = 0;
            const npy_int32 i7 = 7;
            const npy_int32 i9 = 9;
            const npy_int32 i15 = 15;
            const npy_int32 i16 = 16;
            const npy_int32 i22 = 22;
            const npy_int32 i24 = 24;

            const npy_int32 M1 = 2147483647;      //2^31 - 1
            const npy_int32 M2 = 2147462579;      //2^31 - 21069
            const npy_int32 MASK12 = 511;       //2^9 - 1
            const npy_int32 MASK13 = 16777215;  //2^24 - 1
            const npy_int32 MASK2 = 65535;      //2^16 - 1
            const npy_int32 MULT2 = 21069;

            const unsigned int numThreads = blockDim.x * gridDim.x;
            const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
            npy_int32 y1, y2, x11, x12, x13, x21, x22, x23;

            if (idx < Nstreams_used)
            {
            x11 = state_data[idx*6+0];
            x12 = state_data[idx*6+1];
            x13 = state_data[idx*6+2];
            x21 = state_data[idx*6+3];
            x22 = state_data[idx*6+4];
            x23 = state_data[idx*6+5];

            for (int i = idx; i < Nsamples; i += Nstreams_used)
            {
                y1 = ((x12 & MASK12) << i22) + (x12 >> i9) + ((x13 & MASK13) << i7) + (x13 >> i24);
                y1 -= (y1 < 0 || y1 >= M1) ? M1 : 0;
                y1 += x13;
                y1 -= (y1 < 0 || y1 >= M1) ? M1 : 0;
                x13 = x12;
                x12 = x11;
                x11 = y1;

                y1 = ((x21 & MASK2) << i15) + (MULT2 * (x21 >> i16));
                y1 -= (y1 < 0 || y1 >= M2) ? M2 : 0;
                y2 = ((x23 & MASK2) << i15) + (MULT2 * (x23 >> i16));
                y2 -= (y2 < 0 || y2 >= M2) ? M2 : 0;
                y2 += x23;
                y2 -= (y2 < 0 || y2 >= M2) ? M2 : 0;
                y2 += y1;
                y2 -= (y2 < 0 || y2 >= M2) ? M2 : 0;

                x23 = x22;
                x22 = x21;
                x21 = y2;

                if (x11 <= x21) {
                    sample_data[i] = (x11 - x21 + M1) * 4.6566126e-10f;
                }
                else
                {
                    sample_data[i] = (x11 - x21) * 4.6566126e-10f;
                }
            }

            state_data[idx*6+0]= x11;
            state_data[idx*6+1]= x12;
            state_data[idx*6+2]= x13;
            state_data[idx*6+3]= x21;
            state_data[idx*6+4]= x22;
            state_data[idx*6+5]= x23;
            }
        }

        

    namespace {
    struct __struct_compiled_op_03591e6f7304156e3b956ee04cd01af3 {
        PyObject* __ERROR;

        PyObject* storage_V3;
PyObject* storage_V5;
PyObject* storage_V7;
PyObject* storage_V1;
        

        __struct_compiled_op_03591e6f7304156e3b956ee04cd01af3() {
            // This is only somewhat safe because we:
            //  1) Are not a virtual class
            //  2) Do not use any virtual classes in the members
            //  3) Deal with mostly POD and pointers

            // If this changes, we would have to revise this, but for
            // now I am tired of chasing segfaults because
            // initialization code had an error and some pointer has
            // a junk value.
            memset(this, 0, sizeof(*this));
        }
        ~__struct_compiled_op_03591e6f7304156e3b956ee04cd01af3(void) {
            cleanup();
        }

        int init(PyObject* __ERROR, PyObject* storage_V3, PyObject* storage_V5, PyObject* storage_V7, PyObject* storage_V1) {
            Py_XINCREF(storage_V3);
Py_XINCREF(storage_V5);
Py_XINCREF(storage_V7);
Py_XINCREF(storage_V1);
            this->storage_V3 = storage_V3;
this->storage_V5 = storage_V5;
this->storage_V7 = storage_V7;
this->storage_V1 = storage_V1;
            





            this->__ERROR = __ERROR;
            return 0;
        }
        void cleanup(void) {
            __label_1:

double __DUMMY_1;
__label_3:

double __DUMMY_3;
__label_5:

double __DUMMY_5;
__label_7:

double __DUMMY_7;
__label_10:

double __DUMMY_10;

            Py_XDECREF(this->storage_V3);
Py_XDECREF(this->storage_V5);
Py_XDECREF(this->storage_V7);
Py_XDECREF(this->storage_V1);
        }
        int run(void) {
            int __failure = 0;
            
    PyObject* py_V1;
     CudaNdarray * V1;
    PyObject* py_V3;
     CudaNdarray * V3;
    PyObject* py_V5;
    
        PyArrayObject* V5;
        
            typedef npy_int32 dtype_V5;
            
    PyObject* py_V7;
     CudaNdarray * V7;
{

    py_V1 = PyList_GET_ITEM(storage_V1, 0);
    {Py_XINCREF(py_V1);}
    
        if (py_V1 == Py_None)
        {
            V1 = NULL;
        }
        else
        {
            
        assert(py_V1->ob_refcnt >= 2); // There should be at least one ref from the container object,
        // and one ref from the local scope.

        if (CudaNdarray_Check(py_V1))
        {
            //fprintf(stderr, "c_extract CNDA object w refcnt %p %i\n", py_V1, (py_V1->ob_refcnt));
            V1 = (CudaNdarray*)py_V1;
            //std::cerr << "c_extract " << V1 << '\n';
        

                if (V1->nd != 2)
                {
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has rank %i, it was supposed to have rank 2",
                                 V1->nd);
                    V1 = NULL;
                    {
        __failure = 2;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_2;};
                }
                //std::cerr << "c_extract " << V1 << " nd check passed\n";
            

                assert(V1);
                Py_INCREF(py_V1);
            }
            else if (py_V1 == Py_None)
            {
                PyErr_SetString(PyExc_TypeError,
                                "expected a CudaNdarray, not None");
                V1 = NULL;
                {
        __failure = 2;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_2;};
            }
            else
            {
                //fprintf(stderr, "FAILING c_extract CNDA object w refcnt %p %i\n", py_V1, (py_V1->ob_refcnt));
                PyErr_SetString(PyExc_TypeError, "Argument not a CudaNdarray");
                V1 = NULL;
                {
        __failure = 2;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_2;};
            }
            //std::cerr << "c_extract done " << V1 << '\n';
            

        }
        
{

    py_V3 = PyList_GET_ITEM(storage_V3, 0);
    {Py_XINCREF(py_V3);}
    
        assert(py_V3->ob_refcnt >= 2); // There should be at least one ref from the container object,
        // and one ref from the local scope.

        if (CudaNdarray_Check(py_V3))
        {
            //fprintf(stderr, "c_extract CNDA object w refcnt %p %i\n", py_V3, (py_V3->ob_refcnt));
            V3 = (CudaNdarray*)py_V3;
            //std::cerr << "c_extract " << V3 << '\n';
        

                if (V3->nd != 1)
                {
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has rank %i, it was supposed to have rank 1",
                                 V3->nd);
                    V3 = NULL;
                    {
        __failure = 4;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_4;};
                }
                //std::cerr << "c_extract " << V3 << " nd check passed\n";
            

                assert(V3);
                Py_INCREF(py_V3);
            }
            else if (py_V3 == Py_None)
            {
                PyErr_SetString(PyExc_TypeError,
                                "expected a CudaNdarray, not None");
                V3 = NULL;
                {
        __failure = 4;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_4;};
            }
            else
            {
                //fprintf(stderr, "FAILING c_extract CNDA object w refcnt %p %i\n", py_V3, (py_V3->ob_refcnt));
                PyErr_SetString(PyExc_TypeError, "Argument not a CudaNdarray");
                V3 = NULL;
                {
        __failure = 4;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_4;};
            }
            //std::cerr << "c_extract done " << V3 << '\n';
            

{

    py_V5 = PyList_GET_ITEM(storage_V5, 0);
    {Py_XINCREF(py_V5);}
    
            V5 = NULL;
            if (py_V5 == Py_None) {
                // We can either fail here or set V5 to NULL and rely on Ops
                // using tensors to handle the NULL case, but if they fail to do so
                // they'll end up with nasty segfaults, so this is public service.
                PyErr_SetString(PyExc_ValueError, "expected an ndarray, not None");
                {
        __failure = 6;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_6;}
            }
            if (!PyArray_Check(py_V5)) {
                PyErr_SetString(PyExc_ValueError, "expected an ndarray");
                {
        __failure = 6;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_6;}
            }
            // We expect NPY_INT32
            if (!PyArray_ISALIGNED((PyArrayObject*) py_V5)) {
                PyArrayObject * tmp = (PyArrayObject*) py_V5;
                PyErr_Format(PyExc_NotImplementedError,
                             "expected an aligned array of type %ld "
                             "(NPY_INT32), got non-aligned array of type %ld"
                             " with %ld dimensions, with 3 last dims "
                             "%ld, %ld, %ld"
                             " and 3 last strides %ld %ld, %ld.",
                             (long int) NPY_INT32,
                             (long int) PyArray_TYPE((PyArrayObject*) py_V5),
                             (long int) PyArray_NDIM(tmp),
                             (long int) PyArray_NDIM(tmp) >= 3 ?
            PyArray_DIMS(tmp)[PyArray_NDIM(tmp)-3] : -1,
                             (long int) PyArray_NDIM(tmp) >= 2 ?
            PyArray_DIMS(tmp)[PyArray_NDIM(tmp)-2] : -1,
                             (long int) PyArray_NDIM(tmp) >= 1 ?
            PyArray_DIMS(tmp)[PyArray_NDIM(tmp)-1] : -1,
                             (long int) PyArray_NDIM(tmp) >= 3 ?
            PyArray_STRIDES(tmp)[PyArray_NDIM(tmp)-3] : -1,
                             (long int) PyArray_NDIM(tmp) >= 2 ?
            PyArray_STRIDES(tmp)[PyArray_NDIM(tmp)-2] : -1,
                             (long int) PyArray_NDIM(tmp) >= 1 ?
            PyArray_STRIDES(tmp)[PyArray_NDIM(tmp)-1] : -1
            );
                {
        __failure = 6;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_6;}
            }
            // This is a TypeError to be consistent with DEBUG_MODE
            // Note: DEBUG_MODE also tells the name of the container
            if (PyArray_TYPE((PyArrayObject*) py_V5) != NPY_INT32) {
                PyErr_Format(PyExc_TypeError,
                             "expected type_num %d (NPY_INT32) got %d",
                             NPY_INT32, PyArray_TYPE((PyArrayObject*) py_V5));
                {
        __failure = 6;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_6;}
            }
            
        V5 = (PyArrayObject*)(py_V5);
        Py_XINCREF(V5);
        
{

    py_V7 = Py_None;
    {Py_XINCREF(py_V7);}
    V7 = NULL;
{
// Op class GPU_mrg_uniform

        //////// <code generated by mrg_uniform>

        int odims[2];
        int n_elements = 1;
        int n_streams, n_streams_used_in_this_call;
        int must_alloc_sample = ((NULL == V1)
                || !CudaNdarray_Check((PyObject*)V1)
                || !CudaNdarray_is_c_contiguous(V1)
                || (CudaNdarray_NDIM(V1) != 2));

        if (PyArray_NDIM(V5) != 1)
        {
            PyErr_SetString(PyExc_ValueError, "size must be vector");
            {
        __failure = 9;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_9;}
        }
        if (PyArray_DIMS(V5)[0] != 2)
        {
            PyErr_Format(PyExc_ValueError, "size must have length %i (not %i)",
                2, PyArray_DIMS(V5)[0]);
            {
        __failure = 9;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_9;}
        }
        if (PyArray_DESCR(V5)->type_num != NPY_INT32)
        {
            PyErr_SetString(PyExc_ValueError, "size must be int32");
            {
        __failure = 9;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_9;}
        }
        for (int i = 0; i < 2; ++i)
        {
            odims[i] = ((npy_int32*)(PyArray_BYTES(V5) + PyArray_STRIDES(V5)[0] * i))[0];
            n_elements *= odims[i];
            must_alloc_sample = (must_alloc_sample
                    || CudaNdarray_HOST_DIMS(V1)[i] != odims[i]);
        }
        if (must_alloc_sample)
        {
            Py_XDECREF(V1);
            V1 = (CudaNdarray*)CudaNdarray_NewDims(2, odims);
            if(!V1)
            {
                {
        __failure = 9;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_9;};
            }
        }
        if (!CudaNdarray_Check((PyObject*)V3))
        {
            PyErr_Format(PyExc_ValueError, "rstate must be cudandarray");
            {
        __failure = 9;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_9;};
        }

        Py_XDECREF(V7);
        if (1)
        {
            Py_INCREF(V3);
            V7 = V3;
        }
        else
        {
            V7 = (CudaNdarray*)CudaNdarray_Copy(V3);
            if (!V7) {
                PyErr_SetString(PyExc_RuntimeError, "GPU_mrg_uniform: "
                                "could not copy rstate");
                {
        __failure = 9;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_9;}
            }
        }

        if (CudaNdarray_NDIM(V7) != 1)
        {
            PyErr_SetString(PyExc_ValueError, "rstate must be vector");
            {
        __failure = 9;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_9;};
        }
        if (CudaNdarray_HOST_DIMS(V7)[0] % 6)
        {
            PyErr_Format(PyExc_ValueError, "rstate len must be multiple of 6");
            {
        __failure = 9;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_9;};
        }
        n_streams = CudaNdarray_HOST_DIMS(V7)[0]/6;
        n_streams_used_in_this_call = std::min(n_streams, n_elements);

        {
            unsigned int threads_per_block = std::min((unsigned int)n_streams_used_in_this_call, (unsigned int)NUM_VECTOR_OP_THREADS_PER_BLOCK);
            unsigned int n_blocks = std::min(ceil_intdiv((unsigned int)n_streams_used_in_this_call, threads_per_block), (unsigned int)NUM_VECTOR_OP_BLOCKS);

            if (n_streams > (unsigned int)NUM_VECTOR_OP_THREADS_PER_BLOCK * (unsigned int)NUM_VECTOR_OP_BLOCKS)
            {
                PyErr_Format(PyExc_ValueError, "On GPU, n_streams should be at most %u",
                    (unsigned int)NUM_VECTOR_OP_THREADS_PER_BLOCK * (unsigned int)NUM_VECTOR_OP_BLOCKS);
                {
        __failure = 9;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_9;};
            }

            if (threads_per_block * n_blocks < n_streams)
            {
                if (! node_03591e6f7304156e3b956ee04cd01af3_0_printed_warning)
                  fprintf(stderr, "WARNING: unused streams above %i (Tune GPU_mrg get_n_streams)\n", threads_per_block * n_blocks );
                node_03591e6f7304156e3b956ee04cd01af3_0_printed_warning = 1;
            }
            node_03591e6f7304156e3b956ee04cd01af3_0_mrg_uniform<<<n_blocks,threads_per_block>>>(
                CudaNdarray_DEV_DATA(V1),
                (npy_int32*)CudaNdarray_DEV_DATA(V7),
                n_elements, n_streams_used_in_this_call);
        }

        CNDA_THREAD_SYNC;

        {
            hipError_t err = hipGetLastError();
            if( hipSuccess != err)
            {
                PyErr_Format(PyExc_RuntimeError, "Cuda error: %s: %s.\n", "mrg_uniform", hipGetErrorString(err));
                {
        __failure = 9;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_9;};
            }
        }

        //////// </ code generated by mrg_uniform>
        __label_9:

double __DUMMY_9;

}
__label_8:

    if (!__failure) {
      
        //std::cerr << "sync\n";
        if (NULL == V7) {
            // failure: sync None to storage
            Py_XDECREF(py_V7);
            py_V7 = Py_None;
            Py_INCREF(py_V7);
        }
        else
        {
            if (py_V7 != (PyObject*)V7)
            {
                Py_XDECREF(py_V7);
                py_V7 = (PyObject*)V7;
                Py_INCREF(py_V7);
            }
            assert(py_V7->ob_refcnt);
        }
        
      PyObject* old = PyList_GET_ITEM(storage_V7, 0);
      {Py_XINCREF(py_V7);}
      PyList_SET_ITEM(storage_V7, 0, py_V7);
      {Py_XDECREF(old);}
    }
    
        //std::cerr << "cleanup " << py_V7 << " " << V7 << "\n";
        //fprintf(stderr, "c_cleanup CNDA py_object w refcnt %p %i\n", py_V7, (py_V7->ob_refcnt));
        if (V7)
        {
            //fprintf(stderr, "c_cleanup CNDA cn_object w refcnt %p %i\n", V7, (V7->ob_refcnt));
            Py_XDECREF(V7);
        }
        //std::cerr << "cleanup done" << py_V7 << "\n";
        
    {Py_XDECREF(py_V7);}
    
double __DUMMY_8;

}
__label_6:

        if (V5) {
            Py_XDECREF(V5);
        }
        
    {Py_XDECREF(py_V5);}
    
double __DUMMY_6;

}
__label_4:

        //std::cerr << "cleanup " << py_V3 << " " << V3 << "\n";
        //fprintf(stderr, "c_cleanup CNDA py_object w refcnt %p %i\n", py_V3, (py_V3->ob_refcnt));
        if (V3)
        {
            //fprintf(stderr, "c_cleanup CNDA cn_object w refcnt %p %i\n", V3, (V3->ob_refcnt));
            Py_XDECREF(V3);
        }
        //std::cerr << "cleanup done" << py_V3 << "\n";
        
    {Py_XDECREF(py_V3);}
    
double __DUMMY_4;

}
__label_2:

    if (!__failure) {
      
        //std::cerr << "sync\n";
        if (NULL == V1) {
            // failure: sync None to storage
            Py_XDECREF(py_V1);
            py_V1 = Py_None;
            Py_INCREF(py_V1);
        }
        else
        {
            if (py_V1 != (PyObject*)V1)
            {
                Py_XDECREF(py_V1);
                py_V1 = (PyObject*)V1;
                Py_INCREF(py_V1);
            }
            assert(py_V1->ob_refcnt);
        }
        
      PyObject* old = PyList_GET_ITEM(storage_V1, 0);
      {Py_XINCREF(py_V1);}
      PyList_SET_ITEM(storage_V1, 0, py_V1);
      {Py_XDECREF(old);}
    }
    
        //std::cerr << "cleanup " << py_V1 << " " << V1 << "\n";
        //fprintf(stderr, "c_cleanup CNDA py_object w refcnt %p %i\n", py_V1, (py_V1->ob_refcnt));
        if (V1)
        {
            //fprintf(stderr, "c_cleanup CNDA cn_object w refcnt %p %i\n", V1, (V1->ob_refcnt));
            Py_XDECREF(V1);
        }
        //std::cerr << "cleanup done" << py_V1 << "\n";
        
    {Py_XDECREF(py_V1);}
    
double __DUMMY_2;

}

            
        if (__failure) {
            // When there is a failure, this code puts the exception
            // in __ERROR.
            PyObject* err_type = NULL;
            PyObject* err_msg = NULL;
            PyObject* err_traceback = NULL;
            PyErr_Fetch(&err_type, &err_msg, &err_traceback);
            if (!err_type) {err_type = Py_None;Py_INCREF(Py_None);}
            if (!err_msg) {err_msg = Py_None; Py_INCREF(Py_None);}
            if (!err_traceback) {err_traceback = Py_None; Py_INCREF(Py_None);}
            PyObject* old_err_type = PyList_GET_ITEM(__ERROR, 0);
            PyObject* old_err_msg = PyList_GET_ITEM(__ERROR, 1);
            PyObject* old_err_traceback = PyList_GET_ITEM(__ERROR, 2);
            PyList_SET_ITEM(__ERROR, 0, err_type);
            PyList_SET_ITEM(__ERROR, 1, err_msg);
            PyList_SET_ITEM(__ERROR, 2, err_traceback);
            {Py_XDECREF(old_err_type);}
            {Py_XDECREF(old_err_msg);}
            {Py_XDECREF(old_err_traceback);}
        }
        // The failure code is returned to index what code block failed.
        return __failure;
        
        }
    };
    }
    

        static int __struct_compiled_op_03591e6f7304156e3b956ee04cd01af3_executor(__struct_compiled_op_03591e6f7304156e3b956ee04cd01af3* self) {
            return self->run();
        }

        static void __struct_compiled_op_03591e6f7304156e3b956ee04cd01af3_destructor(void* executor, void* self) {
            delete ((__struct_compiled_op_03591e6f7304156e3b956ee04cd01af3*)self);
        }
        
//////////////////////
////  Functions
//////////////////////
static PyObject * instantiate(PyObject * self, PyObject *argtuple) {
  assert(PyTuple_Check(argtuple));
  if (5 != PyTuple_Size(argtuple)){ 
     PyErr_Format(PyExc_TypeError, "Wrong number of arguments, expected 5, got %i", (int)PyTuple_Size(argtuple));
     return NULL;
  }
  __struct_compiled_op_03591e6f7304156e3b956ee04cd01af3* struct_ptr = new __struct_compiled_op_03591e6f7304156e3b956ee04cd01af3();
  if (struct_ptr->init( PyTuple_GET_ITEM(argtuple, 0),PyTuple_GET_ITEM(argtuple, 1),PyTuple_GET_ITEM(argtuple, 2),PyTuple_GET_ITEM(argtuple, 3),PyTuple_GET_ITEM(argtuple, 4) ) != 0) {
    delete struct_ptr;
    return NULL;
  }
  PyObject* thunk = PyCObject_FromVoidPtrAndDesc((void*)(&__struct_compiled_op_03591e6f7304156e3b956ee04cd01af3_executor), struct_ptr, __struct_compiled_op_03591e6f7304156e3b956ee04cd01af3_destructor);
  return thunk; }

//////////////////////
////  Module init
//////////////////////
static PyMethodDef MyMethods[] = {
	{"instantiate", instantiate, METH_VARARGS, "undocumented"} ,
	{NULL, NULL, 0, NULL}
};
PyMODINIT_FUNC init03591e6f7304156e3b956ee04cd01af3(void){
   import_array();
   (void) Py_InitModule("03591e6f7304156e3b956ee04cd01af3", MyMethods);
}
