#include "hip/hip_runtime.h"
#include <Python.h>
#include <iostream>
#include "theano_mod_helper.h"
#include "cuda_ndarray.cuh"
//////////////////////
////  Support Code
//////////////////////


#define INTDIV_POW2(a, b) (a >> b)
#define INTMOD_POW2(a, b) (a & ((1<<b)-1))
        // GpuElemwise{Composite{((i0 * sqrt(clip((i1 - (i2 ** i3)), i4, i5))) / (i1 - (i6 ** i3)))},no_inplace}
// node.op.destroy_map={}
//    Input   0 CudaNdarrayType(float32, scalar)
//    Input   1 CudaNdarrayType(float32, scalar)
//    Input   2 CudaNdarrayType(float32, scalar)
//    Input   3 CudaNdarrayType(float32, scalar)
//    Input   4 CudaNdarrayType(float32, scalar)
//    Input   5 CudaNdarrayType(float32, scalar)
//    Input   6 CudaNdarrayType(float32, scalar)
//    Output  0 CudaNdarrayType(float32, scalar)
static __global__ void kernel_Composite_node_a75fe51b78cd45e598008b19e82e1aec_0_Ccontiguous (unsigned int numEls
	, const float * i0_data
	, const float * i1_data
	, const float * i2_data
	, const float * i3_data
	, const float * i4_data
	, const float * i5_data
	, const float * i6_data
	, float * o0_data
	)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    const float ii_i0_value = i0_data[0];
    const float ii_i1_value = i1_data[0];
    const float ii_i2_value = i2_data[0];
    const float ii_i3_value = i3_data[0];
    const float ii_i4_value = i4_data[0];
    const float ii_i5_value = i5_data[0];
    const float ii_i6_value = i6_data[0];
    for (int i = idx; i < numEls; i += numThreads) {
npy_float32 o0_i;
        {
npy_float32 V_DUMMY_ID__tmp1;
V_DUMMY_ID__tmp1 = pow(ii_i6_value, ii_i3_value);
npy_float32 V_DUMMY_ID__tmp2;
V_DUMMY_ID__tmp2 = pow(ii_i2_value, ii_i3_value);
npy_float32 V_DUMMY_ID__tmp3;
V_DUMMY_ID__tmp3 = ii_i1_value - V_DUMMY_ID__tmp1;
npy_float32 V_DUMMY_ID__tmp4;
V_DUMMY_ID__tmp4 = ii_i1_value - V_DUMMY_ID__tmp2;
npy_float32 V_DUMMY_ID__tmp5;
V_DUMMY_ID__tmp5 = V_DUMMY_ID__tmp4 < ii_i4_value ? ii_i4_value : V_DUMMY_ID__tmp4 > ii_i5_value ? ii_i5_value : V_DUMMY_ID__tmp4;
npy_float32 V_DUMMY_ID__tmp6;
V_DUMMY_ID__tmp6 = sqrt(V_DUMMY_ID__tmp5);
npy_float32 V_DUMMY_ID__tmp7;
V_DUMMY_ID__tmp7 = ii_i0_value * V_DUMMY_ID__tmp6;
o0_i = V_DUMMY_ID__tmp7 / V_DUMMY_ID__tmp3;
}

o0_data[i] = o0_i;
    }
}

        static void can_collapse_node_a75fe51b78cd45e598008b19e82e1aec_0(int nd, const int * dims, const int * strides, int collapse[])
        {
            //can we collapse dims[i] and dims[i-1]
            for(int i=nd-1;i>0;i--){
                if(strides[i]*dims[i]==strides[i-1]){//the dims nd-1 are not strided again dimension nd
                    collapse[i]=1;
                }else collapse[i]=0;
            }
        }
        

        static int callkernel_node_a75fe51b78cd45e598008b19e82e1aec_0(unsigned int numEls, const int d,
            const int * dims,
            const float * i0_data, const int * i0_str, const float * i1_data, const int * i1_str, const float * i2_data, const int * i2_str, const float * i3_data, const int * i3_str, const float * i4_data, const int * i4_str, const float * i5_data, const int * i5_str, const float * i6_data, const int * i6_str,
            float * o0_data, const int * o0_str)
        {
            numEls = 1;
        
int *local_dims=NULL;

            int local_str[1][1];
            int local_ostr[1][1];
            

        int nd_collapse = 0;
        for(int i=0;i<0;i++){//init new dim
          local_dims[i]=dims[i];
        }
        

            for(int i=0;i<0;i++){//init new strides
              local_str[0][i]=i0_str[i];
            }
            

            for(int i=0;i<0;i++){//init new strides
              local_str[1][i]=i1_str[i];
            }
            

            for(int i=0;i<0;i++){//init new strides
              local_str[2][i]=i2_str[i];
            }
            

            for(int i=0;i<0;i++){//init new strides
              local_str[3][i]=i3_str[i];
            }
            

            for(int i=0;i<0;i++){//init new strides
              local_str[4][i]=i4_str[i];
            }
            

            for(int i=0;i<0;i++){//init new strides
              local_str[5][i]=i5_str[i];
            }
            

            for(int i=0;i<0;i++){//init new strides
              local_str[6][i]=i6_str[i];
            }
            

            for(int i=0;i<0;i++){//init new strides
              local_ostr[0][i]=o0_str[i];
            }
            

        for(int id=0;id<nd_collapse;id++){

          bool all_broadcast=true;
          for(int input_id=0;input_id<7;input_id++){
            if(local_str[input_id][id]!=0 || local_dims[id]!=1) all_broadcast= false;
          }
          for(int input_id=0;input_id<1;input_id++){
            if(local_ostr[input_id][id]!=0 || local_dims[id]!=1) all_broadcast= false;
          }
          if(all_broadcast){
            for(int j=id+1;j<nd_collapse;j++)//remove dims i from the array
              local_dims[j-1]=local_dims[j];
            for(int input_id=0;input_id<7;input_id++){
              for(int j=id+1;j<nd_collapse;j++){//remove dims i from the array
                local_str[input_id][j-1]=local_str[input_id][j];
              }
            }
            for(int output_id=0;output_id<1;output_id++){
              for(int j=id+1;j<nd_collapse;j++){//remove dims i from the array
                local_ostr[output_id][j-1]=local_ostr[output_id][j];
              }
            }
            nd_collapse--; id--;
          }
        }
        
int *nd_collapse_ = NULL;

            for(int i=nd_collapse-1;i>0;i--){
              if(nd_collapse_[i]==1){
                local_str[0][i-1]=local_str[0][i];//set new strides
                for(int j=i+1;j<nd_collapse;j++)//remove stride i from the array
                  local_str[0][j-1]=local_str[0][j];
                }
            }
            

            for(int i=nd_collapse-1;i>0;i--){
              if(nd_collapse_[i]==1){
                local_str[1][i-1]=local_str[1][i];//set new strides
                for(int j=i+1;j<nd_collapse;j++)//remove stride i from the array
                  local_str[1][j-1]=local_str[1][j];
                }
            }
            

            for(int i=nd_collapse-1;i>0;i--){
              if(nd_collapse_[i]==1){
                local_str[2][i-1]=local_str[2][i];//set new strides
                for(int j=i+1;j<nd_collapse;j++)//remove stride i from the array
                  local_str[2][j-1]=local_str[2][j];
                }
            }
            

            for(int i=nd_collapse-1;i>0;i--){
              if(nd_collapse_[i]==1){
                local_str[3][i-1]=local_str[3][i];//set new strides
                for(int j=i+1;j<nd_collapse;j++)//remove stride i from the array
                  local_str[3][j-1]=local_str[3][j];
                }
            }
            

            for(int i=nd_collapse-1;i>0;i--){
              if(nd_collapse_[i]==1){
                local_str[4][i-1]=local_str[4][i];//set new strides
                for(int j=i+1;j<nd_collapse;j++)//remove stride i from the array
                  local_str[4][j-1]=local_str[4][j];
                }
            }
            

            for(int i=nd_collapse-1;i>0;i--){
              if(nd_collapse_[i]==1){
                local_str[5][i-1]=local_str[5][i];//set new strides
                for(int j=i+1;j<nd_collapse;j++)//remove stride i from the array
                  local_str[5][j-1]=local_str[5][j];
                }
            }
            

            for(int i=nd_collapse-1;i>0;i--){
              if(nd_collapse_[i]==1){
                local_str[6][i-1]=local_str[6][i];//set new strides
                for(int j=i+1;j<nd_collapse;j++)//remove stride i from the array
                  local_str[6][j-1]=local_str[6][j];
                }
            }
            

            for(int i=nd_collapse-1;i>0;i--){
              if(nd_collapse_[i]==1){
                local_ostr[0][i-1]=local_ostr[0][i];//set new strides
                for(int j=i+1;j<nd_collapse;j++)//remove stride i from the array
                  local_ostr[0][j-1]=local_ostr[0][j];
                }
            }
            

        for(int i=nd_collapse-1;i>0;i--){
          if(nd_collapse_[i]==1){
            local_dims[i-1]*=local_dims[i];//set new dims
            for(int j=i+1;j<nd_collapse;j++)//remove dims i from the array
              local_dims[j-1]=local_dims[j];
          }
        }
        

        for(int i=1, end=nd_collapse;i<end;i++){
          if(nd_collapse_[i]==1)nd_collapse--;
        }
        if(nd_collapse == 1 
){nd_collapse=0;} 
if(numEls==0) return 0;
switch (nd_collapse==0?0:min(0,nd_collapse)) {
case 0: {

                //first use at least a full warp
                int threads_per_block = std::min(numEls,  (unsigned int)32); //WARP SIZE

                //next start adding multiprocessors
                int n_blocks = std::min(numEls/threads_per_block + (numEls % threads_per_block?1:0), (unsigned int)30); // UP TO NUMBER OF MULTIPROCESSORS

                // next start adding more warps per multiprocessor
                if (threads_per_block * n_blocks < numEls)
                    threads_per_block = std::min(numEls/n_blocks, (unsigned int)NUM_VECTOR_OP_THREADS_PER_BLOCK);
                kernel_Composite_node_a75fe51b78cd45e598008b19e82e1aec_0_Ccontiguous<<<n_blocks, threads_per_block>>>(numEls, i0_data, i1_data, i2_data, i3_data, i4_data, i5_data, i6_data, o0_data);

                //std::cerr << "calling callkernel returned\n";
                

                CNDA_THREAD_SYNC;
                hipError_t err = hipGetLastError();
                if( hipSuccess != err)
                {
                    PyErr_Format(PyExc_RuntimeError, "Cuda error: %s: %s.\n    n_blocks=%i threads_per_block=%i\n   Call: %s\n",
                         "GpuElemwise node_a75fe51b78cd45e598008b19e82e1aec_0 Composite", hipGetErrorString(err),
                         n_blocks, threads_per_block,
                         "kernel_Composite_node_a75fe51b78cd45e598008b19e82e1aec_0_Ccontiguous<<<n_blocks, threads_per_block>>>(numEls, i0_data, i1_data, i2_data, i3_data, i4_data, i5_data, i6_data, o0_data)");
                    return -1;

                }
                
                return 0;
                
        } break;
}
return -2;
}


    namespace {
    struct __struct_compiled_op_a75fe51b78cd45e598008b19e82e1aec {
        PyObject* __ERROR;

        PyObject* storage_V3;
PyObject* storage_V5;
PyObject* storage_V7;
PyObject* storage_V9;
PyObject* storage_V11;
PyObject* storage_V13;
PyObject* storage_V15;
PyObject* storage_V1;
        

        __struct_compiled_op_a75fe51b78cd45e598008b19e82e1aec() {
            // This is only somewhat safe because we:
            //  1) Are not a virtual class
            //  2) Do not use any virtual classes in the members
            //  3) Deal with mostly POD and pointers

            // If this changes, we would have to revise this, but for
            // now I am tired of chasing segfaults because
            // initialization code had an error and some pointer has
            // a junk value.
            memset(this, 0, sizeof(*this));
        }
        ~__struct_compiled_op_a75fe51b78cd45e598008b19e82e1aec(void) {
            cleanup();
        }

        int init(PyObject* __ERROR, PyObject* storage_V3, PyObject* storage_V5, PyObject* storage_V7, PyObject* storage_V9, PyObject* storage_V11, PyObject* storage_V13, PyObject* storage_V15, PyObject* storage_V1) {
            Py_XINCREF(storage_V3);
Py_XINCREF(storage_V5);
Py_XINCREF(storage_V7);
Py_XINCREF(storage_V9);
Py_XINCREF(storage_V11);
Py_XINCREF(storage_V13);
Py_XINCREF(storage_V15);
Py_XINCREF(storage_V1);
            this->storage_V3 = storage_V3;
this->storage_V5 = storage_V5;
this->storage_V7 = storage_V7;
this->storage_V9 = storage_V9;
this->storage_V11 = storage_V11;
this->storage_V13 = storage_V13;
this->storage_V15 = storage_V15;
this->storage_V1 = storage_V1;
            









            this->__ERROR = __ERROR;
            return 0;
        }
        void cleanup(void) {
            __label_1:

double __DUMMY_1;
__label_3:

double __DUMMY_3;
__label_5:

double __DUMMY_5;
__label_7:

double __DUMMY_7;
__label_9:

double __DUMMY_9;
__label_11:

double __DUMMY_11;
__label_13:

double __DUMMY_13;
__label_15:

double __DUMMY_15;
__label_18:

double __DUMMY_18;

            Py_XDECREF(this->storage_V3);
Py_XDECREF(this->storage_V5);
Py_XDECREF(this->storage_V7);
Py_XDECREF(this->storage_V9);
Py_XDECREF(this->storage_V11);
Py_XDECREF(this->storage_V13);
Py_XDECREF(this->storage_V15);
Py_XDECREF(this->storage_V1);
        }
        int run(void) {
            int __failure = 0;
            
    PyObject* py_V1;
     CudaNdarray * V1;
    PyObject* py_V3;
     CudaNdarray * V3;
    PyObject* py_V5;
     CudaNdarray * V5;
    PyObject* py_V7;
     CudaNdarray * V7;
    PyObject* py_V9;
     CudaNdarray * V9;
    PyObject* py_V11;
     CudaNdarray * V11;
    PyObject* py_V13;
     CudaNdarray * V13;
    PyObject* py_V15;
     CudaNdarray * V15;
{

    py_V1 = PyList_GET_ITEM(storage_V1, 0);
    {Py_XINCREF(py_V1);}
    
        if (py_V1 == Py_None)
        {
            V1 = NULL;
        }
        else
        {
            
        assert(py_V1->ob_refcnt >= 2); // There should be at least one ref from the container object,
        // and one ref from the local scope.

        if (CudaNdarray_Check(py_V1))
        {
            //fprintf(stderr, "c_extract CNDA object w refcnt %p %i\n", py_V1, (py_V1->ob_refcnt));
            V1 = (CudaNdarray*)py_V1;
            //std::cerr << "c_extract " << V1 << '\n';
        

                if (V1->nd != 0)
                {
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has rank %i, it was supposed to have rank 0",
                                 V1->nd);
                    V1 = NULL;
                    {
        __failure = 2;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_2;};
                }
                //std::cerr << "c_extract " << V1 << " nd check passed\n";
            

                assert(V1);
                Py_INCREF(py_V1);
            }
            else if (py_V1 == Py_None)
            {
                PyErr_SetString(PyExc_TypeError,
                                "expected a CudaNdarray, not None");
                V1 = NULL;
                {
        __failure = 2;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_2;};
            }
            else
            {
                //fprintf(stderr, "FAILING c_extract CNDA object w refcnt %p %i\n", py_V1, (py_V1->ob_refcnt));
                PyErr_SetString(PyExc_TypeError, "Argument not a CudaNdarray");
                V1 = NULL;
                {
        __failure = 2;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_2;};
            }
            //std::cerr << "c_extract done " << V1 << '\n';
            

        }
        
{

    py_V3 = PyList_GET_ITEM(storage_V3, 0);
    {Py_XINCREF(py_V3);}
    
        assert(py_V3->ob_refcnt >= 2); // There should be at least one ref from the container object,
        // and one ref from the local scope.

        if (CudaNdarray_Check(py_V3))
        {
            //fprintf(stderr, "c_extract CNDA object w refcnt %p %i\n", py_V3, (py_V3->ob_refcnt));
            V3 = (CudaNdarray*)py_V3;
            //std::cerr << "c_extract " << V3 << '\n';
        

                if (V3->nd != 0)
                {
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has rank %i, it was supposed to have rank 0",
                                 V3->nd);
                    V3 = NULL;
                    {
        __failure = 4;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_4;};
                }
                //std::cerr << "c_extract " << V3 << " nd check passed\n";
            

                assert(V3);
                Py_INCREF(py_V3);
            }
            else if (py_V3 == Py_None)
            {
                PyErr_SetString(PyExc_TypeError,
                                "expected a CudaNdarray, not None");
                V3 = NULL;
                {
        __failure = 4;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_4;};
            }
            else
            {
                //fprintf(stderr, "FAILING c_extract CNDA object w refcnt %p %i\n", py_V3, (py_V3->ob_refcnt));
                PyErr_SetString(PyExc_TypeError, "Argument not a CudaNdarray");
                V3 = NULL;
                {
        __failure = 4;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_4;};
            }
            //std::cerr << "c_extract done " << V3 << '\n';
            

{

    py_V5 = PyList_GET_ITEM(storage_V5, 0);
    {Py_XINCREF(py_V5);}
    
        assert(py_V5->ob_refcnt >= 2); // There should be at least one ref from the container object,
        // and one ref from the local scope.

        if (CudaNdarray_Check(py_V5))
        {
            //fprintf(stderr, "c_extract CNDA object w refcnt %p %i\n", py_V5, (py_V5->ob_refcnt));
            V5 = (CudaNdarray*)py_V5;
            //std::cerr << "c_extract " << V5 << '\n';
        

                if (V5->nd != 0)
                {
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has rank %i, it was supposed to have rank 0",
                                 V5->nd);
                    V5 = NULL;
                    {
        __failure = 6;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_6;};
                }
                //std::cerr << "c_extract " << V5 << " nd check passed\n";
            

                assert(V5);
                Py_INCREF(py_V5);
            }
            else if (py_V5 == Py_None)
            {
                PyErr_SetString(PyExc_TypeError,
                                "expected a CudaNdarray, not None");
                V5 = NULL;
                {
        __failure = 6;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_6;};
            }
            else
            {
                //fprintf(stderr, "FAILING c_extract CNDA object w refcnt %p %i\n", py_V5, (py_V5->ob_refcnt));
                PyErr_SetString(PyExc_TypeError, "Argument not a CudaNdarray");
                V5 = NULL;
                {
        __failure = 6;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_6;};
            }
            //std::cerr << "c_extract done " << V5 << '\n';
            

{

    py_V7 = PyList_GET_ITEM(storage_V7, 0);
    {Py_XINCREF(py_V7);}
    
        assert(py_V7->ob_refcnt >= 2); // There should be at least one ref from the container object,
        // and one ref from the local scope.

        if (CudaNdarray_Check(py_V7))
        {
            //fprintf(stderr, "c_extract CNDA object w refcnt %p %i\n", py_V7, (py_V7->ob_refcnt));
            V7 = (CudaNdarray*)py_V7;
            //std::cerr << "c_extract " << V7 << '\n';
        

                if (V7->nd != 0)
                {
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has rank %i, it was supposed to have rank 0",
                                 V7->nd);
                    V7 = NULL;
                    {
        __failure = 8;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_8;};
                }
                //std::cerr << "c_extract " << V7 << " nd check passed\n";
            

                assert(V7);
                Py_INCREF(py_V7);
            }
            else if (py_V7 == Py_None)
            {
                PyErr_SetString(PyExc_TypeError,
                                "expected a CudaNdarray, not None");
                V7 = NULL;
                {
        __failure = 8;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_8;};
            }
            else
            {
                //fprintf(stderr, "FAILING c_extract CNDA object w refcnt %p %i\n", py_V7, (py_V7->ob_refcnt));
                PyErr_SetString(PyExc_TypeError, "Argument not a CudaNdarray");
                V7 = NULL;
                {
        __failure = 8;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_8;};
            }
            //std::cerr << "c_extract done " << V7 << '\n';
            

{

    py_V9 = PyList_GET_ITEM(storage_V9, 0);
    {Py_XINCREF(py_V9);}
    
        assert(py_V9->ob_refcnt >= 2); // There should be at least one ref from the container object,
        // and one ref from the local scope.

        if (CudaNdarray_Check(py_V9))
        {
            //fprintf(stderr, "c_extract CNDA object w refcnt %p %i\n", py_V9, (py_V9->ob_refcnt));
            V9 = (CudaNdarray*)py_V9;
            //std::cerr << "c_extract " << V9 << '\n';
        

                if (V9->nd != 0)
                {
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has rank %i, it was supposed to have rank 0",
                                 V9->nd);
                    V9 = NULL;
                    {
        __failure = 10;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_10;};
                }
                //std::cerr << "c_extract " << V9 << " nd check passed\n";
            

                assert(V9);
                Py_INCREF(py_V9);
            }
            else if (py_V9 == Py_None)
            {
                PyErr_SetString(PyExc_TypeError,
                                "expected a CudaNdarray, not None");
                V9 = NULL;
                {
        __failure = 10;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_10;};
            }
            else
            {
                //fprintf(stderr, "FAILING c_extract CNDA object w refcnt %p %i\n", py_V9, (py_V9->ob_refcnt));
                PyErr_SetString(PyExc_TypeError, "Argument not a CudaNdarray");
                V9 = NULL;
                {
        __failure = 10;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_10;};
            }
            //std::cerr << "c_extract done " << V9 << '\n';
            

{

    py_V11 = PyList_GET_ITEM(storage_V11, 0);
    {Py_XINCREF(py_V11);}
    
        assert(py_V11->ob_refcnt >= 2); // There should be at least one ref from the container object,
        // and one ref from the local scope.

        if (CudaNdarray_Check(py_V11))
        {
            //fprintf(stderr, "c_extract CNDA object w refcnt %p %i\n", py_V11, (py_V11->ob_refcnt));
            V11 = (CudaNdarray*)py_V11;
            //std::cerr << "c_extract " << V11 << '\n';
        

                if (V11->nd != 0)
                {
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has rank %i, it was supposed to have rank 0",
                                 V11->nd);
                    V11 = NULL;
                    {
        __failure = 12;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_12;};
                }
                //std::cerr << "c_extract " << V11 << " nd check passed\n";
            

                assert(V11);
                Py_INCREF(py_V11);
            }
            else if (py_V11 == Py_None)
            {
                PyErr_SetString(PyExc_TypeError,
                                "expected a CudaNdarray, not None");
                V11 = NULL;
                {
        __failure = 12;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_12;};
            }
            else
            {
                //fprintf(stderr, "FAILING c_extract CNDA object w refcnt %p %i\n", py_V11, (py_V11->ob_refcnt));
                PyErr_SetString(PyExc_TypeError, "Argument not a CudaNdarray");
                V11 = NULL;
                {
        __failure = 12;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_12;};
            }
            //std::cerr << "c_extract done " << V11 << '\n';
            

{

    py_V13 = PyList_GET_ITEM(storage_V13, 0);
    {Py_XINCREF(py_V13);}
    
        assert(py_V13->ob_refcnt >= 2); // There should be at least one ref from the container object,
        // and one ref from the local scope.

        if (CudaNdarray_Check(py_V13))
        {
            //fprintf(stderr, "c_extract CNDA object w refcnt %p %i\n", py_V13, (py_V13->ob_refcnt));
            V13 = (CudaNdarray*)py_V13;
            //std::cerr << "c_extract " << V13 << '\n';
        

                if (V13->nd != 0)
                {
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has rank %i, it was supposed to have rank 0",
                                 V13->nd);
                    V13 = NULL;
                    {
        __failure = 14;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_14;};
                }
                //std::cerr << "c_extract " << V13 << " nd check passed\n";
            

                assert(V13);
                Py_INCREF(py_V13);
            }
            else if (py_V13 == Py_None)
            {
                PyErr_SetString(PyExc_TypeError,
                                "expected a CudaNdarray, not None");
                V13 = NULL;
                {
        __failure = 14;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_14;};
            }
            else
            {
                //fprintf(stderr, "FAILING c_extract CNDA object w refcnt %p %i\n", py_V13, (py_V13->ob_refcnt));
                PyErr_SetString(PyExc_TypeError, "Argument not a CudaNdarray");
                V13 = NULL;
                {
        __failure = 14;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_14;};
            }
            //std::cerr << "c_extract done " << V13 << '\n';
            

{

    py_V15 = PyList_GET_ITEM(storage_V15, 0);
    {Py_XINCREF(py_V15);}
    
        assert(py_V15->ob_refcnt >= 2); // There should be at least one ref from the container object,
        // and one ref from the local scope.

        if (CudaNdarray_Check(py_V15))
        {
            //fprintf(stderr, "c_extract CNDA object w refcnt %p %i\n", py_V15, (py_V15->ob_refcnt));
            V15 = (CudaNdarray*)py_V15;
            //std::cerr << "c_extract " << V15 << '\n';
        

                if (V15->nd != 0)
                {
                    PyErr_Format(PyExc_RuntimeError,
                                 "c_extract: Some CudaNdarray has rank %i, it was supposed to have rank 0",
                                 V15->nd);
                    V15 = NULL;
                    {
        __failure = 16;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_16;};
                }
                //std::cerr << "c_extract " << V15 << " nd check passed\n";
            

                assert(V15);
                Py_INCREF(py_V15);
            }
            else if (py_V15 == Py_None)
            {
                PyErr_SetString(PyExc_TypeError,
                                "expected a CudaNdarray, not None");
                V15 = NULL;
                {
        __failure = 16;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_16;};
            }
            else
            {
                //fprintf(stderr, "FAILING c_extract CNDA object w refcnt %p %i\n", py_V15, (py_V15->ob_refcnt));
                PyErr_SetString(PyExc_TypeError, "Argument not a CudaNdarray");
                V15 = NULL;
                {
        __failure = 16;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_16;};
            }
            //std::cerr << "c_extract done " << V15 << '\n';
            

{
// Op class GpuElemwise

        //std::cerr << "C_CODE Composite{((i0 * sqrt(clip((i1 - (i2 ** i3)), i4, i5))) / (i1 - (i6 ** i3)))} START\n";
        //standard elemwise size checks
            

            int *dims = NULL;
            

                int *broadcasts_V3 = NULL;
                

                int *broadcasts_V5 = NULL;
                

                int *broadcasts_V7 = NULL;
                

                int *broadcasts_V9 = NULL;
                

                int *broadcasts_V11 = NULL;
                

                int *broadcasts_V13 = NULL;
                

                int *broadcasts_V15 = NULL;
                

        //std::cerr << "C_CODE Composite{((i0 * sqrt(clip((i1 - (i2 ** i3)), i4, i5))) / (i1 - (i6 ** i3)))} checking input V3\n";
        if (0 != V3->nd)
        {
            PyErr_Format(PyExc_TypeError,
                         "need 0 dims, not %i", V3->nd);
            {
        __failure = 17;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_17;};
        }
        for (int i = 0; i< 0; ++i)
        {
            dims[i] = (dims[i] == 1) ? CudaNdarray_HOST_DIMS(V3)[i] : dims[i];
            if ((!(broadcasts_V3[i] &&
                 CudaNdarray_HOST_DIMS(V3)[i] == 1)) &&
                (dims[i] != CudaNdarray_HOST_DIMS(V3)[i]))
            {
                //std::cerr << "C_CODE Composite{((i0 * sqrt(clip((i1 - (i2 ** i3)), i4, i5))) / (i1 - (i6 ** i3)))} checking input V3 failed\n";
                PyErr_Format(PyExc_ValueError,
                             "GpuElemwise. Input dimension mis-match. Input"
                             " 0 (indices start at 0) has shape[%i] == %i"
                             ", but the output's size on that axis is %i.",
                             i,
                             CudaNdarray_HOST_DIMS(V3)[i],
                             dims[i]
                            );
                {
        __failure = 17;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_17;};
            }
        }
            

        //std::cerr << "C_CODE Composite{((i0 * sqrt(clip((i1 - (i2 ** i3)), i4, i5))) / (i1 - (i6 ** i3)))} checking input V5\n";
        if (0 != V5->nd)
        {
            PyErr_Format(PyExc_TypeError,
                         "need 0 dims, not %i", V5->nd);
            {
        __failure = 17;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_17;};
        }
        for (int i = 0; i< 0; ++i)
        {
            dims[i] = (dims[i] == 1) ? CudaNdarray_HOST_DIMS(V5)[i] : dims[i];
            if ((!(broadcasts_V5[i] &&
                 CudaNdarray_HOST_DIMS(V5)[i] == 1)) &&
                (dims[i] != CudaNdarray_HOST_DIMS(V5)[i]))
            {
                //std::cerr << "C_CODE Composite{((i0 * sqrt(clip((i1 - (i2 ** i3)), i4, i5))) / (i1 - (i6 ** i3)))} checking input V5 failed\n";
                PyErr_Format(PyExc_ValueError,
                             "GpuElemwise. Input dimension mis-match. Input"
                             " 1 (indices start at 0) has shape[%i] == %i"
                             ", but the output's size on that axis is %i.",
                             i,
                             CudaNdarray_HOST_DIMS(V5)[i],
                             dims[i]
                            );
                {
        __failure = 17;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_17;};
            }
        }
            

        //std::cerr << "C_CODE Composite{((i0 * sqrt(clip((i1 - (i2 ** i3)), i4, i5))) / (i1 - (i6 ** i3)))} checking input V7\n";
        if (0 != V7->nd)
        {
            PyErr_Format(PyExc_TypeError,
                         "need 0 dims, not %i", V7->nd);
            {
        __failure = 17;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_17;};
        }
        for (int i = 0; i< 0; ++i)
        {
            dims[i] = (dims[i] == 1) ? CudaNdarray_HOST_DIMS(V7)[i] : dims[i];
            if ((!(broadcasts_V7[i] &&
                 CudaNdarray_HOST_DIMS(V7)[i] == 1)) &&
                (dims[i] != CudaNdarray_HOST_DIMS(V7)[i]))
            {
                //std::cerr << "C_CODE Composite{((i0 * sqrt(clip((i1 - (i2 ** i3)), i4, i5))) / (i1 - (i6 ** i3)))} checking input V7 failed\n";
                PyErr_Format(PyExc_ValueError,
                             "GpuElemwise. Input dimension mis-match. Input"
                             " 2 (indices start at 0) has shape[%i] == %i"
                             ", but the output's size on that axis is %i.",
                             i,
                             CudaNdarray_HOST_DIMS(V7)[i],
                             dims[i]
                            );
                {
        __failure = 17;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_17;};
            }
        }
            

        //std::cerr << "C_CODE Composite{((i0 * sqrt(clip((i1 - (i2 ** i3)), i4, i5))) / (i1 - (i6 ** i3)))} checking input V9\n";
        if (0 != V9->nd)
        {
            PyErr_Format(PyExc_TypeError,
                         "need 0 dims, not %i", V9->nd);
            {
        __failure = 17;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_17;};
        }
        for (int i = 0; i< 0; ++i)
        {
            dims[i] = (dims[i] == 1) ? CudaNdarray_HOST_DIMS(V9)[i] : dims[i];
            if ((!(broadcasts_V9[i] &&
                 CudaNdarray_HOST_DIMS(V9)[i] == 1)) &&
                (dims[i] != CudaNdarray_HOST_DIMS(V9)[i]))
            {
                //std::cerr << "C_CODE Composite{((i0 * sqrt(clip((i1 - (i2 ** i3)), i4, i5))) / (i1 - (i6 ** i3)))} checking input V9 failed\n";
                PyErr_Format(PyExc_ValueError,
                             "GpuElemwise. Input dimension mis-match. Input"
                             " 3 (indices start at 0) has shape[%i] == %i"
                             ", but the output's size on that axis is %i.",
                             i,
                             CudaNdarray_HOST_DIMS(V9)[i],
                             dims[i]
                            );
                {
        __failure = 17;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_17;};
            }
        }
            

        //std::cerr << "C_CODE Composite{((i0 * sqrt(clip((i1 - (i2 ** i3)), i4, i5))) / (i1 - (i6 ** i3)))} checking input V11\n";
        if (0 != V11->nd)
        {
            PyErr_Format(PyExc_TypeError,
                         "need 0 dims, not %i", V11->nd);
            {
        __failure = 17;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_17;};
        }
        for (int i = 0; i< 0; ++i)
        {
            dims[i] = (dims[i] == 1) ? CudaNdarray_HOST_DIMS(V11)[i] : dims[i];
            if ((!(broadcasts_V11[i] &&
                 CudaNdarray_HOST_DIMS(V11)[i] == 1)) &&
                (dims[i] != CudaNdarray_HOST_DIMS(V11)[i]))
            {
                //std::cerr << "C_CODE Composite{((i0 * sqrt(clip((i1 - (i2 ** i3)), i4, i5))) / (i1 - (i6 ** i3)))} checking input V11 failed\n";
                PyErr_Format(PyExc_ValueError,
                             "GpuElemwise. Input dimension mis-match. Input"
                             " 4 (indices start at 0) has shape[%i] == %i"
                             ", but the output's size on that axis is %i.",
                             i,
                             CudaNdarray_HOST_DIMS(V11)[i],
                             dims[i]
                            );
                {
        __failure = 17;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_17;};
            }
        }
            

        //std::cerr << "C_CODE Composite{((i0 * sqrt(clip((i1 - (i2 ** i3)), i4, i5))) / (i1 - (i6 ** i3)))} checking input V13\n";
        if (0 != V13->nd)
        {
            PyErr_Format(PyExc_TypeError,
                         "need 0 dims, not %i", V13->nd);
            {
        __failure = 17;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_17;};
        }
        for (int i = 0; i< 0; ++i)
        {
            dims[i] = (dims[i] == 1) ? CudaNdarray_HOST_DIMS(V13)[i] : dims[i];
            if ((!(broadcasts_V13[i] &&
                 CudaNdarray_HOST_DIMS(V13)[i] == 1)) &&
                (dims[i] != CudaNdarray_HOST_DIMS(V13)[i]))
            {
                //std::cerr << "C_CODE Composite{((i0 * sqrt(clip((i1 - (i2 ** i3)), i4, i5))) / (i1 - (i6 ** i3)))} checking input V13 failed\n";
                PyErr_Format(PyExc_ValueError,
                             "GpuElemwise. Input dimension mis-match. Input"
                             " 5 (indices start at 0) has shape[%i] == %i"
                             ", but the output's size on that axis is %i.",
                             i,
                             CudaNdarray_HOST_DIMS(V13)[i],
                             dims[i]
                            );
                {
        __failure = 17;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_17;};
            }
        }
            

        //std::cerr << "C_CODE Composite{((i0 * sqrt(clip((i1 - (i2 ** i3)), i4, i5))) / (i1 - (i6 ** i3)))} checking input V15\n";
        if (0 != V15->nd)
        {
            PyErr_Format(PyExc_TypeError,
                         "need 0 dims, not %i", V15->nd);
            {
        __failure = 17;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_17;};
        }
        for (int i = 0; i< 0; ++i)
        {
            dims[i] = (dims[i] == 1) ? CudaNdarray_HOST_DIMS(V15)[i] : dims[i];
            if ((!(broadcasts_V15[i] &&
                 CudaNdarray_HOST_DIMS(V15)[i] == 1)) &&
                (dims[i] != CudaNdarray_HOST_DIMS(V15)[i]))
            {
                //std::cerr << "C_CODE Composite{((i0 * sqrt(clip((i1 - (i2 ** i3)), i4, i5))) / (i1 - (i6 ** i3)))} checking input V15 failed\n";
                PyErr_Format(PyExc_ValueError,
                             "GpuElemwise. Input dimension mis-match. Input"
                             " 6 (indices start at 0) has shape[%i] == %i"
                             ", but the output's size on that axis is %i.",
                             i,
                             CudaNdarray_HOST_DIMS(V15)[i],
                             dims[i]
                            );
                {
        __failure = 17;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_17;};
            }
        }
            

        for (int i = 0; (i< 0) && (V1); ++i) {
            if (dims[i] != CudaNdarray_HOST_DIMS(V1)[i])
            {
                Py_DECREF(V1);
                V1 = NULL;
            }
        }
        if (V1 && !CudaNdarray_is_c_contiguous(V1))
        {
            Py_XDECREF(V1);
            V1 = NULL;
        }
        if (NULL == V1)
        {
            V1 = (CudaNdarray*)CudaNdarray_New();
            if (!V1)
            {
                //error string already set
                {
        __failure = 17;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_17;};
            }
            if (CudaNdarray_alloc_contiguous(V1, 0, dims))
            {
                //error string already set
                Py_DECREF(V1);
                V1 = NULL;
                {
        __failure = 17;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_17;};
            }
        }
        //std::cerr << "ELEMWISE NEW V1 nd" << V1->nd << "\n";
        //std::cerr << "ELEMWISE NEW V1 data" << V1->devdata << "\n";
        

        {
            //new block so that failure gotos don't skip over variable initialization
            //std::cerr << "calling callkernel\n";
            if (callkernel_node_a75fe51b78cd45e598008b19e82e1aec_0(1, 0, dims
            

                        , CudaNdarray_DEV_DATA(V3), CudaNdarray_HOST_STRIDES(V3)
            

                        , CudaNdarray_DEV_DATA(V5), CudaNdarray_HOST_STRIDES(V5)
            

                        , CudaNdarray_DEV_DATA(V7), CudaNdarray_HOST_STRIDES(V7)
            

                        , CudaNdarray_DEV_DATA(V9), CudaNdarray_HOST_STRIDES(V9)
            

                        , CudaNdarray_DEV_DATA(V11), CudaNdarray_HOST_STRIDES(V11)
            

                        , CudaNdarray_DEV_DATA(V13), CudaNdarray_HOST_STRIDES(V13)
            

                        , CudaNdarray_DEV_DATA(V15), CudaNdarray_HOST_STRIDES(V15)
            

                        , CudaNdarray_DEV_DATA(V1), CudaNdarray_HOST_STRIDES(V1)
            

                        ))
            {
                 // error
            

                Py_DECREF(V1);
                V1 = NULL;
                

                {
        __failure = 17;
        if (!PyErr_Occurred()) {
            PyErr_SetString(PyExc_RuntimeError,
                "Unexpected error in an Op's C code. "
                "No Python exception was set.");
            }
        goto __label_17;};
            }
            else // no error
            {
            }
        }
        //std::cerr << "C_CODE Composite{((i0 * sqrt(clip((i1 - (i2 ** i3)), i4, i5))) / (i1 - (i6 ** i3)))} END\n";
        
__label_17:

double __DUMMY_17;

}
__label_16:

        //std::cerr << "cleanup " << py_V15 << " " << V15 << "\n";
        //fprintf(stderr, "c_cleanup CNDA py_object w refcnt %p %i\n", py_V15, (py_V15->ob_refcnt));
        if (V15)
        {
            //fprintf(stderr, "c_cleanup CNDA cn_object w refcnt %p %i\n", V15, (V15->ob_refcnt));
            Py_XDECREF(V15);
        }
        //std::cerr << "cleanup done" << py_V15 << "\n";
        
    {Py_XDECREF(py_V15);}
    
double __DUMMY_16;

}
__label_14:

        //std::cerr << "cleanup " << py_V13 << " " << V13 << "\n";
        //fprintf(stderr, "c_cleanup CNDA py_object w refcnt %p %i\n", py_V13, (py_V13->ob_refcnt));
        if (V13)
        {
            //fprintf(stderr, "c_cleanup CNDA cn_object w refcnt %p %i\n", V13, (V13->ob_refcnt));
            Py_XDECREF(V13);
        }
        //std::cerr << "cleanup done" << py_V13 << "\n";
        
    {Py_XDECREF(py_V13);}
    
double __DUMMY_14;

}
__label_12:

        //std::cerr << "cleanup " << py_V11 << " " << V11 << "\n";
        //fprintf(stderr, "c_cleanup CNDA py_object w refcnt %p %i\n", py_V11, (py_V11->ob_refcnt));
        if (V11)
        {
            //fprintf(stderr, "c_cleanup CNDA cn_object w refcnt %p %i\n", V11, (V11->ob_refcnt));
            Py_XDECREF(V11);
        }
        //std::cerr << "cleanup done" << py_V11 << "\n";
        
    {Py_XDECREF(py_V11);}
    
double __DUMMY_12;

}
__label_10:

        //std::cerr << "cleanup " << py_V9 << " " << V9 << "\n";
        //fprintf(stderr, "c_cleanup CNDA py_object w refcnt %p %i\n", py_V9, (py_V9->ob_refcnt));
        if (V9)
        {
            //fprintf(stderr, "c_cleanup CNDA cn_object w refcnt %p %i\n", V9, (V9->ob_refcnt));
            Py_XDECREF(V9);
        }
        //std::cerr << "cleanup done" << py_V9 << "\n";
        
    {Py_XDECREF(py_V9);}
    
double __DUMMY_10;

}
__label_8:

        //std::cerr << "cleanup " << py_V7 << " " << V7 << "\n";
        //fprintf(stderr, "c_cleanup CNDA py_object w refcnt %p %i\n", py_V7, (py_V7->ob_refcnt));
        if (V7)
        {
            //fprintf(stderr, "c_cleanup CNDA cn_object w refcnt %p %i\n", V7, (V7->ob_refcnt));
            Py_XDECREF(V7);
        }
        //std::cerr << "cleanup done" << py_V7 << "\n";
        
    {Py_XDECREF(py_V7);}
    
double __DUMMY_8;

}
__label_6:

        //std::cerr << "cleanup " << py_V5 << " " << V5 << "\n";
        //fprintf(stderr, "c_cleanup CNDA py_object w refcnt %p %i\n", py_V5, (py_V5->ob_refcnt));
        if (V5)
        {
            //fprintf(stderr, "c_cleanup CNDA cn_object w refcnt %p %i\n", V5, (V5->ob_refcnt));
            Py_XDECREF(V5);
        }
        //std::cerr << "cleanup done" << py_V5 << "\n";
        
    {Py_XDECREF(py_V5);}
    
double __DUMMY_6;

}
__label_4:

        //std::cerr << "cleanup " << py_V3 << " " << V3 << "\n";
        //fprintf(stderr, "c_cleanup CNDA py_object w refcnt %p %i\n", py_V3, (py_V3->ob_refcnt));
        if (V3)
        {
            //fprintf(stderr, "c_cleanup CNDA cn_object w refcnt %p %i\n", V3, (V3->ob_refcnt));
            Py_XDECREF(V3);
        }
        //std::cerr << "cleanup done" << py_V3 << "\n";
        
    {Py_XDECREF(py_V3);}
    
double __DUMMY_4;

}
__label_2:

    if (!__failure) {
      
        //std::cerr << "sync\n";
        if (NULL == V1) {
            // failure: sync None to storage
            Py_XDECREF(py_V1);
            py_V1 = Py_None;
            Py_INCREF(py_V1);
        }
        else
        {
            if (py_V1 != (PyObject*)V1)
            {
                Py_XDECREF(py_V1);
                py_V1 = (PyObject*)V1;
                Py_INCREF(py_V1);
            }
            assert(py_V1->ob_refcnt);
        }
        
      PyObject* old = PyList_GET_ITEM(storage_V1, 0);
      {Py_XINCREF(py_V1);}
      PyList_SET_ITEM(storage_V1, 0, py_V1);
      {Py_XDECREF(old);}
    }
    
        //std::cerr << "cleanup " << py_V1 << " " << V1 << "\n";
        //fprintf(stderr, "c_cleanup CNDA py_object w refcnt %p %i\n", py_V1, (py_V1->ob_refcnt));
        if (V1)
        {
            //fprintf(stderr, "c_cleanup CNDA cn_object w refcnt %p %i\n", V1, (V1->ob_refcnt));
            Py_XDECREF(V1);
        }
        //std::cerr << "cleanup done" << py_V1 << "\n";
        
    {Py_XDECREF(py_V1);}
    
double __DUMMY_2;

}

            
        if (__failure) {
            // When there is a failure, this code puts the exception
            // in __ERROR.
            PyObject* err_type = NULL;
            PyObject* err_msg = NULL;
            PyObject* err_traceback = NULL;
            PyErr_Fetch(&err_type, &err_msg, &err_traceback);
            if (!err_type) {err_type = Py_None;Py_INCREF(Py_None);}
            if (!err_msg) {err_msg = Py_None; Py_INCREF(Py_None);}
            if (!err_traceback) {err_traceback = Py_None; Py_INCREF(Py_None);}
            PyObject* old_err_type = PyList_GET_ITEM(__ERROR, 0);
            PyObject* old_err_msg = PyList_GET_ITEM(__ERROR, 1);
            PyObject* old_err_traceback = PyList_GET_ITEM(__ERROR, 2);
            PyList_SET_ITEM(__ERROR, 0, err_type);
            PyList_SET_ITEM(__ERROR, 1, err_msg);
            PyList_SET_ITEM(__ERROR, 2, err_traceback);
            {Py_XDECREF(old_err_type);}
            {Py_XDECREF(old_err_msg);}
            {Py_XDECREF(old_err_traceback);}
        }
        // The failure code is returned to index what code block failed.
        return __failure;
        
        }
    };
    }
    

        static int __struct_compiled_op_a75fe51b78cd45e598008b19e82e1aec_executor(__struct_compiled_op_a75fe51b78cd45e598008b19e82e1aec* self) {
            return self->run();
        }

        static void __struct_compiled_op_a75fe51b78cd45e598008b19e82e1aec_destructor(void* executor, void* self) {
            delete ((__struct_compiled_op_a75fe51b78cd45e598008b19e82e1aec*)self);
        }
        
//////////////////////
////  Functions
//////////////////////
static PyObject * instantiate(PyObject * self, PyObject *argtuple) {
  assert(PyTuple_Check(argtuple));
  if (9 != PyTuple_Size(argtuple)){ 
     PyErr_Format(PyExc_TypeError, "Wrong number of arguments, expected 9, got %i", (int)PyTuple_Size(argtuple));
     return NULL;
  }
  __struct_compiled_op_a75fe51b78cd45e598008b19e82e1aec* struct_ptr = new __struct_compiled_op_a75fe51b78cd45e598008b19e82e1aec();
  if (struct_ptr->init( PyTuple_GET_ITEM(argtuple, 0),PyTuple_GET_ITEM(argtuple, 1),PyTuple_GET_ITEM(argtuple, 2),PyTuple_GET_ITEM(argtuple, 3),PyTuple_GET_ITEM(argtuple, 4),PyTuple_GET_ITEM(argtuple, 5),PyTuple_GET_ITEM(argtuple, 6),PyTuple_GET_ITEM(argtuple, 7),PyTuple_GET_ITEM(argtuple, 8) ) != 0) {
    delete struct_ptr;
    return NULL;
  }
  PyObject* thunk = PyCObject_FromVoidPtrAndDesc((void*)(&__struct_compiled_op_a75fe51b78cd45e598008b19e82e1aec_executor), struct_ptr, __struct_compiled_op_a75fe51b78cd45e598008b19e82e1aec_destructor);
  return thunk; }

//////////////////////
////  Module init
//////////////////////
static PyMethodDef MyMethods[] = {
	{"instantiate", instantiate, METH_VARARGS, "undocumented"} ,
	{NULL, NULL, 0, NULL}
};
PyMODINIT_FUNC inita75fe51b78cd45e598008b19e82e1aec(void){
   (void) Py_InitModule("a75fe51b78cd45e598008b19e82e1aec", MyMethods);
}
